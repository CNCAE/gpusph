#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int, char *[]) {
	int numDevs = 0;
#define BUFSIZE 8192
	char namebuf[BUFSIZE];

	hipInit(0);

	if (hipGetDeviceCount(&numDevs) != hipSuccess)
		return 1;
	for (int d = 0; d < numDevs; ++d) {
		hipDevice_t dev;
		int ccmaj, ccmin;
		if (hipDeviceGet(&dev, d) != hipSuccess)
			return d+2;
		if (hipDeviceGetName(namebuf, BUFSIZE, dev) != hipSuccess)
			return 250;
#if CUDA_VERSION < 5000
		if (cuDeviceComputeCapability(&ccmaj, &ccmin, dev) != CUDA_SUCCESS)
			return 251;
#else
		if (hipDeviceGetAttribute(&ccmaj, hipDeviceAttributeComputeCapabilityMajor, dev) != hipSuccess)
			return 251;
		if (hipDeviceGetAttribute(&ccmin, hipDeviceAttributeComputeCapabilityMinor, dev) != hipSuccess)
			return 252;
#endif
		printf("%d\t%d.%d\t%s\n", d, ccmaj, ccmin, namebuf);
	}

	return 0;
}
