#include "hip/hip_runtime.h"
/*  Copyright 2011 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

	Istituto de Nazionale di Geofisica e Vulcanologia
          Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */

#ifndef _FORCES_KERNEL_
#define _FORCES_KERNEL_

#include "particledefine.h"
#include "textures.cuh"

#define GPU_CODE
#include "kahan.h"
#undef GPU_CODE

texture<float, 2, hipReadModeElementType> demTex;	// DEM

namespace cuforces {
__constant__ uint d_maxneibsnum_time_neibindexinterleave;

__constant__ float	d_wcoeff_cubicspline;			// coeff = 1/(Pi h^3)
__constant__ float	d_wcoeff_quadratic;				// coeff = 15/(16 Pi h^3)
__constant__ float	d_wcoeff_wendland;				// coeff = 21/(16 Pi h^3)

__constant__ float	d_fcoeff_cubicspline;			// coeff = 3/(4Pi h^4)
__constant__ float	d_fcoeff_quadratic;				// coeff = 15/(32Pi h^4)
__constant__ float	d_fcoeff_wendland;				// coeff = 105/(128Pi h^5)

__constant__ int    d_numfluids;					// number of different fluids

__constant__ float	d_rho0[MAX_FLUID_TYPES];		// rest density of fluids

// Speed of sound constants
__constant__ float	d_bcoeff[MAX_FLUID_TYPES];
__constant__ float	d_gammacoeff[MAX_FLUID_TYPES];
__constant__ float	d_sscoeff[MAX_FLUID_TYPES];
__constant__ float	d_sspowercoeff[MAX_FLUID_TYPES];

__constant__ float3	d_gravity;						// gravity (vector)

__constant__ float	d_ferrari;				// coefficient for Ferrari correction

// LJ boundary repusion force comuting
__constant__ float	d_dcoeff;
__constant__ float	d_p1coeff;
__constant__ float	d_p2coeff;
__constant__ float	d_r0;

// Monaghan-Kaijar boundary repulsion force constants
// This is typically the square of the maximum velocity, or gravity times the maximum height
__constant__ float	d_MK_K;
// This is typically the distance between boundary particles
__constant__ float	d_MK_d;
// This is typically the ration between h and the distance between boundary particles
__constant__ float	d_MK_beta;

__constant__ float	d_visccoeff;
__constant__ float	d_epsartvisc;

__constant__ float3	d_dispvect;					// displacment vector for periodic boundaries

// Constants used for DEM
__constant__ float	d_ewres;
__constant__ float	d_nsres;
__constant__ float	d_demdx;
__constant__ float	d_demdy;
__constant__ float	d_demdxdy;
__constant__ float	d_demzmin;

__constant__ float	d_partsurf;						// particle surface

// Definition of planes for geometrical boundaries
__constant__ uint	d_numplanes;
__constant__ float4	d_planes[MAXPLANES];
__constant__ float	d_plane_div[MAXPLANES];

// Sub-Particle Scale (SPS) Turbulence parameters
__constant__ float	d_smagfactor;
__constant__ float	d_kspsfactor;

// Free surface detection
__constant__ float	d_cosconeanglefluid;
__constant__ float	d_cosconeanglenonfluid;

// Rigid body data (test version)
__device__ float3	d_force;
__device__ float3	d_torque;
__constant__ float3 d_rbcg[MAXBODIES];
__constant__ uint	d_rbstartindex[MAXBODIES];
__constant__ float d_objectobjectdf;
__constant__ float d_objectboundarydf;


typedef struct sym33mat {
	float a11;
	float a12;
	float a13;
	float a22;
	float a23;
	float a33;
} sym33mat;


/************************************************************************************************************/
/*							  Functions used by the differents CUDA kernels							   */
/************************************************************************************************************/

/********************************************* SPH kernels **************************************************/
// Return kernel value at distance r, for a given smoothing length
template<KernelType kerneltype>
__device__ __forceinline__ float
W(const float r, const float slength);


// Cubic Spline kernel
template<>
__device__ __forceinline__ float
W<CUBICSPLINE>(const float r, const float slength)
{
	float val = 0.0f;
	const float R = r/slength;

	if (R < 1)
		val = 1.0f - 1.5f*R*R + 0.75f*R*R*R;			// val = 1 - 3/2 R^2 + 3/4 R^3
	else
		val = 0.25f*(2.0f - R)*(2.0f - R)*(2.0f - R);	// val = 1/4 (2 - R)^3

	val *= d_wcoeff_cubicspline;						// coeff = 1/(Pi h^3)

	return val;
}


// Qudratic kernel
template<>
__device__ __forceinline__ float
W<QUADRATIC>(const float r, const float slength)
{
	float val = 0.0f;
	const float R = r/slength;

	val = 0.25f*R*R - R + 1.0f;		// val = 1/4 R^2 -  R + 1
	val *= d_wcoeff_quadratic;		// coeff = 15/(16 Pi h^3)

	return val;
}


// Wendland kernel
template<>
__device__ __forceinline__ float
W<WENDLAND>(float r, float slength)
{
	const float R = r/slength;

	float val = 1.0f - 0.5f*R;
	val *= val;
	val *= val;						// val = (1 - R/2)^4
	val *= 1.0f + 2.0f*R;			// val = (2R + 1)(1 - R/2)^4*
	val *= d_wcoeff_wendland;		// coeff = 21/(16 Pi h^3)
	return val;
}


// Return 1/r dW/dr at distance r, for a given smoothing length
template<KernelType kerneltype>
__device__ __forceinline__ float
F(const float r, const float slength);


template<>
__device__ __forceinline__ float
F<CUBICSPLINE>(const float r, const float slength)
{
	float val = 0.0f;
	const float R = r/slength;

	if (R < 1.0f)
		val = (-4.0f + 3.0f*R)/slength;		// val = (-4 + 3R)/h
	else
		val = -(-2.0f + R)*(-2.0f + R)/r;	// val = -(-2 + R)^2/r
	val *= d_fcoeff_cubicspline;			// coeff = 3/(4Pi h^4)

	return val;
}


template<>
__device__ __forceinline__ float
F<QUADRATIC>(const float r, const float slength)
{
	const float R = r/slength;

	float val = (-2.0f + R)/r;		// val = (-2 + R)/r
	val *= d_fcoeff_quadratic;		// coeff = 15/(32Pi h^4)

	return val;
}


template<> 
__device__ __forceinline__ float
F<WENDLAND>(const float r, const float slength)
{
	const float qm2 = r/slength - 2.0f;	// val = (-2 + R)^3
	float val = qm2*qm2*qm2*d_fcoeff_wendland;
	return val;
}
/************************************************************************************************************/


/********************** Equation of state, speed of sound, repulsive force **********************************/
// Equation of state: pressure from density, where i is the fluid kind, not particle_id
__device__ __forceinline__ float
P(const float rho, const uint i)
{
	return d_bcoeff[i]*(__powf(rho/d_rho0[i], d_gammacoeff[i]) - 1);
}

// Inverted equation of state: density from pressure
__device__ __forceinline__ float
rho(const float P, const uint i)
{
	return d_rho0[i]*__powf(P/d_bcoeff[i] + 1, 1.0f/d_gammacoeff[i]);
}

// Sound speed computed from density
__device__ __forceinline__ float
soundSpeed(const float rho, const uint i)
{
	return d_sscoeff[i]*__powf(rho/d_rho0[i], d_sspowercoeff[i]);
}


// Lennard-Jones boundary repulsion force
__device__ __forceinline__ float
LJForce(const float r)
{
	float force = 0.0f;

	if (r <= d_r0)
		force = d_dcoeff*(__powf(d_r0/r, d_p1coeff) - __powf(d_r0/r, d_p2coeff))/(r*r);

	return force;
}

// Monaghan-Kajtar boundary repulsion force doi:10.1016/j.cpc.2009.05.008
// to be multiplied by r_aj vector
// we allow the fluid particle mass mass_f to be different from the
// boundary particle mass mass_b even though they are typically the same
// (except for multi-phase fluids)
__device__ __forceinline__ float
MKForce(const float r, const float slength,
		const float mass_f, const float mass_b)
{
	// MK always uses the 1D cubic or quintic Wendland spline
	float w = 0.0f;

	float force = 0.0f;

	// Wendland has radius 2
	if (r <= 2*slength) {	//TODO: fixme use influenceradius
		float qq = r/slength;
		w = 1.8f * __powf(1.0f - 0.5f*qq, 4.0f) * (2.0f*qq + 1.0f);  //TODO: optimize
		// float dist = r - d_MK_d;
		float dist = max(d_epsartvisc, r - d_MK_d);
		force = d_MK_K*w*2*mass_b/(d_MK_beta * dist * r * (mass_f+mass_b));
	}

	return force;
}
/************************************************************************************************************/


/***************************************** Viscosities *******************************************************/
// Artificial viscosity s
__device__ __forceinline__ float
artvisc(	const float	vel_dot_pos,
			const float	rho,
			const float	neib_rho,
			const float	sspeed,
			const float	neib_sspeed,
			const float	r,
			const float	slength)
{
	return vel_dot_pos*slength*d_visccoeff*(sspeed + neib_sspeed)/
									((r*r + d_epsartvisc)*(rho + neib_rho));
}


// ATTENTION: for all non artificial viscosity
// µ is the dynamic viscosity (ρν)

// Scalar part of viscosity using Morris 1997
// expression 21 p218 when all particles have the same viscosity
// in this case d_visccoeff = 4 nu
// returns 4.mj.nu/(ρi + ρj) (1/r ∂Wij/∂r)
__device__ __forceinline__ float
laminarvisc_kinematic(	const float	rho,
						const float	neib_rho,
						const float	neib_mass,
						const float	f)
{
	return neib_mass*d_visccoeff*f/(rho + neib_rho);
}


// Same behaviour as laminarvisc but for particle
// dependent viscosity.
// returns mj.(µi + µi)/(ρi.ρj) (1/r ∂Wij/∂r)
__device__ __forceinline__ float
laminarvisc_dynamic(const float	rho,
					const float	neib_rho,
					const float	neib_mass,
					const float	f,
					const float	visc,
					const float	neib_visc)
{
	return neib_mass*(visc + neib_visc)*f/(rho*neib_rho);
}
/************************************************************************************************************/


/*********************************** Adptative time stepping ************************************************/
// Function called at the end of the forces or powerlawVisc function doing
// a per block maximum reduction
__device__ __forceinline__ void
dtadaptBlockReduce(	float*	sm_max,
					float*	cfl)
{
	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
	{
		__syncthreads();
		if (threadIdx.x < s) 
		{
			sm_max[threadIdx.x] = max(sm_max[threadIdx.x + s], sm_max[threadIdx.x]);
		}
	}

	// write result for this block to global mem
	if (!threadIdx.x)
		cfl[blockIdx.x] = sm_max[0];
}
/************************************************************************************************************/


/********************************* Periodic boundary management *********************************************/
// Function returning the neigbor index, position, relative distance and velocity
template<bool periodicbound>
__device__ __forceinline__ void
getNeibData(const float4	pos,
			const float		influenceradius,
			uint&			neib_index,
			float4&			neib_pos,
			float3&			relPos,
			float&			r);


// In case of periodic boundaries we add the displacement
// vector when needed
template<>
__device__ __forceinline__ void
getNeibData<true>(	const float4	pos,
					const float		influenceradius,
					uint&			neib_index,
					float4&			neib_pos,
					float3&			relPos,
					float&			r)
{
	int3 periodic = make_int3(0);
	if (neib_index & WARPXPLUS)
		periodic.x = 1;
	else if (neib_index & WARPXMINUS)
		periodic.x = -1;
	if (neib_index & WARPYPLUS)
		periodic.y = 1;
	else if (neib_index & WARPYMINUS)
		periodic.y = -1;
	if (neib_index & WARPZPLUS)
		periodic.z = 1;
	else if (neib_index & WARPZMINUS)
		periodic.z = -1;

	neib_index &= NOWARP;

	neib_pos = tex1Dfetch(posTex, neib_index);

	relPos.x = pos.x - neib_pos.x;
	relPos.y = pos.y - neib_pos.y;
	relPos.z = pos.z - neib_pos.z;
	r = length(relPos);
	if (periodic.x || periodic.y || periodic.z) {
		if (r > influenceradius) {
			relPos += periodic*d_dispvect;
			r = length(relPos);
		}
	}
}


template<>
__device__ __forceinline__ void
getNeibData<false>(	const float4	pos,
					const float		influenceradius,
					uint&			neib_index,
					float4&			neib_pos,
					float3&			relPos,
					float&			r)
{
	neib_pos = tex1Dfetch(posTex, neib_index);

	relPos.x = pos.x - neib_pos.x;
	relPos.y = pos.y - neib_pos.y;
	relPos.z = pos.z - neib_pos.z;
	r = length(relPos);
}

template<bool periodicbound>
__device__ __forceinline__ void
getNeibData(const float4	pos,
			const float4*	posArray,
			const float		influenceradius,
			uint&			neib_index,
			float4&			neib_pos,
			float3&			relPos,
			float&			r);


// In case of periodic boundaries we add the displacement
// vector when needed
template<>
__device__ __forceinline__ void
getNeibData<true>(	const float4	pos,
					const float4*	posArray,
					const float		influenceradius,
					uint&			neib_index,
					float4&			neib_pos,
					float3&			relPos,
					float&			r)
{
	int3 periodic = make_int3(0);
	if (neib_index & WARPXPLUS)
		periodic.x = 1;
	else if (neib_index & WARPXMINUS)
		periodic.x = -1;
	if (neib_index & WARPYPLUS)
		periodic.y = 1;
	else if (neib_index & WARPYMINUS)
		periodic.y = -1;
	if (neib_index & WARPZPLUS)
		periodic.z = 1;
	else if (neib_index & WARPZMINUS)
		periodic.z = -1;

	neib_index &= NOWARP;

	neib_pos = posArray[neib_index];

	relPos.x = pos.x - neib_pos.x;
	relPos.y = pos.y - neib_pos.y;
	relPos.z = pos.z - neib_pos.z;
	r = length(relPos);
	if (periodic.x || periodic.y || periodic.z) {
		if (r > influenceradius) {
			relPos += periodic*d_dispvect;
			r = length(relPos);
		}
	}
}


template<>
__device__ __forceinline__ void
getNeibData<false>(	const float4	pos,
					const float4*	posArray,
					const float		influenceradius,
					uint&			neib_index,
					float4&			neib_pos,
					float3&			relPos,
					float&			r)
{
	neib_pos = posArray[neib_index];

	relPos.x = pos.x - neib_pos.x;
	relPos.y = pos.y - neib_pos.y;
	relPos.z = pos.z - neib_pos.z;
	r = length(relPos);
}
/************************************************************************************************************/


/******************** Functions for computing repulsive force directly from DEM *****************************/
// TODO: check for the maximum timestep

// Normal and viscous force wrt to solid boundary
__device__ __forceinline__ float
PlaneForce(	const float4	pos,
			const float4	plane,
			const float		l,
			const float3	vel,
			const float		dynvisc,
			float4&			force)
{
	const float r = abs(dot(as_float3(pos), as_float3(plane)) + plane.w)/l;
	if (r < d_r0) {
		const float DvDt = LJForce(r);
		// Unitary normal vector of the surface
		const float3 relPos = make_float3(plane)*r/l;

		force.x += DvDt*relPos.x;
		force.y += DvDt*relPos.y;
		force.z += DvDt*relPos.z;

		// normal velocity component
		const float normal = dot(vel, relPos)/r;
		const float3 v_n = normal*relPos/r;
		// tangential velocity component
		const float3 v_t = vel - v_n;

		// f = -µ u/∆n

		// viscosity
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0-r*r)/(pos.w*r);
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0*3/(M_PI*2)-r*r)/(pos.w*r);
		const float coeff = -dynvisc*d_partsurf/(pos.w*r);

		// coeff should not be higher than needed to nil v_t in the maximum allowed dt
		// coefficients are negative, so the smallest in absolute value is the biggest

		/*
		float fmag = length(as_float3(force));
		float coeff2 = -sqrt(fmag/slength)/(d_dtadaptfactor*d_dtadaptfactor);
		if (coeff2 < -d_epsartvisc)
			coeff = max(coeff, coeff2);
			*/

		force.x += coeff*v_t.x;
		force.y += coeff*v_t.y;
		force.z += coeff*v_t.z;

		return -coeff;
	}

	return 0.0f;
}

__device__ __forceinline__ float
GeometryForce(	const float4	pos,
				const float3	vel,
				const float		dynvisc,
				float4&			force)
{
	float coeff_max = 0.0f;
	for (uint i = 0; i < d_numplanes; ++i) {
		float coeff = PlaneForce(pos, d_planes[i], d_plane_div[i], vel, dynvisc, force);
		if (coeff > coeff_max)
			coeff_max = coeff;
	}

	return coeff_max;
}


__device__ __forceinline__ float
DemInterpol(const texture<float, 2, hipReadModeElementType> texref, 
			const float x, 
			const float y)
{
	return tex2D(texref, x/d_ewres + 0.5f, y/d_nsres + 0.5f);
}


__device__ __forceinline__ float
DemLJForce(	const texture<float, 2, hipReadModeElementType> texref,
			const float4	pos,
			const float3	vel,
			const float		dynvisc,
			float4&			force)
{
	const float z0 = DemInterpol(texref, pos.x, pos.y);
	if (pos.z - z0 < d_demzmin) {
		const float z1 = DemInterpol(texref, pos.x + d_demdx, pos.y);
		const float z2 = DemInterpol(texref, pos.x, pos.y + d_demdy);
		const float a = d_demdy*(z0 - z1);
		const float b = d_demdx*(z0 - z2);
		const float c = d_demdxdy;	// demdx*demdy
		const float d = -a*pos.x - b*pos.y - c*z0;
		const float l = sqrt(a*a+b*b+c*c);
		return PlaneForce(pos, make_float4(a, b, c, d), l, vel, dynvisc, force);
	}
	return 0;
}

/************************************************************************************************************/

/************************************************************************************************************/
/*		   Kernels for computing SPS tensor and SPS viscosity												*/
/************************************************************************************************************/

// Compute the Sub-Particle-Stress (SPS) Tensor matrix for all Particles
// WITHOUT Kernel correction
// Procedure:
// (1) compute velocity gradients
// (2) compute turbulent eddy viscosity (non-dynamic)
// (3) compute turbulent shear stresses
// (4) return SPS tensor matrix (tau) divided by rho^2
template<KernelType kerneltype, bool periodicbound>
__global__ void
__launch_bounds__(BLOCK_SIZE_SPS, MIN_BLOCKS_SPS)
SPSstressMatrixDevice(	const float4* posArray,
						float2*		tau0,
						float2*		tau1,
						float2*		tau2,
						const uint*	neibsList,
						const uint	numParticles,
						const float	slength,
						const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// compute SPS matrix only for fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	// read particle data from sorted arrays
	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	const float4 vel = tex1Dfetch(velTex, index);

	// SPS stress matrix elements
	sym33mat tau;
//	tau.a11 = 0.0f;   // tau11 = tau_xx
//	tau.a12 = 0.0f;   // tau12 = tau_xy
//	tau.a13 = 0.0f;   // tau13 = tau_xz
//	tau.a22 = 0.0f;   // tau22 = tau_yy
//	tau.a23 = 0.0f;   // tau23 = tau_yz
//	tau.a33 = 0.0f;   // tau33 = tau_zz

	// Gradients of the the velocity components
	float3 dvx = make_float3(0.0f);
	float3 dvy = make_float3(0.0f);
	float3 dvz = make_float3(0.0f);

	// first loop over all the neighbors for the Velocity Gradients
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		#if( __COMPUTE__ >= 20)							
		getNeibData<periodicbound>(pos, posArray, influenceradius, neib_index, neib_pos, relPos, r);
		#else
		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		#endif
		const float4 neib_vel = tex1Dfetch(velTex, neib_index);
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && FLUID(neib_info)) {
			const float f = F<kerneltype>(r, slength)*neib_pos.w/neib_vel.w;	// 1/r ∂Wij/∂r Vj

			float3 relVel;
			relVel.x = vel.x - neib_vel.x;
			relVel.y = vel.y - neib_vel.y;
			relVel.z = vel.z - neib_vel.z;

			// Velocity Gradients
			dvx -= relVel.x*relPos*f;	// dvx = -∑mj/ρj vxij (ri - rj)/r ∂Wij/∂r
			dvy -= relVel.y*relPos*f;	// dvy = -∑mj/ρj vyij (ri - rj)/r ∂Wij/∂r
			dvz -= relVel.y*relPos*f;	// dvy = -∑mj/ρj vzij (ri - rj)/r ∂Wij/∂r
			}
		} // end of loop through neighbors

	// Calculate Sub-Particle Scale viscosity
	// and special turbulent terms
	float SijSij_bytwo = 2.0f*(dvx.x*dvx.x + dvy.y*dvy.y + dvz.z*dvz.z);	// 2*SijSij = 2.0((∂vx/∂x)^2 + (∂vy/∂yx)^2 + (∂vz/∂z)^2)
	float temp = dvx.y + dvy.x;		// 2*SijSij += (∂vx/∂y + ∂vy/∂x)^2
	tau.a12 = temp;
	SijSij_bytwo += temp*temp;
	temp = dvx.z + dvz.x;			// 2*SijSij += (∂vx/∂z + ∂vz/∂x)^2
	tau.a13 = temp;
	SijSij_bytwo += temp*temp;
	temp = dvy.z + dvz.y;			// 2*SijSij += (∂vy/∂z + ∂vz/∂y)^2
	tau.a23 = temp;
	SijSij_bytwo += temp*temp;
	float S = sqrtf(SijSij_bytwo);
	float nu_SPS = d_smagfactor*S;		// Dalrymple & Rogers (2006): eq. (12)
	float divu_SPS = 0.6666666666f*nu_SPS*(dvx.x + dvy.y + dvz.z);
	float Blinetal_SPS = d_kspsfactor*SijSij_bytwo;

	// Shear Stress matrix = TAU (pronounced taf)
	// Dalrymple & Rogers (2006): eq. (10)
	tau.a11 = nu_SPS*(dvx.x + dvx.x) - divu_SPS - Blinetal_SPS;	// tau11 = tau_xx/ρ^2
	tau.a11 /= vel.w;
	tau.a12 *= nu_SPS/vel.w;								// tau12 = tau_xy/ρ^2
	tau.a13 *= nu_SPS/vel.w;								// tau13 = tau_xz/ρ^2
	tau.a22 = nu_SPS*(dvy.y + dvy.y) - divu_SPS - Blinetal_SPS;	// tau22 = tau_yy/ρ^2
	tau.a22 /= vel.w;
	tau.a23 *= nu_SPS/vel.w;								// tau23 = tau_yz/ρ^2
	tau.a33 = nu_SPS*(dvz.z + dvz.z) - divu_SPS - Blinetal_SPS;	// tau33 = tau_zz/ρ^2
	tau.a33 /= vel.w;

	tau0[index] = make_float2(tau.a11, tau.a12);
	tau1[index] = make_float2(tau.a13, tau.a22);
	tau2[index] = make_float2(tau.a23, tau.a33);
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   Gamma calculations						    */
/************************************************************************************************************/
template<KernelType kerneltype>
__device__ __forceinline__ float4
gradGamma(	const float slength,
		const float r,
		const float4 boundElement)
{
	float4 retval = W<kerneltype>(r, slength) * boundElement.w * boundElement;
	retval.w = 0;
	return retval;
}


template<KernelType kerneltype, bool periodicbound>
__global__ void
initGradGammaDevice(	float4*		newPos,
			float4*		virtualVel,
			float4*		gradGam,
			const uint*	neibsList,
			const uint	numParticles,
			const float	slength,
			const float	inflRadius)
{
	const uint index = INTMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if(index < numParticles) {
		float4 pos = tex1Dfetch(posTex, index);
		const particleinfo info = tex1Dfetch(infoTex, index);
		
		// Taking info account self contribution in summation
		float4 gGam = make_float4(0.0f);
		float4 virtVel = make_float4(0.0f);
		
		// Compute gradient of gamma for fluid only
		if(FLUID(info)) {
			//uint counter = 0; //DEBUG

			// Loop over all neighbors
			for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave; i += NEIBINDEX_INTERLEAVE) {
				uint neibIndex = neibsList[d_maxneibsnum_time_neibindexinterleave * lane + offset + i];
				
				if(neibIndex == 0xffffffff) break;
				
				float4 neibPos;
				float3 relPos;
				float r;
				
				getNeibData<periodicbound>(pos, inflRadius, neibIndex, neibPos, relPos, r);
				
				const particleinfo neibInfo = tex1Dfetch(infoTex, neibIndex);
				
				if(r < inflRadius && BOUNDARY(neibInfo)) {
					const float4 boundElement = tex1Dfetch(boundTex, neibIndex);
					gGam += gradGamma<kerneltype>(slength, r, boundElement);
					//counter++; //DEBUG
				}
			}
			//DEBUG output
			//if(counter && ((pos.x < 0.1 && pos.y < 0.1) || (pos.x > 1.35 && pos.y > 1.35)) )
			//	printf("X: %g\tY: %g\tZ: %g\tnumBound: %d\n", pos.x, pos.y, pos.z, counter);
			
			//Set the virtual displacement
			float magnitude = length(make_float3(gGam));
			if (magnitude > 1.e-10) {
				virtVel = -1.0 * inflRadius * gGam / magnitude;
				virtVel.w = 0.0;
			}
		}
		
		// Set gamma to 1
		gGam.w = 1.0;
		
		gradGam[index] = gGam;
		virtualVel[index].x = virtVel.x;
		virtualVel[index].y = virtVel.y;
		virtualVel[index].z = virtVel.z;
		newPos[index] = pos - virtVel;
	}
}


template<KernelType kerneltype, bool periodicbound>
__global__ void
updateGammaDevice(	float4*		newGam,
			const uint*	neibsList,
			const uint	numParticles,
			const float	slength,
			const float	inflRadius,
			const float	virtDt)
{
	const uint index = INTMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);

	if(index < numParticles) {
		float4 pos = tex1Dfetch(posTex, index);
		const particleinfo info = tex1Dfetch(infoTex, index);
		float3 vel = make_float3(tex1Dfetch(velTex, index));
		float4 oldGam = tex1Dfetch(gamTex, index);

		float4 gGam = make_float4(0.0f);
		float deltaGam = 0.0;

		// Compute gradient of gamma for fluid only
		if(FLUID(info)) {
			// Loop over all neighbors
			for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave; i += NEIBINDEX_INTERLEAVE) {
				uint neibIndex = neibsList[d_maxneibsnum_time_neibindexinterleave * lane + offset + i];

				if(neibIndex == 0xffffffff) break;

				float4 neibPos;
				float3 relPos;
				float r;
				
				getNeibData<periodicbound>(pos, inflRadius, neibIndex, neibPos, relPos, r);

				const particleinfo neibInfo = tex1Dfetch(infoTex, neibIndex);

				if(r < inflRadius && BOUNDARY(neibInfo)) {
					const float4 boundElement = tex1Dfetch(boundTex, neibIndex);
					const float4 gradGamma_as = gradGamma<kerneltype>(slength, r, boundElement);
					gGam += gradGamma_as;
					deltaGam += dot(make_float3(gradGamma_as), vel);
				}
			}

			//Update gamma value
			float magnitude = length(make_float3(gGam));
			if (magnitude > 1.e-10) {
				gGam.w = oldGam.w + deltaGam * 0.5*virtDt;
			}
			else
				gGam.w = 1.0;
		}

		newGam[index] = gGam;
	}
}


template<KernelType kerneltype, bool periodicbound>
__global__ void
updateGammaPrCorDevice( float4*		newPos,
			float4*		newGam,
			const uint*	neibsList,
			const uint	numParticles,
			const float	slength,
			const float	inflRadius,
			const float	virtDt)
{
	const uint index = INTMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);

	if(index < numParticles) {
		float4 oldpos = tex1Dfetch(posTex, index);
		float4 newpos = newPos[index];
		const particleinfo info = tex1Dfetch(infoTex, index);
		float3 vel = make_float3(tex1Dfetch(velTex, index));
		float4 oldGam = tex1Dfetch(gamTex, index);

		float4 gGam = make_float4(0.0f);
		float deltaGam = 0.0;
		deltaGam += dot(make_float3(oldGam), vel); //FIXME: It is incorrect for moving boundaries

		// Compute gradient of gamma for fluid only
		if(FLUID(info)) {

			// Loop over all neighbors
			for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave; i += NEIBINDEX_INTERLEAVE) {
				uint neibIndex = neibsList[d_maxneibsnum_time_neibindexinterleave * lane + offset + i];

				if(neibIndex == 0xffffffff) break;

				float4 neibPos;
				float3 relPos;
				float r;

				getNeibData<periodicbound>(newpos, newPos, inflRadius, neibIndex, neibPos, relPos, r);

				const particleinfo neibInfo = tex1Dfetch(infoTex, neibIndex);

				if(r < inflRadius && BOUNDARY(neibInfo)) {
					const float4 boundElement = tex1Dfetch(boundTex, neibIndex);
					const float4 gradGamma_as = gradGamma<kerneltype>(slength, r, boundElement);
					gGam += gradGamma_as;
					deltaGam += dot(make_float3(gradGamma_as), vel);
				}
			}

			//Update gamma value
			float magnitude = length(make_float3(gGam));
			if (magnitude > 1.e-10) {
				gGam.w = oldGam.w + deltaGam * 0.25*virtDt;
			}
			else
				gGam.w = 1.0;
		}

		newGam[index] = gGam;
	}
}


//FIXME: Modify this kernel taking into account periodic boundary
//template<KernelType kerneltype, bool periodicbound>
__global__ void
updatePositionsDevice(	float4*	newPos,
			float	virtDt,
			uint	numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if(index < numParticles) {
		float4 pos = tex1Dfetch(posTex, index);
		const particleinfo info = tex1Dfetch(infoTex, index);
		float4 vel = tex1Dfetch(velTex, index);

		if(FLUID(info)) {
			pos.x += virtDt * vel.x;
			pos.y += virtDt * vel.y;
			pos.z += virtDt * vel.z;
		}

//		if (periodicbound) {
//			if (d_dispvect.x) {
//				if (pos.x >= d_maxlimit.x)
//					pos.x -= d_dispvect.x;
//				else if (pos.x < d_minlimit.x)
//					pos.x += d_dispvect.x;
//			}
//			if (d_dispvect.y) {
//				if (pos.y >= d_maxlimit.y)
//					pos.y -= d_dispvect.y;
//				else if (pos.y < d_minlimit.y)
//					pos.y += d_dispvect.y;
//			}
//			if (d_dispvect.z) {
//				if (pos.z >= d_maxlimit.z)
//					pos.z -= d_dispvect.z;
//				else if (pos.z < d_minlimit.z)
//					pos.z += d_dispvect.z;
//			}
//		}

		newPos[index] = pos;
	}
}

__global__ void
updateBoundValuesDevice(	float4*		oldVel,
				float*		oldPressure,
				const uint	numParticles,
				bool		initStep)
{
	const uint index = INTMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	
	if(index < numParticles) {
		const particleinfo info = tex1Dfetch(infoTex, index);
		const vertexinfo vertices = tex1Dfetch(vertTex, index);
		const float ro1 = oldVel[vertices.x].w;
		const float ro2 = oldVel[vertices.y].w;
		const float ro3 = oldVel[vertices.z].w;
		const float pres1 = oldPressure[vertices.x];
		const float pres2 = oldPressure[vertices.y];
		const float pres3 = oldPressure[vertices.z];

		if (BOUNDARY(info)) {
			oldVel[index].w = (ro1 + ro2 + ro3)/3.f;
			oldPressure[index] = (pres1 + pres2 + pres3)/3.f;
		}
		//FIXME: it should be implemented somewhere in initializeGammaAndGradGamma
		//FIXME: keeping initial velocity values, if given
		if (initStep && FLUID(info)) {
			oldVel[index].x = 0;
			oldVel[index].y = 0;
			oldVel[index].z = 0;
		}
	}
}

template<KernelType kerneltype, bool periodicbound >
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
dynamicBoundConditionsDevice(	const float4*	oldPos,
				float4*		oldVel,
				float*		oldPressure,
				const uint*	neibsList,
				const uint	numParticles,
				const float	slength,
				const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (!VERTEX(info))
		return;

	#if( __COMPUTE__ >= 20)
	const float4 pos = oldPos[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	const float vel = length(make_float3(oldVel[index]));

	// in contrast to Shepard filter particle itself doesn't contribute into summation
	float temp1 = 0;
	float temp2 = 0;
	float alpha = 0;

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		#if( __COMPUTE__ >= 20)
		getNeibData<periodicbound>(pos, oldPos, influenceradius, neib_index, neib_pos, relPos, r);
		#else
		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		#endif

//		const float neib_rho = tex1Dfetch(velTex, neib_index).w;
		const float neib_rho = oldVel[neib_index].w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);
		const float neib_pres = P(neib_rho, PART_FLUID_NUM(neib_info));
		const float neib_vel = length(make_float3(oldVel[neib_index]));

		if (r < influenceradius && FLUID(neib_info)) {
			const float w = W<kerneltype>(r, slength)*neib_pos.w;
			temp1 += w;
			temp2 += w/neib_rho*(neib_pres/neib_rho + dot(d_gravity,relPos) + 0.5*(neib_vel*neib_vel-vel*vel));
			alpha += w/neib_rho;
		}
	}

	if(alpha)
	{
		oldVel[index].w = temp1/alpha; //FIXME: this can be included directly in the next line
		oldPressure[index] = temp2*oldVel[index].w/alpha;
		oldVel[index].w = rho(oldPressure[index], PART_FLUID_NUM(info));
	}
}

template<KernelType kerneltype, bool periodicbound >
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
calcProbeDevice(	float4*		oldPos,
			float4*		oldVel,
			float*		oldPressure,
			const uint*	neibsList,
			const uint	numParticles,
			const float	slength,
			const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	const particleinfo info = tex1Dfetch(infoTex, index);
	// kernel is only run for probe particles
	if (!PROBE(info))
		return;

	#if( __COMPUTE__ >= 20)
	const float4 pos = oldPos[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// in contrast to Shepard filter particle itself doesn't contribute into summation
	float pressure = 0;
	float alpha = 0;
	uint num_neib = 0;

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		#if( __COMPUTE__ >= 20)
		getNeibData<periodicbound>(pos, oldPos, influenceradius, neib_index, neib_pos, relPos, r);
		#else
		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		#endif

		const float neib_rho = oldVel[neib_index].w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && FLUID(neib_info)) {
			const float w = W<kerneltype>(r, slength)*neib_pos.w/neib_rho;
			pressure += w*P(neib_rho, PART_FLUID_NUM(neib_info));
			alpha += w;
			num_neib++;
		}
	}

	if(alpha)
	{
		oldPos[index].w = alpha;
	}
	if(num_neib > 10)
		oldPressure[index] = pressure/alpha;
	else
		oldPressure[index] = 0;
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   Kernels for computing acceleration without gradient correction					 */
/************************************************************************************************************/

/* Normal kernels */
#include "forces_kernel.xsphdt.inc"

/************************************************************************************************************/


/************************************************************************************************************/
/*					   Kernels for XSPH, Shepard and MLS corrections									   */
/************************************************************************************************************/

// This kernel computes the Shepard correction
template<KernelType kerneltype, bool periodicbound >
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
shepardDevice(	const float4*	posArray,
				float4*			newVel,
				const uint*		neibsList,
				const uint		numParticles,
				const float		slength,
				const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// normalize kernel only if the given particle is a fluid one
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info) && !VERTEX(info))
		return;

	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 vel = tex1Dfetch(velTex, index);

	// taking into account self contribution in summation
	float temp1 = pos.w*W<kerneltype>(0, slength);
	float temp2 = temp1/vel.w ;

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		#if( __COMPUTE__ >= 20)							
		getNeibData<periodicbound>(pos, posArray, influenceradius, neib_index, neib_pos, relPos, r);
		#else
		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		#endif

		const float neib_rho = tex1Dfetch(velTex, neib_index).w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && (FLUID(neib_info)/* || VERTEX(neib_info)*/)) {
			const float w = W<kerneltype>(r, slength)*neib_pos.w;
			temp1 += w;
			temp2 += w/neib_rho;
		}
	}

	vel.w = temp1/temp2;
	newVel[index] = vel;
}


// This kernel computes the MLS correction
template<KernelType kerneltype, bool periodicbound>
__global__ void
__launch_bounds__(BLOCK_SIZE_MLS, MIN_BLOCKS_MLS)
MlsDevice(	const float4*	posArray,
			float4*			newVel,
			const uint*		neibsList,
			const uint		numParticles,
			const float		slength,
			const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// computing MLS matrix only for fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 vel = tex1Dfetch(velTex, index);

	// MLS matrix elements
	float a11 = 0.0f, a12 = 0.0f, a13 = 0.0f, a14 = 0.0f;
	float a22 = 0.0f, a23 = 0.0f, a24 = 0.0f;
	float a33 = 0.0f, a34 = 0.0f;
	float a44 = 0.0f;

	// number of neighbors
	int neibs_num = 0;

	// taking into account self contribution in MLS matrix construction
	a11 = W<kerneltype>(0, slength)*pos.w/vel.w;

	// first loop over all the neighbors for the MLS matrix
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		#if( __COMPUTE__ >= 20)							
		getNeibData<periodicbound>(pos, posArray, influenceradius, neib_index, neib_pos, relPos, r);
		#else
		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		#endif

		const float neib_rho = tex1Dfetch(velTex, neib_index).w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// interaction between two particles
		if (r < influenceradius && FLUID(neib_info)) {
			neibs_num ++;
			const float w = W<kerneltype>(r, slength)*neib_pos.w/neib_rho;	// Wij*Vj
			a11 += w;						// a11 = ∑Wij*Vj
			a12 += relPos.x*w;				// a12 = ∑(xi - xj)*Wij*Vj
			a13 += relPos.y*w;				// a13 = ∑(yi - yj)*Wij*Vj
			a14 += relPos.z*w;				// a14 = ∑(zi - zj)*Wij*Vj
			a22 += relPos.x*relPos.x*w;		// a22 = ∑(xi - xj)^2*Wij*Vj
			a23 += relPos.x*relPos.y*w;		// a23 = ∑(xi - xj)(yi - yj)*Wij*Vj
			a24 += relPos.x*relPos.z*w;		// a23 = ∑(xi - xj)(zi - zj)*Wij*Vj
			a33 += relPos.y*relPos.y*w;		// a33 = ∑(yi - yj)^2*Wij*Vj
			a34 += relPos.y*relPos.z*w;		// a33 = ∑(yi - yj)(zi - zj)*Wij*Vj
			a44 += relPos.z*relPos.z*w;		// a33 = ∑(yi - yj)^2*Wij*Vj
		}
	} // end of first loop trough neighbors

	// safe inverse of MLS matrix
	// the matrix is inverted only if |det|/max|aij|^4 > EPSDET
	// and if the number of fluids neighbors if above a minimum
	// value, otherwise no correction is applied
	float maxa = fmaxf(fabsf(a11), fabsf(a12));
	maxa = fmaxf(maxa, fabsf(a13));
	maxa = fmaxf(maxa, fabsf(a14));
	maxa = fmaxf(maxa, fabsf(a22));
	maxa = fmaxf(maxa, fabsf(a23));
	maxa = fmaxf(maxa, fabsf(a24));
	maxa = fmaxf(maxa, fabsf(a33));
	maxa = fmaxf(maxa, fabsf(a34));
	maxa = fmaxf(maxa, fabsf(a44));
	maxa *= maxa;
	maxa *= maxa;
	float det = a11*(a22*a33*a44 + a23*a34*a24 + a24*a23*a34 - a22*a34*a34 - a23*a23*a44 - a24*a33*a24)
			  + a12*(a12*a34*a34 + a23*a13*a44 + a24*a33*a14 - a12*a33*a44 - a23*a34*a14 - a24*a13*a34)
			  + a13*(a12*a23*a44 + a22*a34*a14 + a24*a13*a24 - a12*a34*a24 - a22*a13*a44 - a24*a23*a14)
			  + a14*(a12*a33*a24 + a22*a13*a34 + a23*a23*a14 - a12*a23*a34 - a22*a33*a14 - a23*a13*a24);
	if (det > maxa*EPSDETMLS && neibs_num > MINCORRNEIBSMLS) {  // FIXME: should be |det| ?????
		// first row of inverse matrix
		det = 1/det;
		const float b11 = (a22*a33*a44 + a23*a34*a24 + a24*a23*a34 - a22*a34*a34 - a23*a23*a44 - a24*a33*a24)*det;
		const float b21 = (a12*a34*a34 + a23*a13*a44 + a24*a33*a14 - a12*a33*a44 - a23*a34*a14 - a24*a13*a34)*det;
		const float b31 = (a12*a23*a44 + a22*a34*a14 + a24*a13*a24 - a12*a34*a24 - a22*a13*a44 - a24*a23*a14)*det;
		const float b41 = (a12*a33*a24 + a22*a13*a34 + a23*a23*a14 - a12*a23*a34 - a22*a33*a14 - a23*a13*a24)*det;

		// taking into account self contribution in density summation
		vel.w = b11*W<kerneltype>(0, slength)*pos.w;

		// second loop over all the neighbors for correction
		for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
			uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

			if (neib_index == 0xffffffff) break;

			float4 neib_pos;
			float3 relPos;
			float r;

			#if( __COMPUTE__ >= 20)							
			getNeibData<periodicbound>(pos, posArray, influenceradius, neib_index, neib_pos, relPos, r);
			#else
			getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
			#endif
			const float neib_rho = tex1Dfetch(velTex, neib_index).w;
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			// interaction between two particles
			if (r < influenceradius && FLUID(neib_info)) {
				const float w = W<kerneltype>(r, slength)*neib_pos.w;	 // ρj*Wij*Vj = mj*Wij
				vel.w += (b11 + b21*relPos.x + b31*relPos.y
							+ b41*relPos.z)*w;	 // ρ = ∑(ß0 + ß1(xi - xj) + ß2(yi - yj))*Wij*Vj
			}
		}  // end of second loop trough neighbors
	} else {
		// Resort to Shepard filter in absence of invertible matrix
		// see also shepardDevice. TODO: share the code
		// we use a11 and a12 for temp1, temp2
		a11 = pos.w*W<kerneltype>(0, slength);
		a12 = a11/vel.w;

			// loop over all neighbors
		for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
			uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

				if (neib_index == 0xffffffff) break;

				float4 neib_pos;
				float3 relPos;
				float r;

				#if( __COMPUTE__ >= 20)							
				getNeibData<periodicbound>(pos, posArray, influenceradius, neib_index, neib_pos, relPos, r);
				#else
				getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
				#endif
				const float neib_rho = tex1Dfetch(velTex, neib_index).w;
				const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

				// interaction between two particles
				if (r < influenceradius && FLUID(neib_info)) {
						// ρj*Wij*Vj = mj*Wij
						const float w = W<kerneltype>(r, slength)*neib_pos.w;
						// ρ = ∑(ß0 + ß1(xi - xj) + ß2(yi - yj))*Wij*Vj
						a11 += w;
						a12 +=w/neib_rho;
				}
		}  // end of second loop through neighbors

		vel.w = a11/a12;
	}

	newVel[index] = vel;
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   CFL max kernel																		*/
/************************************************************************************************************/
template <unsigned int blockSize>
__global__ void
fmaxDevice(float *g_idata, float *g_odata, const uint n)
{
	extern __shared__ float sdata[];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	float myMax = 0;

	// we reduce multiple elements per thread.  The number is determined by the 
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{         
		myMax = max(myMax, g_idata[i]);
		// ensure we don't read out of bounds
		if (i + blockSize < n) 
			myMax = max(myMax, g_idata[i + blockSize]);
		i += gridSize;
	} 

	// each thread puts its local sum into shared memory 
	sdata[tid] = myMax;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = myMax = max(myMax,sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = myMax = max(myMax,sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = myMax = max(myMax,sdata[tid +  64]); } __syncthreads(); }

	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and induce incorrect behavior.
	if (tid < 32)
	{
		volatile float* smem = sdata;
		if (blockSize >=  64) { smem[tid] = myMax = max(myMax, smem[tid + 32]); }
		if (blockSize >=  32) { smem[tid] = myMax = max(myMax, smem[tid + 16]); }
		if (blockSize >=  16) { smem[tid] = myMax = max(myMax, smem[tid +  8]); }
		if (blockSize >=   8) { smem[tid] = myMax = max(myMax, smem[tid +  4]); }
		if (blockSize >=   4) { smem[tid] = myMax = max(myMax, smem[tid +  2]); }
		if (blockSize >=   2) { smem[tid] = myMax = max(myMax, smem[tid +  1]); }
	}

	// write result for this block to global mem 
	if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   Parallel reduction kernels															*/
/************************************************************************************************************/

extern __shared__ float4 shmem4[];

extern "C" __global__
void calcEnergies(
		const float4* pPos,
		const float4* pVel,
		const particleinfo* pInfo,
		uint	numParticles,
		uint	numFluids,
		float4* output
		)
{
	// shared memory for this kernel should be sized to
	// blockDim.x*numFluids*sizeof(float4)*2

	uint gid = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	uint stride = INTMUL(gridDim.x,blockDim.x);
	// .x kinetic, .y potential, .z internal
	float4 energy[MAX_FLUID_TYPES], E_k[MAX_FLUID_TYPES];

#pragma unroll
	for (uint i = 0; i < MAX_FLUID_TYPES; ++i)
		energy[i] = E_k[i] = make_float4(0.0f);

	while (gid < numParticles) {
		float4 pos = pPos[gid];
		float4 vel = pVel[gid];
		particleinfo pinfo = pInfo[gid];
		if (FLUID(pinfo)) {
			uint fluid_num = PART_FLUID_NUM(pinfo);
			float v2 = kahan_sqlength(as_float3(vel));
			float gh = kahan_dot(d_gravity, as_float3(pos));
			kahan_add(energy[fluid_num].x, pos.w*v2/2, E_k[fluid_num].x);
			kahan_add(energy[fluid_num].y, -pos.w*gh, E_k[fluid_num].y);
			// internal elastic energy
			float gamma = d_gammacoeff[fluid_num];
			float gm1 = d_gammacoeff[fluid_num]-1;
			float rho0 = d_rho0[fluid_num];
			float elen = __powf(vel.w/rho0, gm1)/gm1 + rho0/vel.w - gamma/gm1;
			float ssp = soundSpeed(vel.w, fluid_num);
			elen *= ssp*ssp/gamma;
			kahan_add(energy[fluid_num].z, pos.w*elen, E_k[fluid_num].z);
		}
		gid += stride;
	}

	uint lid = threadIdx.x;
	for (uint offset = blockDim.x/2; offset; offset >>= 1) {
		stride = offset*numFluids; // stride between fields in shmem4 memory
		if (lid >= offset && lid < 2*offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = lid + offset*i;
				shmem4[idx] = energy[i];
				idx += stride;
				shmem4[idx] = E_k[i];
			}
		}
		__syncthreads();
		if (lid < offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = lid + offset*(i+1);
				float4 other = shmem4[idx];
				idx += stride;
				float4 oth_k = shmem4[idx];
				kahan_add(energy[i].x, oth_k.x, E_k[i].x);
				kahan_add(energy[i].x, other.x, E_k[i].x);
				kahan_add(energy[i].y, oth_k.y, E_k[i].y);
				kahan_add(energy[i].y, other.y, E_k[i].y);
				kahan_add(energy[i].z, oth_k.z, E_k[i].z);
				kahan_add(energy[i].z, other.z, E_k[i].z);
			}
		}
	}

	if (lid == 0) {
		for (uint i = 0; i < numFluids; ++i) {
			output[blockIdx.x + INTMUL(gridDim.x,i)] = energy[i];
			output[blockIdx.x + INTMUL(gridDim.x,numFluids+i)] = E_k[i];
		}
	}
}

// final reduction stage
extern "C" __global__
void calcEnergies2(
		float4* buffer,
		uint	prev_blocks,
		uint	numFluids)
{
	// shared memory for this kernel should be sized to
	// blockDim.x*numFluids*sizeof(float4)*2

	uint gid = threadIdx.x;
	float4 energy[MAX_FLUID_TYPES];
	float4 E_k[MAX_FLUID_TYPES];
	for (uint i = 0; i < numFluids; ++i) {
		if (gid < prev_blocks) {
			energy[i] = buffer[gid + prev_blocks*i];
			E_k[i] = buffer[gid + prev_blocks*(numFluids+i)];
		} else {
			energy[i] = E_k[i] = make_float4(0.0f);
		}
	}

	uint stride;
	for (uint offset = blockDim.x/2; offset; offset >>= 1) {
		stride = offset*numFluids; // stride between fields in shmem4 memory
		if (gid >= offset && gid < 2*offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = gid + offset*i;
				shmem4[idx] = energy[i];
				idx += stride;
				shmem4[idx] = E_k[i];
			}
		}
		__syncthreads();
		if (gid < offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = gid + offset*(i+1);
				float4 other = shmem4[idx];
				idx += stride;
				float4 oth_k = shmem4[idx];
				kahan_add(energy[i].x, oth_k.x, E_k[i].x);
				kahan_add(energy[i].x, other.x, E_k[i].x);
				kahan_add(energy[i].y, oth_k.y, E_k[i].y);
				kahan_add(energy[i].y, other.y, E_k[i].y);
				kahan_add(energy[i].z, oth_k.z, E_k[i].z);
				kahan_add(energy[i].z, other.z, E_k[i].z);
			}
		}
	}

	if (gid == 0) {
		for (uint i = 0; i < numFluids; ++i)
			buffer[i] = energy[i] + E_k[i];
	}
}


/************************************************************************************************************/
/*					   Auxiliary kernels used for post processing										    */
/************************************************************************************************************/

// This kernel compute the vorticity field
template<KernelType kerneltype, bool periodicbound>
__global__ void
calcVortDevice(	float3*		vorticity,
				const uint*	neibsList,
				const uint	numParticles,
				const float	slength,
				const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// computing vorticity only for fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	float4 pos = tex1Dfetch(posTex, index);
	float4 vel = tex1Dfetch(velTex, index);

	// MLS matrix elements
	float3 vort = make_float3(0.0f);

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		const float4 neib_vel = tex1Dfetch(velTex, neib_index);
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// interaction between two particles
		if (r < influenceradius && FLUID(neib_info)) {
			float3 relVel;
			relVel.x = vel.x - neib_vel.x;
			relVel.y = vel.y - neib_vel.y;
			relVel.z = vel.z - neib_vel.z;
			const float f = F<kerneltype>(r, slength)*neib_pos.w/neib_vel.w;	// ∂Wij/∂r*Vj
			// vxij = vxi - vxj and same for vyij and vzij
			vort.x += f*(relVel.y*relPos.z - relVel.z*relPos.y);		// vort.x = ∑(vyij(zi - zj) - vzij*(yi - yj))*∂Wij/∂r*Vj
			vort.y += f*(relVel.z*relPos.x - relVel.x*relPos.z);		// vort.y = ∑(vzij(xi - xj) - vxij*(zi - zj))*∂Wij/∂r*Vj
			vort.z += f*(relVel.x*relPos.y - relVel.y*relPos.x);		// vort.x = ∑(vxij(yi - yj) - vyij*(xi - xj))*∂Wij/∂r*Vj
		}
	} // end of loop trough neighbors

	vorticity[index] = vort;
}


// Testpoints
// This kernel compute the velocity at testpoints
template<KernelType kerneltype, bool periodicbound >
__global__ void
calcTestpointsVelocityDevice(	float4*		newVel,
								const uint*	neibsList,
								const uint	numParticles,
								const float	slength,
								const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	const particleinfo info = tex1Dfetch(infoTex, index);
	if(type(info) != TESTPOINTSPART)
		return;
	
	float4 pos = tex1Dfetch(posTex, index);
	float4 vel = tex1Dfetch(velTex, index);
	
	float4 temp = make_float4(0.0f);

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		const float4 neib_vel = tex1Dfetch(velTex, neib_index);
        const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && FLUID(neib_info)) {
			const float w = W<kerneltype>(r, slength)*neib_pos.w/neib_vel.w;	// Wij*mj
			temp.x += w*neib_vel.x;
			temp.y += w*neib_vel.y;
			temp.z += w*neib_vel.z;
			//Pressure
			temp.w += w*P(neib_vel.w, object(neib_info));

		}
	}

	vel = temp;

	newVel[index] = vel;
}


// Free surface detection
// This kernel detects the surface particles
template<KernelType kerneltype, bool periodicbound, bool savenormals>
__global__ void
calcSurfaceparticleDevice(	float4*			normals,
							particleinfo*	newInfo,
							const uint*		neibsList,
							const uint		numParticles,
							const float		slength,
							const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = threadIdx.x & (NEIBINDEX_INTERLEAVE - 1);
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	particleinfo info = tex1Dfetch(infoTex, index);

	if (NOT_FLUID(info)) {
		newInfo[index] = info;		
		return;
	}

	float4 pos = tex1Dfetch(posTex, index);
	float4 normal = make_float4(0.0f);
	
	info.x &= ~SURFACE_PARTICLE_FLAG;
	normal.w = W<kerneltype>(0.0f, slength)*pos.w;

	// loop over all the neighbors (First loop)
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];

		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		const float neib_density = tex1Dfetch(velTex, neib_index).w;

		if (r < influenceradius) {
			const float f = F<kerneltype>(r, slength)* neib_pos.w /neib_density; // 1/r ∂Wij/∂r Vj
			normal.x -= f * relPos.x;
			normal.y -= f * relPos.y;
			normal.z -= f * relPos.z;
			normal.w += W<kerneltype>(r, slength)*neib_pos.w;	// Wij*mj ;

		}
	}

	float normal_length = length(as_float3(normal));

	//Checking the planes
	for (uint i = 0; i < d_numplanes; ++i) {
		float r = abs(dot(as_float3(pos), as_float3(d_planes[i])) + d_planes[i].w)/d_plane_div[i];
		if (r < influenceradius) {
			as_float3(normal) += as_float3(d_planes[i])* normal_length;
			normal_length = length(as_float3(normal));
		}
	}

	// loop over all the neighbors (Second loop)
	int nc = 0;
	for(uint i = 0; i < d_maxneibsnum_time_neibindexinterleave ; i += NEIBINDEX_INTERLEAVE) {
		uint neib_index = neibsList[d_maxneibsnum_time_neibindexinterleave*lane + i + offset];
		
		if (neib_index == 0xffffffff) break;

		float4 neib_pos;
		float3 relPos;
		float r;

		float cosconeangle;

		getNeibData<periodicbound>(pos, influenceradius, neib_index, neib_pos, relPos, r);
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius) {
			float criteria = -(normal.x * relPos.x + normal.y * relPos.y + normal.z * relPos.z);
			if (FLUID(neib_info))
				cosconeangle = d_cosconeanglefluid;
			else
				cosconeangle = d_cosconeanglenonfluid;

			if (criteria > r*normal_length*cosconeangle)
				nc++;
		}

	}

	if (!nc)
		info.x |= SURFACE_PARTICLE_FLAG;

	newInfo[index] = info;

	if (savenormals) {
		normal.x /= normal_length;
		normal.y /= normal_length;
		normal.z /= normal_length;
		normals[index] = normal;
		}

}
/************************************************************************************************************/

} //namespace cuforces
#endif
