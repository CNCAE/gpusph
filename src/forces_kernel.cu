#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */

#ifndef _FORCES_KERNEL_
#define _FORCES_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "vector_math.h"

#define GPU_CODE
#include "kahan.h"
#undef GPU_CODE

texture<float, 2, hipReadModeElementType> demTex;	// DEM

namespace cuforces {
__constant__ uint d_maxneibsnum_time_numparticles;

__constant__ float	d_wcoeff_cubicspline;			// coeff = 1/(Pi h^3)
__constant__ float	d_wcoeff_quadratic;				// coeff = 15/(16 Pi h^3)
__constant__ float	d_wcoeff_wendland;				// coeff = 21/(16 Pi h^3)

__constant__ float	d_fcoeff_cubicspline;			// coeff = 3/(4Pi h^4)
__constant__ float	d_fcoeff_quadratic;				// coeff = 15/(32Pi h^4)
__constant__ float	d_fcoeff_wendland;				// coeff = 105/(128Pi h^5)

__constant__ int    d_numfluids;					// number of different fluids

__constant__ float	d_rho0[MAX_FLUID_TYPES];		// rest density of fluids

// Speed of sound constants
__constant__ float	d_bcoeff[MAX_FLUID_TYPES];
__constant__ float	d_gammacoeff[MAX_FLUID_TYPES];
__constant__ float	d_sscoeff[MAX_FLUID_TYPES];
__constant__ float	d_sspowercoeff[MAX_FLUID_TYPES];

__constant__ float3	d_gravity;						// gravity (vector)

// LJ boundary repusion force comuting
__constant__ float	d_dcoeff;
__constant__ float	d_p1coeff;
__constant__ float	d_p2coeff;
__constant__ float	d_r0;

// Monaghan-Kaijar boundary repulsion force constants
// This is typically the square of the maximum velocity, or gravity times the maximum height
__constant__ float	d_MK_K;
// This is typically the distance between boundary particles
__constant__ float	d_MK_d;
// This is typically the ration between h and the distance between boundary particles
__constant__ float	d_MK_beta;

__constant__ float	d_visccoeff;
__constant__ float	d_epsartvisc;


// Constants used for DEM
__constant__ float	d_ewres;
__constant__ float	d_nsres;
__constant__ float	d_demdx;
__constant__ float	d_demdy;
__constant__ float	d_demdxdy;
__constant__ float	d_demzmin;

__constant__ float	d_partsurf;						// particle surface

// Definition of planes for geometrical boundaries
__constant__ uint	d_numplanes;
__constant__ float4	d_planes[MAXPLANES];
__constant__ float	d_plane_div[MAXPLANES];

// Sub-Particle Scale (SPS) Turbulence parameters
__constant__ float	d_smagfactor;
__constant__ float	d_kspsfactor;

// Free surface detection
__constant__ float	d_cosconeanglefluid;
__constant__ float	d_cosconeanglenonfluid;

// Rigid body data (test version)
__device__ float3	d_force;
__device__ float3	d_torque;
__constant__ float3 d_rbcg[MAXBODIES];
__constant__ uint	d_rbstartindex[MAXBODIES];
__constant__ float d_objectobjectdf;
__constant__ float d_objectboundarydf;

// Grid data
__constant__ float3 d_worldOrigin;
__constant__ uint3	d_gridSize;
__constant__ float3 d_cellSize;

// Neibdata cell number to offset
__constant__ char3 d_cell_to_offset[27];

typedef struct sym33mat {
	float a11;
	float a12;
	float a13;
	float a22;
	float a23;
	float a33;
} sym33mat;


/************************************************************************************************************/
/*							  Functions used by the differents CUDA kernels							   */
/************************************************************************************************************/

/********************************************* SPH kernels **************************************************/
// Return kernel value at distance r, for a given smoothing length
template<KernelType kerneltype>
__device__ __forceinline__ float
W(const float r, const float slength);


// Cubic Spline kernel
template<>
__device__ __forceinline__ float
W<CUBICSPLINE>(const float r, const float slength)
{
	float val = 0.0f;
	const float R = r/slength;

	if (R < 1)
		val = 1.0f - 1.5f*R*R + 0.75f*R*R*R;			// val = 1 - 3/2 R^2 + 3/4 R^3
	else
		val = 0.25f*(2.0f - R)*(2.0f - R)*(2.0f - R);	// val = 1/4 (2 - R)^3

	val *= d_wcoeff_cubicspline;						// coeff = 1/(Pi h^3)

	return val;
}


// Qudratic kernel
template<>
__device__ __forceinline__ float
W<QUADRATIC>(const float r, const float slength)
{
	float val = 0.0f;
	const float R = r/slength;

	val = 0.25f*R*R - R + 1.0f;		// val = 1/4 R^2 -  R + 1
	val *= d_wcoeff_quadratic;		// coeff = 15/(16 Pi h^3)

	return val;
}


// Wendland kernel
template<>
__device__ __forceinline__ float
W<WENDLAND>(float r, float slength)
{
	const float R = r/slength;

	float val = 1.0f - 0.5f*R;
	val *= val;
	val *= val;						// val = (1 - R/2)^4
	val *= 1.0f + 2.0f*R;			// val = (2R + 1)(1 - R/2)^4*
	val *= d_wcoeff_wendland;		// coeff = 21/(16 Pi h^3)
	return val;
}


// Return 1/r dW/dr at distance r, for a given smoothing length
template<KernelType kerneltype>
__device__ __forceinline__ float
F(const float r, const float slength);


template<>
__device__ __forceinline__ float
F<CUBICSPLINE>(const float r, const float slength)
{
	float val = 0.0f;
	const float R = r/slength;

	if (R < 1.0f)
		val = (-4.0f + 3.0f*R)/slength;		// val = (-4 + 3R)/h
	else
		val = -(-2.0f + R)*(-2.0f + R)/r;	// val = -(-2 + R)^2/r
	val *= d_fcoeff_cubicspline;			// coeff = 3/(4Pi h^4)

	return val;
}


template<>
__device__ __forceinline__ float
F<QUADRATIC>(const float r, const float slength)
{
	const float R = r/slength;

	float val = (-2.0f + R)/r;		// val = (-2 + R)/r
	val *= d_fcoeff_quadratic;		// coeff = 15/(32Pi h^4)

	return val;
}


template<> 
__device__ __forceinline__ float
F<WENDLAND>(const float r, const float slength)
{
	const float qm2 = r/slength - 2.0f;	// val = (-2 + R)^3
	float val = qm2*qm2*qm2*d_fcoeff_wendland;
	return val;
}
/************************************************************************************************************/


/********************** Equation of state, speed of sound, repulsive force **********************************/
// Equation of state: pressure from density, where i is the fluid kind, not particle_id
__device__ __forceinline__ float
P(const float rho, const uint i)
{
	return d_bcoeff[i]*(__powf(rho/d_rho0[i], d_gammacoeff[i]) - 1);
}


// Sound speed computed from density
__device__ __forceinline__ float
soundSpeed(const float rho, const uint i)
{
	return d_sscoeff[i]*__powf(rho/d_rho0[i], d_sspowercoeff[i]);
}


// Lennard-Jones boundary repulsion force
__device__ __forceinline__ float
LJForce(const float r)
{
	float force = 0.0f;

	if (r <= d_r0)
		force = d_dcoeff*(__powf(d_r0/r, d_p1coeff) - __powf(d_r0/r, d_p2coeff))/(r*r);

	return force;
}

// Monaghan-Kajtar boundary repulsion force doi:10.1016/j.cpc.2009.05.008
// to be multiplied by r_aj vector
// we allow the fluid particle mass mass_f to be different from the
// boundary particle mass mass_b even though they are typically the same
// (except for multi-phase fluids)
__device__ __forceinline__ float
MKForce(const float r, const float slength,
		const float mass_f, const float mass_b)
{
	// MK always uses the 1D cubic or quintic Wendland spline
	float w = 0.0f;

	float force = 0.0f;

	// Wendland has radius 2
	if (r <= 2*slength) {	//TODO: fixme use influenceradius
		float qq = r/slength;
		w = 1.8f * __powf(1.0f - 0.5f*qq, 4.0f) * (2.0f*qq + 1.0f);  //TODO: optimize
		// float dist = r - d_MK_d;
		float dist = max(d_epsartvisc, r - d_MK_d);
		force = d_MK_K*w*2*mass_b/(d_MK_beta * dist * r * (mass_f+mass_b));
	}

	return force;
}
/************************************************************************************************************/


/***************************************** Viscosities *******************************************************/
// Artificial viscosity s
__device__ __forceinline__ float
artvisc(	const float	vel_dot_pos,
			const float	rho,
			const float	neib_rho,
			const float	sspeed,
			const float	neib_sspeed,
			const float	r,
			const float	slength)
{
	return vel_dot_pos*slength*d_visccoeff*(sspeed + neib_sspeed)/
									((r*r + d_epsartvisc)*(rho + neib_rho));
}


// ATTENTION: for all non artificial viscosity
// µ is the dynamic viscosity (ρν)

// Scalar part of viscosity using Morris 1997
// expression 21 p218 when all particles have the same viscosity
// in this case d_visccoeff = 4 nu
// returns 4.mj.nu/(ρi + ρj) (1/r ∂Wij/∂r)
__device__ __forceinline__ float
laminarvisc_kinematic(	const float	rho,
						const float	neib_rho,
						const float	neib_mass,
						const float	f)
{
	return neib_mass*d_visccoeff*f/(rho + neib_rho);
}


// Same behaviour as laminarvisc but for particle
// dependent viscosity.
// returns mj.(µi + µi)/(ρi.ρj) (1/r ∂Wij/∂r)
__device__ __forceinline__ float
laminarvisc_dynamic(const float	rho,
					const float	neib_rho,
					const float	neib_mass,
					const float	f,
					const float	visc,
					const float	neib_visc)
{
	return neib_mass*(visc + neib_visc)*f/(rho*neib_rho);
}
/************************************************************************************************************/


/*********************************** Adptative time stepping ************************************************/
// Function called at the end of the forces or powerlawVisc function doing
// a per block maximum reduction
__device__ __forceinline__ void
dtadaptBlockReduce(	float*	sm_max,
					float*	cfl)
{
	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
	{
		__syncthreads();
		if (threadIdx.x < s) 
		{
			sm_max[threadIdx.x] = max(sm_max[threadIdx.x + s], sm_max[threadIdx.x]);
		}
	}

	// write result for this block to global mem
	if (!threadIdx.x)
		cfl[blockIdx.x] = sm_max[0];
}
/************************************************************************************************************/


/********************************* Neighbor data access management ******************************************/
/// Compute grid position from hash value
/*! Compute the grid position corresponding to the given hash. The position
 * 	should be in the range [0, gridSize.x - 1]x[0, gridSize.y - 1]x[0, gridSize.z - 1].
 *
 *	\param[in] gridHash : hash value
 *
 *	\return grid position
 *
 *	Note : no test is done by this function to ensure that hash value is valid.
 */
__device__ __forceinline__ int3
calcGridPosFromHash(const hashKey fullGridHash)
{
	const uint gridHash = (uint)(fullGridHash >> GRIDHASH_BITSHIFT);
	int3 gridPos;
	int temp = INTMUL(d_gridSize.y, d_gridSize.x);
	gridPos.z = gridHash/temp;
	temp = gridHash - gridPos.z*temp;
	gridPos.y = temp/d_gridSize.x;
	gridPos.x = temp - gridPos.y*d_gridSize.x;

	return gridPos;
}


/// Compute hash value from grid position
/*! Compute the hash value corresponding to the given position. If the position
 * 	is not in the range [0, gridSize.x - 1]x[0, gridSize.y - 1]x[0, gridSize.z - 1]
 * 	we have periodic boundary and the grid position is updated according to the
 * 	chosen periodicity.
 *
 *	\param[in] gridPos : grid position
 *
 *	\return hash value
 *
 *	Note : no test is done by this function to ensure that grid position is within the
 *	range and no clamping is done
 */
//TODO: implement other periodicity than XPERIODIC and templatize
__device__ __forceinline__ uint
calcGridHash(int3 gridPos)
{
	if (gridPos.x < 0) gridPos.x = d_gridSize.x - 1;
	if (gridPos.x >= d_gridSize.x) gridPos.x = 0;
	return INTMUL(INTMUL(gridPos.z, d_gridSize.y), d_gridSize.x) + INTMUL(gridPos.y, d_gridSize.x) + gridPos.x;
}


#define CELLNUMENCODED		(1U<<11)
#define NEIBINDEXMASK		(0x7FF)

/// Return neighbor index and add cell offset vector to current position
/*! For given neighbor data this function compute the neighbor index
 *  and subtract, if necessary, the neighbor cell offset vector to the
 *  current particle position. This last operation is done only
 *  when the neighbor cell change and result is stored in pos_corr.
 *
 *	\param[in] pos : current particle's positions
 *	\param[out] pos_corr : pos - current neighbor cell offset
 *	\param[in] cellStart : cells first particle index
 *	\param[in] neibdata : neighbor data
 *	\param[in,out] neib_cellnum : current neighbor cell number (0...27)
 *	\param[in,out] neib_cell_base_index : index of first particle of the current cell
 *
 * 	\return neighbor index
 *
 * Note: neib_cell_num and neib_cell_base_index must be persistent along
 * getNeibIndex calls.
 */
__device__ __forceinline__ uint
getNeibIndex(const float4	pos,
			float3& 		pos_corr,
			const uint*		cellStart,
			neibdata		neib_data,
			const int3		gridPos,
			char&			neib_cellnum,
			uint&			neib_cell_base_index)
{
	if (neib_data >= CELLNUMENCODED) {
		// Update current neib cell number
		neib_cellnum = (neib_data >> 11) - 1;

		// Compute neighbor index relative to belonging cell
		neib_data &= NEIBINDEXMASK;

		// Substract current cell offset vector to pos
		pos_corr = as_float3(pos) - d_cell_to_offset[neib_cellnum]*d_cellSize;

		// Compute index of the first particle in the current cell
		neib_cell_base_index = cellStart[calcGridHash(gridPos + d_cell_to_offset[neib_cellnum])];
	}

	// Compute and return neighbor index
	return neib_cell_base_index + neib_data;
}
/************************************************************************************************************/


/******************** Functions for computing repulsive force directly from DEM *****************************/
// TODO: check for the maximum timestep

// Normal and viscous force wrt to solid boundary
__device__ __forceinline__ float
PlaneForce(	const float3 &	pos,
			const float 	mass,
			const float4 &	plane,
			const float		l,
			const float3&	vel,
			const float		dynvisc,
			float4&			force)
{
	const float r = abs(dot(pos, as_float3(plane)) + plane.w)/l;
	if (r < d_r0) {
		const float DvDt = LJForce(r);
		// Unitary normal vector of the surface
		const float3 relPos = make_float3(plane)*r/l;

		as_float3(force) += DvDt*relPos;

		// tangential velocity component
		const float3 v_t = vel - dot(vel, relPos)/r*relPos/r; //TODO: check

		// f = -µ u/∆n

		// viscosity
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0-r*r)/(pos.w*r);
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0*3/(M_PI*2)-r*r)/(pos.w*r);
		const float coeff = -dynvisc*d_partsurf/(mass*r);

		// coeff should not be higher than needed to nil v_t in the maximum allowed dt
		// coefficients are negative, so the smallest in absolute value is the biggest

		/*
		float fmag = length(as_float3(force));
		float coeff2 = -sqrt(fmag/slength)/(d_dtadaptfactor*d_dtadaptfactor);
		if (coeff2 < -d_epsartvisc)
			coeff = max(coeff, coeff2);
			*/

		as_float3(force) += coeff*v_t;

		return -coeff;
	}

	return 0.0f;
}

__device__ __forceinline__ float
GeometryForce(	const float3&	pos,
				const float		mass,
				const float3&	vel,
				const float		dynvisc,
				float4&			force)
{
	float coeff_max = 0.0f;
	for (uint i = 0; i < d_numplanes; ++i) {
		float coeff = PlaneForce(pos, mass, d_planes[i], d_plane_div[i], vel, dynvisc, force);
		if (coeff > coeff_max)
			coeff_max = coeff;
	}

	return coeff_max;
}


__device__ __forceinline__ float
DemInterpol(const texture<float, 2, hipReadModeElementType> texref, 
			const float x, 
			const float y)
{
	return tex2D(texref, x/d_ewres + 0.5f, y/d_nsres + 0.5f);
}


__device__ __forceinline__ float
DemLJForce(	const texture<float, 2, hipReadModeElementType> texref,
			const float3&	pos,
			const float		mass,
			const float3&	vel,
			const float		dynvisc,
			float4&			force)
{
	const float z0 = DemInterpol(texref, pos.x, pos.y);
	if (pos.z - z0 < d_demzmin) {
		const float z1 = DemInterpol(texref, pos.x + d_demdx, pos.y);
		const float z2 = DemInterpol(texref, pos.x, pos.y + d_demdy);
		const float a = d_demdy*(z0 - z1);
		const float b = d_demdx*(z0 - z2);
		const float c = d_demdxdy;	// demdx*demdy
		const float d = -a*pos.x - b*pos.y - c*z0;
		const float l = sqrt(a*a+b*b+c*c);
		return PlaneForce(pos, mass, make_float4(a, b, c, d), l, vel, dynvisc, force);
	}
	return 0;
}

/************************************************************************************************************/

/************************************************************************************************************/
/*		   Kernels for computing SPS tensor and SPS viscosity												*/
/************************************************************************************************************/

// Compute the Sub-Particle-Stress (SPS) Tensor matrix for all Particles
// WITHOUT Kernel correction
// Procedure:
// (1) compute velocity gradients
// (2) compute turbulent eddy viscosity (non-dynamic)
// (3) compute turbulent shear stresses
// (4) return SPS tensor matrix (tau) divided by rho^2
template<KernelType kerneltype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SPS, MIN_BLOCKS_SPS)
SPSstressMatrixDevice(	const float4* posArray,
						float2*		tau0,
						float2*		tau1,
						float2*		tau2,
						const uint*	particleHash,
						const uint*	cellStart,
						const neibdata*	neibsList,
						const uint	numParticles,
						const float	slength,
						const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// compute SPS matrix only for fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	// read particle data from sorted arrays
	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	const float4 vel = tex1Dfetch(velTex, index);

	// SPS stress matrix elements
	sym33mat tau;

	// Gradients of the the velocity components
	float3 dvx = make_float3(0.0f);
	float3 dvy = make_float3(0.0f);
	float3 dvz = make_float3(0.0f);

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromHash(particleHash[index]);

	// Persistent variables across getNeibData calls
	char neib_cellnum = -1;
	uint neib_cell_base_index = 0;

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - posArray[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif
		const float r = length(as_float3(relPos));

		// Compute relative velocity
		// Now relVel is a float4 and neib density is stored in relVel.w
		const float4 relVel = as_float3(vel) - tex1Dfetch(velTex, neib_index);
        const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && FLUID(neib_info)) {
			const float f = F<kerneltype>(r, slength)*relPos.w/relVel.w;	// 1/r ∂Wij/∂r Vj

			// Velocity Gradients
			dvx -= relVel.x*as_float3(relPos)*f;	// dvx = -∑mj/ρj vxij (ri - rj)/r ∂Wij/∂r
			dvy -= relVel.y*as_float3(relPos)*f;	// dvy = -∑mj/ρj vyij (ri - rj)/r ∂Wij/∂r
			dvz -= relVel.z*as_float3(relPos)*f;	// dvz = -∑mj/ρj vzij (ri - rj)/r ∂Wij/∂r
			}
		} // end of loop through neighbors

	// Calculate Sub-Particle Scale viscosity
	// and special turbulent terms
	float SijSij_bytwo = 2.0f*(dvx.x*dvx.x + dvy.y*dvy.y + dvz.z*dvz.z);	// 2*SijSij = 2.0((∂vx/∂x)^2 + (∂vy/∂yx)^2 + (∂vz/∂z)^2)
	float temp = dvx.y + dvy.x;		// 2*SijSij += (∂vx/∂y + ∂vy/∂x)^2
	tau.a12 = temp;
	SijSij_bytwo += temp*temp;
	temp = dvx.z + dvz.x;			// 2*SijSij += (∂vx/∂z + ∂vz/∂x)^2
	tau.a13 = temp;
	SijSij_bytwo += temp*temp;
	temp = dvy.z + dvz.y;			// 2*SijSij += (∂vy/∂z + ∂vz/∂y)^2
	tau.a23 = temp;
	SijSij_bytwo += temp*temp;
	float S = sqrtf(SijSij_bytwo);
	float nu_SPS = d_smagfactor*S;		// Dalrymple & Rogers (2006): eq. (12)
	float divu_SPS = 0.6666666666f*nu_SPS*(dvx.x + dvy.y + dvz.z);
	float Blinetal_SPS = d_kspsfactor*SijSij_bytwo;

	// Shear Stress matrix = TAU (pronounced taf)
	// Dalrymple & Rogers (2006): eq. (10)
	tau.a11 = nu_SPS*(dvx.x + dvx.x) - divu_SPS - Blinetal_SPS;	// tau11 = tau_xx/ρ^2
	tau.a11 /= vel.w;
	tau.a12 *= nu_SPS/vel.w;								// tau12 = tau_xy/ρ^2
	tau.a13 *= nu_SPS/vel.w;								// tau13 = tau_xz/ρ^2
	tau.a22 = nu_SPS*(dvy.y + dvy.y) - divu_SPS - Blinetal_SPS;	// tau22 = tau_yy/ρ^2
	tau.a22 /= vel.w;
	tau.a23 *= nu_SPS/vel.w;								// tau23 = tau_yz/ρ^2
	tau.a33 = nu_SPS*(dvz.z + dvz.z) - divu_SPS - Blinetal_SPS;	// tau33 = tau_zz/ρ^2
	tau.a33 /= vel.w;

	tau0[index] = make_float2(tau.a11, tau.a12);
	tau1[index] = make_float2(tau.a13, tau.a22);
	tau2[index] = make_float2(tau.a23, tau.a33);
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   Kernels for computing acceleration without gradient correction					 */
/************************************************************************************************************/

/* Normal kernels */
#include "forces_kernel.xsphdt.inc"

/************************************************************************************************************/


/************************************************************************************************************/
/*					   Kernels for XSPH, Shepard and MLS corrections									   */
/************************************************************************************************************/

// This kernel computes the Sheppard correction
template<KernelType kerneltype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
shepardDevice(	const float4*	posArray,
				float4*			newVel,
				const uint*		particleHash,
				const uint*		cellStart,
				const neibdata*	neibsList,
				const uint		numParticles,
				const float		slength,
				const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// normalize kernel only if the given particle is a fluid one
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 vel = tex1Dfetch(velTex, index);

	// taking into account self contribution in summation
	float temp1 = pos.w*W<kerneltype>(0, slength);
	float temp2 = temp1/vel.w ;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromHash(particleHash[index]);

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;

	// loop over all the neighbors
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - posArray[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif
		const float r = length(as_float3(relPos));

		const float neib_rho = tex1Dfetch(velTex, neib_index).w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && FLUID(neib_info)) {
			const float w = W<kerneltype>(r, slength)*relPos.w;
			temp1 += w;
			temp2 += w/neib_rho;
		}
	}

	vel.w = temp1/temp2;
	newVel[index] = vel;
}


// This kernel computes the MLS correction
template<KernelType kerneltype>
__global__ void
__launch_bounds__(BLOCK_SIZE_MLS, MIN_BLOCKS_MLS)
MlsDevice(	const float4*	posArray,
			float4*			newVel,
			const uint*		particleHash,
			const uint*		cellStart,
			const neibdata*	neibsList,
			const uint		numParticles,
			const float		slength,
			const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// computing MLS matrix only for fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 vel = tex1Dfetch(velTex, index);

	// MLS matrix elements
	float a11 = 0.0f, a12 = 0.0f, a13 = 0.0f, a14 = 0.0f;
	float a22 = 0.0f, a23 = 0.0f, a24 = 0.0f;
	float a33 = 0.0f, a34 = 0.0f;
	float a44 = 0.0f;

	// number of neighbors
	int neibs_num = 0;

	// taking into account self contribution in MLS matrix construction
	a11 = W<kerneltype>(0, slength)*pos.w/vel.w;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromHash(particleHash[index]);

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;

	// First loop over all neighbors
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - posArray[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif
		const float r = length(as_float3(relPos));

		const float neib_rho = tex1Dfetch(velTex, neib_index).w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// interaction between two particles
		if (r < influenceradius && FLUID(neib_info)) {
			neibs_num ++;
			const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;	// Wij*Vj
			a11 += w;						// a11 = ∑Wij*Vj
			a12 += relPos.x*w;				// a12 = ∑(xi - xj)*Wij*Vj
			a13 += relPos.y*w;				// a13 = ∑(yi - yj)*Wij*Vj
			a14 += relPos.z*w;				// a14 = ∑(zi - zj)*Wij*Vj
			a22 += relPos.x*relPos.x*w;		// a22 = ∑(xi - xj)^2*Wij*Vj
			a23 += relPos.x*relPos.y*w;		// a23 = ∑(xi - xj)(yi - yj)*Wij*Vj
			a24 += relPos.x*relPos.z*w;		// a23 = ∑(xi - xj)(zi - zj)*Wij*Vj
			a33 += relPos.y*relPos.y*w;		// a33 = ∑(yi - yj)^2*Wij*Vj
			a34 += relPos.y*relPos.z*w;		// a33 = ∑(yi - yj)(zi - zj)*Wij*Vj
			a44 += relPos.z*relPos.z*w;		// a33 = ∑(yi - yj)^2*Wij*Vj
		}
	} // end of first loop trough neighbors

	// Resetting persistent variables across getNeibData
	neib_cellnum = 0;
	neib_cell_base_index = 0;

	// safe inverse of MLS matrix
	// the matrix is inverted only if |det|/max|aij|^4 > EPSDET
	// and if the number of fluids neighbors if above a minimum
	// value, otherwise no correction is applied
	float maxa = fmaxf(fabsf(a11), fabsf(a12));
	maxa = fmaxf(maxa, fabsf(a13));
	maxa = fmaxf(maxa, fabsf(a14));
	maxa = fmaxf(maxa, fabsf(a22));
	maxa = fmaxf(maxa, fabsf(a23));
	maxa = fmaxf(maxa, fabsf(a24));
	maxa = fmaxf(maxa, fabsf(a33));
	maxa = fmaxf(maxa, fabsf(a34));
	maxa = fmaxf(maxa, fabsf(a44));
	maxa *= maxa;
	maxa *= maxa;
	float det = a11*(a22*a33*a44 + a23*a34*a24 + a24*a23*a34 - a22*a34*a34 - a23*a23*a44 - a24*a33*a24)
			  + a12*(a12*a34*a34 + a23*a13*a44 + a24*a33*a14 - a12*a33*a44 - a23*a34*a14 - a24*a13*a34)
			  + a13*(a12*a23*a44 + a22*a34*a14 + a24*a13*a24 - a12*a34*a24 - a22*a13*a44 - a24*a23*a14)
			  + a14*(a12*a33*a24 + a22*a13*a34 + a23*a23*a14 - a12*a23*a34 - a22*a33*a14 - a23*a13*a24);
	if (det > maxa*EPSDETMLS && neibs_num > MINCORRNEIBSMLS) {  // FIXME: should be |det| ?????
		// first row of inverse matrix
		det = 1/det;
		const float b11 = (a22*a33*a44 + a23*a34*a24 + a24*a23*a34 - a22*a34*a34 - a23*a23*a44 - a24*a33*a24)*det;
		const float b21 = (a12*a34*a34 + a23*a13*a44 + a24*a33*a14 - a12*a33*a44 - a23*a34*a14 - a24*a13*a34)*det;
		const float b31 = (a12*a23*a44 + a22*a34*a14 + a24*a13*a24 - a12*a34*a24 - a22*a13*a44 - a24*a23*a14)*det;
		const float b41 = (a12*a33*a24 + a22*a13*a34 + a23*a23*a14 - a12*a23*a34 - a22*a33*a14 - a23*a13*a24)*det;

		// taking into account self contribution in density summation
		vel.w = b11*W<kerneltype>(0, slength)*pos.w;

		// loop over all the neighbors (Second loop)
		for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
			neibdata neib_data = neibsList[i + index];

			if (neib_data == 0xffff) break;

			float3 pos_corr;
			const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
						neib_cellnum, neib_cell_base_index);

			// Compute relative position vector and distance
			// Now relPos is a float4 and neib mass is stored in relPos.w
			#if( __COMPUTE__ >= 20)
			const float4 relPos = pos_corr - posArray[neib_index];
			#else
			const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
			#endif
			const float r = length(as_float3(relPos));

			const float neib_rho = tex1Dfetch(velTex, neib_index).w;
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			// interaction between two particles
			if (r < influenceradius && FLUID(neib_info)) {
				const float w = W<kerneltype>(r, slength)*relPos.w;	 // ρj*Wij*Vj = mj*Wij
				vel.w += (b11 + b21*relPos.x + b31*relPos.y
							+ b41*relPos.z)*w;	 // ρ = ∑(ß0 + ß1(xi - xj) + ß2(yi - yj))*Wij*Vj
			}
		}  // end of second loop trough neighbors
	} else {
		// Resort to Sheppard filter in absence of invertible matrix
		// see also shepardDevice. TODO: share the code
		// we use a11 and a12 for temp1, temp2
		a11 = pos.w*W<kerneltype>(0, slength);
		a12 = a11/vel.w;

		// loop over all the neighbors (Second loop)
		for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
			neibdata neib_data = neibsList[i + index];

			if (neib_data == 0xffff) break;

			float3 pos_corr;
			const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
						neib_cellnum, neib_cell_base_index);

			// Compute relative position vector and distance
			// Now relPos is a float4 and neib mass is stored in relPos.w
			#if( __COMPUTE__ >= 20)
			const float4 relPos = pos_corr - posArray[neib_index];
			#else
			const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
			#endif
			const float r = length(as_float3(relPos));

			const float neib_rho = tex1Dfetch(velTex, neib_index).w;
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			// interaction between two particles
			if (r < influenceradius && FLUID(neib_info)) {
					// ρj*Wij*Vj = mj*Wij
					const float w = W<kerneltype>(r, slength)*relPos.w;
					// ρ = ∑(ß0 + ß1(xi - xj) + ß2(yi - yj))*Wij*Vj
					a11 += w;
					a12 +=w/neib_rho;
			}
		}  // end of second loop through neighbors

		vel.w = a11/a12;
	}

	newVel[index] = vel;
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   CFL max kernel																		*/
/************************************************************************************************************/
template <unsigned int blockSize>
__global__ void
fmaxDevice(float *g_idata, float *g_odata, const uint n)
{
	extern __shared__ float sdata[];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	float myMax = 0;

	// we reduce multiple elements per thread.  The number is determined by the 
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{         
		myMax = max(myMax, g_idata[i]);
		// ensure we don't read out of bounds
		if (i + blockSize < n) 
			myMax = max(myMax, g_idata[i + blockSize]);
		i += gridSize;
	} 

	// each thread puts its local sum into shared memory 
	sdata[tid] = myMax;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = myMax = max(myMax,sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = myMax = max(myMax,sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = myMax = max(myMax,sdata[tid +  64]); } __syncthreads(); }

	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and induce incorrect behavior.
	if (tid < 32)
	{
		volatile float* smem = sdata;
		if (blockSize >=  64) { smem[tid] = myMax = max(myMax, smem[tid + 32]); }
		if (blockSize >=  32) { smem[tid] = myMax = max(myMax, smem[tid + 16]); }
		if (blockSize >=  16) { smem[tid] = myMax = max(myMax, smem[tid +  8]); }
		if (blockSize >=   8) { smem[tid] = myMax = max(myMax, smem[tid +  4]); }
		if (blockSize >=   4) { smem[tid] = myMax = max(myMax, smem[tid +  2]); }
		if (blockSize >=   2) { smem[tid] = myMax = max(myMax, smem[tid +  1]); }
	}

	// write result for this block to global mem 
	if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   Parallel reduction kernels															*/
/************************************************************************************************************/

extern __shared__ float4 shmem4[];

extern "C" __global__
void calcEnergies(
		const float4* pPos,
		const float4* pVel,
		const particleinfo* pInfo,
		uint	numParticles,
		uint	numFluids,
		float4* output
		)
{
	// shared memory for this kernel should be sized to
	// blockDim.x*numFluids*sizeof(float4)*2

	uint gid = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	uint stride = INTMUL(gridDim.x,blockDim.x);
	// .x kinetic, .y potential, .z internal
	float4 energy[MAX_FLUID_TYPES], E_k[MAX_FLUID_TYPES];

#pragma unroll
	for (uint i = 0; i < MAX_FLUID_TYPES; ++i)
		energy[i] = E_k[i] = make_float4(0.0f);

	while (gid < numParticles) {
		float4 pos = pPos[gid];
		float4 vel = pVel[gid];
		particleinfo pinfo = pInfo[gid];
		if (FLUID(pinfo)) {
			uint fluid_num = PART_FLUID_NUM(pinfo);
			float v2 = kahan_sqlength(as_float3(vel));
			float gh = kahan_dot(d_gravity, as_float3(pos));
			kahan_add(energy[fluid_num].x, pos.w*v2/2, E_k[fluid_num].x);
			kahan_add(energy[fluid_num].y, -pos.w*gh, E_k[fluid_num].y);
			// internal elastic energy
			float gamma = d_gammacoeff[fluid_num];
			float gm1 = d_gammacoeff[fluid_num]-1;
			float rho0 = d_rho0[fluid_num];
			float elen = __powf(vel.w/rho0, gm1)/gm1 + rho0/vel.w - gamma/gm1;
			float ssp = soundSpeed(vel.w, fluid_num);
			elen *= ssp*ssp/gamma;
			kahan_add(energy[fluid_num].z, pos.w*elen, E_k[fluid_num].z);
		}
		gid += stride;
	}

	uint lid = threadIdx.x;
	for (uint offset = blockDim.x/2; offset; offset >>= 1) {
		stride = offset*numFluids; // stride between fields in shmem4 memory
		if (lid >= offset && lid < 2*offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = lid + offset*i;
				shmem4[idx] = energy[i];
				idx += stride;
				shmem4[idx] = E_k[i];
			}
		}
		__syncthreads();
		if (lid < offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = lid + offset*(i+1);
				float4 other = shmem4[idx];
				idx += stride;
				float4 oth_k = shmem4[idx];
				kahan_add(energy[i].x, oth_k.x, E_k[i].x);
				kahan_add(energy[i].x, other.x, E_k[i].x);
				kahan_add(energy[i].y, oth_k.y, E_k[i].y);
				kahan_add(energy[i].y, other.y, E_k[i].y);
				kahan_add(energy[i].z, oth_k.z, E_k[i].z);
				kahan_add(energy[i].z, other.z, E_k[i].z);
			}
		}
	}

	if (lid == 0) {
		for (uint i = 0; i < numFluids; ++i) {
			output[blockIdx.x + INTMUL(gridDim.x,i)] = energy[i];
			output[blockIdx.x + INTMUL(gridDim.x,numFluids+i)] = E_k[i];
		}
	}
}

// final reduction stage
extern "C" __global__
void calcEnergies2(
		float4* buffer,
		uint	prev_blocks,
		uint	numFluids)
{
	// shared memory for this kernel should be sized to
	// blockDim.x*numFluids*sizeof(float4)*2

	uint gid = threadIdx.x;
	float4 energy[MAX_FLUID_TYPES];
	float4 E_k[MAX_FLUID_TYPES];
	for (uint i = 0; i < numFluids; ++i) {
		if (gid < prev_blocks) {
			energy[i] = buffer[gid + prev_blocks*i];
			E_k[i] = buffer[gid + prev_blocks*(numFluids+i)];
		} else {
			energy[i] = E_k[i] = make_float4(0.0f);
		}
	}

	uint stride;
	for (uint offset = blockDim.x/2; offset; offset >>= 1) {
		stride = offset*numFluids; // stride between fields in shmem4 memory
		if (gid >= offset && gid < 2*offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = gid + offset*i;
				shmem4[idx] = energy[i];
				idx += stride;
				shmem4[idx] = E_k[i];
			}
		}
		__syncthreads();
		if (gid < offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = gid + offset*(i+1);
				float4 other = shmem4[idx];
				idx += stride;
				float4 oth_k = shmem4[idx];
				kahan_add(energy[i].x, oth_k.x, E_k[i].x);
				kahan_add(energy[i].x, other.x, E_k[i].x);
				kahan_add(energy[i].y, oth_k.y, E_k[i].y);
				kahan_add(energy[i].y, other.y, E_k[i].y);
				kahan_add(energy[i].z, oth_k.z, E_k[i].z);
				kahan_add(energy[i].z, other.z, E_k[i].z);
			}
		}
	}

	if (gid == 0) {
		for (uint i = 0; i < numFluids; ++i)
			buffer[i] = energy[i] + E_k[i];
	}
}


/************************************************************************************************************/
/*					   Auxiliary kernels used for post processing										    */
/************************************************************************************************************/

// This kernel compute the vorticity field
template<KernelType kerneltype>
__global__ void
calcVortDevice(	float3*		vorticity,
				const uint*	particleHash,
				const uint*	cellStart,
				const neibdata*	neibsList,
				const uint	numParticles,
				const float	slength,
				const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// computing vorticity only for fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);
	if (NOT_FLUID(info))
		return;

	const float4 pos = tex1Dfetch(posTex, index);
	const float4 vel = tex1Dfetch(velTex, index);

	float3 vort = make_float3(0.0f);

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromHash(particleHash[index]);

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;

	// First loop over all neighbors
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		const float r = length(as_float3(relPos));

		// Compute relative velocity
		// Now relVel is a float4 and neib density is stored in relVel.w
		const float4 relVel = as_float3(vel) - tex1Dfetch(velTex, neib_index);
        const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Compute vorticity
		if (r < influenceradius && FLUID(neib_info)) {
			const float f = F<kerneltype>(r, slength)*relPos.w/relVel.w;	// ∂Wij/∂r*Vj
			// vxij = vxi - vxj and same for vyij and vzij
			vort.x += f*(relVel.y*relPos.z - relVel.z*relPos.y);		// vort.x = ∑(vyij(zi - zj) - vzij*(yi - yj))*∂Wij/∂r*Vj
			vort.y += f*(relVel.z*relPos.x - relVel.x*relPos.z);		// vort.y = ∑(vzij(xi - xj) - vxij*(zi - zj))*∂Wij/∂r*Vj
			vort.z += f*(relVel.x*relPos.y - relVel.y*relPos.x);		// vort.x = ∑(vxij(yi - yj) - vyij*(xi - xj))*∂Wij/∂r*Vj
		}
	} // end of loop trough neighbors

	vorticity[index] = vort;
}


// Testpoints
// This kernel compute the velocity at testpoints
template<KernelType kerneltype>
__global__ void
calcTestpointsVelocityDevice(	float4*		newVel,
								const uint*	particleHash,
								const uint*	cellStart,
								const neibdata*	neibsList,
								const uint	numParticles,
								const float	slength,
								const float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	const particleinfo info = tex1Dfetch(infoTex, index);
	if(type(info) != TESTPOINTSPART)
		return;
	
	const float4 pos = tex1Dfetch(posTex, index);
	float4 vel = tex1Dfetch(velTex, index);
	
	float4 temp = make_float4(0.0f);

	// Compute grid position of current particle
	int3 gridPos = calcGridPosFromHash(particleHash[index]);

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;

	// First loop over all neighbors
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		const float r = length(as_float3(relPos));

		const float4 neib_vel = tex1Dfetch(velTex, neib_index);
        const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius && FLUID(neib_info)) {
			const float w = W<kerneltype>(r, slength)*relPos.w/neib_vel.w;	// Wij*mj
			temp.x += w*neib_vel.x;
			temp.y += w*neib_vel.y;
			temp.z += w*neib_vel.z;
			//Pressure
			temp.w += w*P(neib_vel.w, object(neib_info));

		}
	}

	vel = temp;

	newVel[index] = vel;
}


// Free surface detection
// This kernel detects the surface particles
template<KernelType kerneltype, bool savenormals>
__global__ void
calcSurfaceparticleDevice(	float4*			normals,
							particleinfo*	newInfo,
							const uint*		particleHash,
							const uint*		cellStart,
							const neibdata*	neibsList,
							const uint		numParticles,
							const float		slength,
							const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	particleinfo info = tex1Dfetch(infoTex, index);

	if (NOT_FLUID(info)) {
		newInfo[index] = info;		
		return;
	}

	const float4 pos = tex1Dfetch(posTex, index);
	float4 normal = make_float4(0.0f);
	
	// Compute grid position of current particle
	int3 gridPos = calcGridPosFromHash(particleHash[index]);

	info.x &= ~SURFACE_PARTICLE_FLAG;
	normal.w = W<kerneltype>(0.0f, slength)*pos.w;

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;

	// First loop over all neighbors
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		const float r = length(as_float3(relPos));

		const float neib_density = tex1Dfetch(velTex, neib_index).w;

		if (r < influenceradius) {
			const float f = F<kerneltype>(r, slength)*relPos.w /neib_density; // 1/r ∂Wij/∂r Vj
			normal.x -= f * relPos.x;
			normal.y -= f * relPos.y;
			normal.z -= f * relPos.z;
			normal.w += W<kerneltype>(r, slength)*relPos.w;	// Wij*mj ;

		}
	}

	float normal_length = length(as_float3(normal));

	//Checking the planes
	// TODO: fix me for homogenous precision
	for (uint i = 0; i < d_numplanes; ++i) {
		float r = abs(dot(as_float3(pos), as_float3(d_planes[i])) + d_planes[i].w)/d_plane_div[i];
		if (r < influenceradius) {
			as_float3(normal) += as_float3(d_planes[i])* normal_length;
			normal_length = length(as_float3(normal));
		}
	}

	// Second loop over all neighbors

	// Resetting grid position of current particle
	gridPos = calcGridPosFromHash(particleHash[index]);

	// Resetting persistent variables across getNeibData
	neib_cellnum = 0;
	neib_cell_base_index = 0;

	// loop over all the neighbors (Second loop)
	int nc = 0;
	for(uint i = 0; i < d_maxneibsnum_time_numparticles; i += numParticles) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		float3 pos_corr;
		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		const float r = length(as_float3(relPos));

		float cosconeangle;

		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius) {
			float criteria = -(normal.x * relPos.x + normal.y * relPos.y + normal.z * relPos.z);
			if (FLUID(neib_info))
				cosconeangle = d_cosconeanglefluid;
			else
				cosconeangle = d_cosconeanglenonfluid;

			if (criteria > r*normal_length*cosconeangle)
				nc++;
		}

	}

	if (!nc)
		info.x |= SURFACE_PARTICLE_FLAG;

	newInfo[index] = info;

	if (savenormals) {
		normal.x /= normal_length;
		normal.y /= normal_length;
		normal.z /= normal_length;
		normals[index] = normal;
		}

}
/************************************************************************************************************/
}
#endif
