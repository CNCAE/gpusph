#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdexcept>

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/functional.h>

#include "textures.cuh"
#include "forces.cuh"

#include "utils.h"
#include "cuda_call.h"

#include "forces_params.h"

hipArray*  dDem = NULL;

/* Auxiliary data for parallel reductions */
size_t	reduce_blocks = 0;
size_t	reduce_blocksize_max = 0;
size_t	reduce_bs2 = 0;
size_t	reduce_shmem_max = 0;
void*	reduce_buffer = NULL;

#include "forces_kernel.cu"

/// static inline methods for fmax reduction

static inline void
reducefmax(	const int	size,
			const int	threads,
			const int	blocks,
			float		*d_idata,
			float		*d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

	switch (threads)
	{
		case 512:
			cuforces::fmaxDevice<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 256:
			cuforces::fmaxDevice<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 128:
			cuforces::fmaxDevice<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 64:
			cuforces::fmaxDevice<64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 32:
			cuforces::fmaxDevice<32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 16:
			cuforces::fmaxDevice<16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  8:
			cuforces::fmaxDevice<8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  4:
			cuforces::fmaxDevice<4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  2:
			cuforces::fmaxDevice<2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  1:
			cuforces::fmaxDevice<1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	}
}


static inline uint nextPow2(uint x )
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


#define MIN(x,y) ((x < y) ? x : y)
static inline void
getNumBlocksAndThreads(	const uint	n,
						const uint	maxBlocks,
						const uint	maxThreads,
						uint		&blocks,
						uint		&threads)
{
	threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
	blocks = MIN(maxBlocks, blocks);
}

static inline float
cflmax( const uint	n,
		float*		cfl,
		float*		tempCfl)
{
	uint numBlocks = 0;
	uint numThreads = 0;
	float max = 0.0f;

	getNumBlocksAndThreads(n, MAX_BLOCKS_FMAX, BLOCK_SIZE_FMAX, numBlocks, numThreads);

	// execute the kernel
	reducefmax(n, numThreads, numBlocks, cfl, tempCfl);

	// check if kernel execution generated an error
	CUT_CHECK_ERROR("fmax kernel execution failed");

	// TODO this can be done in just two calls
	uint s = numBlocks;
	while(s > 1)
	{
		uint threads = 0, blocks = 0;
		getNumBlocksAndThreads(s, MAX_BLOCKS_FMAX, BLOCK_SIZE_FMAX, blocks, threads);

		reducefmax(s, threads, blocks, tempCfl, tempCfl);
		CUT_CHECK_ERROR("fmax kernel execution failed");

		s = (s + (threads*2-1)) / (threads*2);
	}

	CUDA_SAFE_CALL(hipMemcpy(&max, tempCfl, sizeof(float), hipMemcpyDeviceToHost));

	return max;
}



/// Methods of the CUDAForcesEngine class

// Since for the time being each method needs a
//   template<blah blah blah> [return type] class CUDAForces<blahblahblah>::
// before it, and it's boring to type, we do a small auxiliary macro FORCES_RET
// that wraps the return type.
// TODO this will go away when the class will be turned into an actual include-only
// template class definition

#define FORCES_RET(ret_type) \
template< \
	KernelType kerneltype, \
	SPHFormulation sph_formulation, \
	ViscosityType visctype, \
	BoundaryType boundarytype, \
	flag_t simflags \
> \
ret_type \
CUDAForcesEngine<kerneltype, sph_formulation, visctype, boundarytype, simflags>::

FORCES_RET(bool)
needs_eulerVel = (boundarytype == SA_BOUNDARY &&
			(visctype == KEPSVISC || (simflags & ENABLE_INLET_OUTLET)));


FORCES_RET(void)
setconstants(const SimParams *simparams, const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles)
{
	// Setting kernels and kernels derivative factors
	float h = simparams->slength;
	float h3 = h*h*h;
	float h4 = h3*h;
	float h5 = h4*h;
	float kernelcoeff = 1.0f/(M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_wcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_wcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 21.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_wcoeff_wendland), &kernelcoeff, sizeof(float)));

	kernelcoeff = 3.0f/(4.0f*M_PI*h4);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_fcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(32.0f*M_PI*h4);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_fcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 105.0f/(128.0f*M_PI*h5);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_fcoeff_wendland), &kernelcoeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_numfluids), &physparams->numFluids, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_rho0), &physparams->rho0, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_bcoeff), &physparams->bcoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gammacoeff), &physparams->gammacoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_sscoeff), &physparams->sscoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_sspowercoeff), &physparams->sspowercoeff, MAX_FLUID_TYPES*sizeof(float)));

	// compute (and upload) square of sound speeds, needed for Ferrari
	float sqC0[MAX_FLUID_TYPES];
	for (uint i = 0; i < MAX_FLUID_TYPES; ++i) {
		sqC0[i]  = physparams->sscoeff[i];
		sqC0[i] *= sqC0[i];
	}
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_sqC0), sqC0, MAX_FLUID_TYPES*sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gravity), &physparams->gravity, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_dcoeff), &physparams->dcoeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_p1coeff), &physparams->p1coeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_p2coeff), &physparams->p2coeff, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_MK_K), &physparams->MK_K, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_MK_d), &physparams->MK_d, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_MK_beta), &physparams->MK_beta, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_r0), &physparams->r0, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_visccoeff), &physparams->visccoeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_epsartvisc), &physparams->epsartvisc, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_ewres), &physparams->ewres, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_nsres), &physparams->nsres, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demdx), &physparams->demdx, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demdy), &physparams->demdy, sizeof(float)));
	float demdxdy = physparams->demdx*physparams->demdy;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demdxdy), &demdxdy, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demzmin), &physparams->demzmin, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_smagfactor), &physparams->smagfactor, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_kspsfactor), &physparams->kspsfactor, sizeof(float)));

	float partsurf = physparams->partsurf;
	if (partsurf == 0.0f)
		partsurf = physparams->r0*physparams->r0;
		// partsurf = (6.0 - M_PI)*physparams->r0*physparams->r0/4;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_partsurf), &partsurf, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_cosconeanglefluid), &physparams->cosconeanglefluid, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_cosconeanglenonfluid), &physparams->cosconeanglenonfluid, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_objectobjectdf), &physparams->objectobjectdf, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_objectboundarydf), &physparams->objectboundarydf, sizeof(float)));

	idx_t neiblist_end = simparams->maxneibsnum*allocatedParticles;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_neiblist_end), &neiblist_end, sizeof(idx_t)));

	// Neibs cell to offset table
	char3 cell_to_offset[27];
	for(char z=-1; z<=1; z++) {
		for(char y=-1; y<=1; y++) {
			for(char x=-1; x<=1; x++) {
				int i = (x + 1) + (y + 1)*3 + (z + 1)*9;
				cell_to_offset[i] =  make_char3(x, y, z);
			}
		}
	}
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_cell_to_offset), cell_to_offset, 27*sizeof(char3)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gridSize), &gridSize, sizeof(uint3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_ferrari), &simparams->ferrari, sizeof(float)));
}


FORCES_RET(void)
getconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->numFluids, HIP_SYMBOL(cuforces::d_numfluids), sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->rho0, HIP_SYMBOL(cuforces::d_rho0), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->gravity, HIP_SYMBOL(cuforces::d_gravity), sizeof(float3), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->bcoeff, HIP_SYMBOL(cuforces::d_bcoeff), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->gammacoeff, HIP_SYMBOL(cuforces::d_gammacoeff), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->sscoeff, HIP_SYMBOL(cuforces::d_sscoeff), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->sspowercoeff, HIP_SYMBOL(cuforces::d_sspowercoeff), MAX_FLUID_TYPES*sizeof(float), 0));

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->dcoeff, HIP_SYMBOL(cuforces::d_dcoeff), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->p1coeff, HIP_SYMBOL(cuforces::d_p1coeff), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->p2coeff, HIP_SYMBOL(cuforces::d_p2coeff), sizeof(float), 0));

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->MK_K, HIP_SYMBOL(cuforces::d_MK_K), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->MK_d, HIP_SYMBOL(cuforces::d_MK_d), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->MK_beta, HIP_SYMBOL(cuforces::d_MK_beta), sizeof(float), 0));

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->r0, HIP_SYMBOL(cuforces::d_r0), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->visccoeff, HIP_SYMBOL(cuforces::d_visccoeff), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsartvisc, HIP_SYMBOL(cuforces::d_epsartvisc), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->ewres, HIP_SYMBOL(cuforces::d_ewres), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->nsres, HIP_SYMBOL(cuforces::d_nsres), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->demdx, HIP_SYMBOL(cuforces::d_demdx), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->demdy, HIP_SYMBOL(cuforces::d_demdy), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->demzmin, HIP_SYMBOL(cuforces::d_demzmin), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->smagfactor, HIP_SYMBOL(cuforces::d_smagfactor), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->kspsfactor, HIP_SYMBOL(cuforces::d_kspsfactor), sizeof(float)));
}

FORCES_RET(void)
setplanes(int numPlanes, const float *planesDiv, const float4 *planes)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_planes), planes, numPlanes*sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_plane_div), planesDiv, numPlanes*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_numplanes), &numPlanes, sizeof(uint)));
}

FORCES_RET(void)
setgravity(float3 const& gravity)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gravity), &gravity, sizeof(float3)));
}

FORCES_RET(void)
setrbcg(const float3* cg, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_rbcg), cg, numbodies*sizeof(float3)));
}

FORCES_RET(void)
setrbstart(const int* rbfirstindex, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_rbstartindex), rbfirstindex, numbodies*sizeof(int)));
}

FORCES_RET(void)
bind_textures(
	const	float4	*pos,
	const	float4	*vel,
	const	float4	*eulerVel,
	const	float4	*oldGGam,
	const	float4	*boundelem,
	const	particleinfo	*info,
	const	float	*keps_tke,
	const	float	*keps_eps,
	uint	numParticles)
{
	// bind textures to read all particles, not only internal ones
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	if (needs_eulerVel) {
		if (!eulerVel)
			throw std::invalid_argument("eulerVel not set but needed");
		CUDA_SAFE_CALL(hipBindTexture(0, eulerVelTex, eulerVel, numParticles*sizeof(float4)));
	} else {
		if (eulerVel)
			cerr << "eulerVel set but not used" << endl;
	}

	if (boundarytype == SA_BOUNDARY) {
		CUDA_SAFE_CALL(hipBindTexture(0, gamTex, oldGGam, numParticles*sizeof(float4)));
		CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelem, numParticles*sizeof(float4)));
	}

	if (visctype == KEPSVISC) {
		CUDA_SAFE_CALL(hipBindTexture(0, keps_kTex, keps_tke, numParticles*sizeof(float)));
		CUDA_SAFE_CALL(hipBindTexture(0, keps_eTex, keps_eps, numParticles*sizeof(float)));
	}
}

FORCES_RET(void)
unbind_textures()
{
	// TODO FIXME why are SPS textures unbound here but bound in sps?
	// shouldn't we bind them in bind_textures() instead?
	if (visctype == SPSVISC) {
		CUDA_SAFE_CALL(hipUnbindTexture(tau0Tex));
		CUDA_SAFE_CALL(hipUnbindTexture(tau1Tex));
		CUDA_SAFE_CALL(hipUnbindTexture(tau2Tex));
	}

	if (visctype == KEPSVISC) {
		CUDA_SAFE_CALL(hipUnbindTexture(keps_kTex));
		CUDA_SAFE_CALL(hipUnbindTexture(keps_eTex));
	}

	if (boundarytype == SA_BOUNDARY) {
		CUDA_SAFE_CALL(hipUnbindTexture(gamTex));
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	}

	if (needs_eulerVel)
		CUDA_SAFE_CALL(hipUnbindTexture(eulerVelTex));

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
}

// returns the number of elements in the (starting) fmax array, assuming n particles.
// this is _exactly_ the number of blocks in the grid launch for the forces kernel over n
// particles, since the forces kernel pre-reduces the cfl values, producing one value
// per block instead of one per particle
// TODO FIXME reorganize this reduction stuff
FORCES_RET(uint)
getFmaxElements(const uint n)
{
	return div_up(n, min(BLOCK_SIZE_FORCES, n));
}


FORCES_RET(uint)
getFmaxTempElements(const uint n)
{
	uint numBlocks, numThreads;
	getNumBlocksAndThreads(n, MAX_BLOCKS_FMAX, BLOCK_SIZE_FMAX, numBlocks, numThreads);
	return numBlocks;
}



FORCES_RET(float)
dtreduce(	float	slength,
			float	dtadaptfactor,
			float	visccoeff,
			float	*cfl,
			float	*cflTVisc,
			float	*tempCfl,
			uint	numBlocks)
{
	// cfl holds one value per block in the forces kernel call,
	// so it holds numBlocks elements
	float maxcfl = cflmax(numBlocks, cfl, tempCfl);
	float dt = dtadaptfactor*sqrtf(slength/maxcfl);

	if (visctype != ARTVISC) {
		/* Stability condition from viscosity h²/ν */
		float dt_visc = slength*slength/visccoeff;
		switch (visctype) {
			case KINEMATICVISC:
			case SPSVISC:
			/* ν = visccoeff/4 for kinematic viscosity */
				dt_visc *= 4;
				break;

			case DYNAMICVISC:
			/* ν = visccoeff for dynamic viscosity */
				break;
			case KEPSVISC:
				dt_visc = slength*slength/(visccoeff + cflmax(numBlocks, cflTVisc, tempCfl));
				break;
			default:
				throw invalid_argument("unknown viscosity in dtreduce");
			}
		dt_visc *= 0.125;
		if (dt_visc < dt)
			dt = dt_visc;
	}

	// check if last kernel invocation generated an error
	CUT_CHECK_ERROR("Forces kernel execution failed");

	return dt;
}

// Returns numBlock for delayed dt reduction in case of striping
FORCES_RET(uint)
basicstep(
	const	float4	*pos,
	const	float2	* const vertPos[],
	const	float4	*vel,
			float4	*forces,
			float2	*contupd,
	const	float4	*oldGGam,
			float4	*newGGam,
	const	float4	*boundelem,
			float4	*rbforces,
			float4	*rbtorques,
			float4	*xsph,
	const	particleinfo	*info,
	const	hashKey	*particleHash,
	const	uint	*cellStart,
	const	neibdata*neibsList,
			uint	numParticles,
			uint	fromParticle,
			uint	toParticle,
			float	deltap,
			float	slength,
			float	dtadaptfactor,
			float	influenceradius,
	const	float	epsilon,
			uint	*IOwaterdepth,
			float	visccoeff,
			float	*turbvisc,
			float	*keps_tke,
			float	*keps_eps,
			float3	*keps_dkde,
			float	*cfl,
			float	*cflTVisc,
			float	*tempCfl,
			uint	cflOffset)
{
	int dummy_shared = 0;

	const uint numParticlesInRange = toParticle - fromParticle;
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_FORCES, numParticlesInRange);
	uint numBlocks = div_up(numParticlesInRange, numThreads);
	#if (__COMPUTE__ == 20)
	if (visctype == SPSVISC)
		dummy_shared = 3328 - dtadapt*BLOCK_SIZE_FORCES*4;
	else
		dummy_shared = 2560 - dtadapt*BLOCK_SIZE_FORCES*4;
	#endif

	// TODO why are the booleans here as parameters?
	// FIXME forces_param should be based on simflags too
	forces_params<kerneltype, boundarytype, visctype,
		simflags & ENABLE_DTADAPT,
		simflags & ENABLE_XSPH,
		simflags & ENABLE_INLET_OUTLET> params(
			forces, contupd, rbforces, rbtorques,
			pos, particleHash, cellStart, neibsList, fromParticle, toParticle,
			deltap, slength, influenceradius,
			simflags & ENABLE_DEM,
			cfl, cflTVisc, cflOffset,
			xsph,
			newGGam, vertPos, epsilon,
			simflags & ENABLE_MOVING_BODIES,
			IOwaterdepth, simflags & ENABLE_WATER_DEPTH,
			keps_dkde, turbvisc);

	// FIXME forcesDevice should use simflags, not the neverending pile of booleans
	cuforces::forcesDevice<kerneltype, sph_formulation, boundarytype, visctype,
		simflags & ENABLE_DTADAPT, simflags & ENABLE_XSPH, simflags & ENABLE_INLET_OUTLET>
			<<< numBlocks, numThreads, dummy_shared >>>(params);

	return numBlocks;
}

FORCES_RET(void)
setDEM(const float *hDem, int width, int height)
{
	// Allocating, reading and copying DEM
	unsigned int size = width*height*sizeof(float);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL( hipMallocArray( &dDem, &channelDesc, width, height ));
	CUDA_SAFE_CALL( hipMemcpyToArray( dDem, 0, 0, hDem, size, hipMemcpyHostToDevice));

	demTex.addressMode[0] = hipAddressModeClamp;
	demTex.addressMode[1] = hipAddressModeClamp;
	demTex.filterMode = hipFilterModeLinear;
	demTex.normalized = false;

	CUDA_SAFE_CALL( hipBindTextureToArray(demTex, dDem, channelDesc));
}

FORCES_RET(void)
unsetDEM()
{
	CUDA_SAFE_CALL(hipFreeArray(dDem));
}

FORCES_RET(uint)
round_particles(uint numparts)
{
	return (numparts/BLOCK_SIZE_FORCES)*BLOCK_SIZE_FORCES;
}

FORCES_RET(void)
reduceRbForces(	float4	*forces,
				float4	*torques,
				uint	*rbnum,
				uint	*lastindex,
				float3	*totalforce,
				float3	*totaltorque,
				uint	numbodies,
				uint	numBodiesParticles)
{
	thrust::device_ptr<float4> forces_devptr = thrust::device_pointer_cast(forces);
	thrust::device_ptr<float4> torques_devptr = thrust::device_pointer_cast(torques);
	thrust::device_ptr<uint> rbnum_devptr = thrust::device_pointer_cast(rbnum);
	thrust::equal_to<uint> binary_pred;
	thrust::plus<float4> binary_op;

	// For the segmented scan, we use rbnum (number of object per object particle) as key (first and second parameters
	// of inclusive_scan_by_key are the begin and the end of the array of keys); forces or torques as input and output
	// the scan is in place); equal_to as data-key operator and plus as scan operator. The sums are in the last position
	// of each segment (thus we retrieve them by using lastindex values).

	thrust::inclusive_scan_by_key(rbnum_devptr, rbnum_devptr + numBodiesParticles,
				forces_devptr, forces_devptr, binary_pred, binary_op);
	thrust::inclusive_scan_by_key(rbnum_devptr, rbnum_devptr + numBodiesParticles,
				torques_devptr, torques_devptr, binary_pred, binary_op);

	for (uint i = 0; i < numbodies; i++) {
		float4 temp;
		void * ddata = (void *) (forces + lastindex[i]);
		CUDA_SAFE_CALL(hipMemcpy((void *) &temp, ddata, sizeof(float4), hipMemcpyDeviceToHost));
		totalforce[i] = as_float3(temp);

		ddata = (void *) (torques + lastindex[i]);
		CUDA_SAFE_CALL(hipMemcpy((void *) &temp, ddata, sizeof(float4), hipMemcpyDeviceToHost));
		totaltorque[i] = as_float3(temp);
		}
}

// The instances that we want to actually instantiate are defined
// in a programmatically-generated file:

#ifndef FORCES_INSTANCE_FILE
#error "No instance file defined for forces!"
#else
#include STR(FORCES_INSTANCE_FILE)
#endif

/// CUDAViscEngine should be moved elsewhere

template<ViscosityType visctype,
	KernelType kerneltype,
	BoundaryType boundarytype>
void
CUDAViscEngineHelper<visctype, kerneltype, boundarytype>::process(
			float2	*tau[],
	const	float4	*pos,
	const	float4	*vel,
	const	particleinfo	*info,
	const	hashKey	*particleHash,
	const	uint	*cellStart,
	const	neibdata*neibsList,
			uint	numParticles,
			uint	particleRangeEnd,
			float	slength,
			float	influenceradius)
{ /* default, does nothing */ }

/// Partial specialization for SPSVISC. Partial specializations
/// redefine the whole helper struct, not just the method, since
/// C++ does not allow partial function/method template specializations
/// (which is why we have the Helper struct in the first place
template<KernelType kerneltype,
	BoundaryType boundarytype>
struct CUDAViscEngineHelper<SPSVISC, kerneltype, boundarytype>
{
	static void
	process(float2	*tau[],
	const	float4	*pos,
	const	float4	*vel,
	const	particleinfo	*info,
	const	hashKey	*particleHash,
	const	uint	*cellStart,
	const	neibdata*neibsList,
			uint	numParticles,
			uint	particleRangeEnd,
			float	slength,
			float	influenceradius)
{
	int dummy_shared = 0;
	// bind textures to read all particles, not only internal ones
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	uint numThreads = min(BLOCK_SIZE_SPS, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	cuforces::SPSstressMatrixDevice<kerneltype, boundarytype == DYN_BOUNDARY>
		<<<numBlocks, numThreads, dummy_shared>>>
		(pos, tau[0], tau[1], tau[2], particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("SPS kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, tau0Tex, tau[0], numParticles*sizeof(float2)));
	CUDA_SAFE_CALL(hipBindTexture(0, tau1Tex, tau[1], numParticles*sizeof(float2)));
	CUDA_SAFE_CALL(hipBindTexture(0, tau2Tex, tau[2], numParticles*sizeof(float2)));
}
};

// The instances that we want to actually instantiate are defined
// in a programmatically-generated file:

#ifndef VISC_INSTANCE_FILE
#error "No instance file defined for viscosities!"
#else
#include STR(VISC_INSTANCE_FILE)
#endif

/// Other methods TODO will need to move elsewhere

template<KernelType kerneltype, BoundaryType boundarytype>
struct CUDAFilterEngineHelper<SHEPARD_FILTER, kerneltype, boundarytype>
{
	static void process(
		const	float4	*pos,
		const	float4	*oldVel,
				float4	*newVel,
		const	particleinfo	*info,
		const	hashKey	*particleHash,
		const	uint	*cellStart,
		const	neibdata*neibsList,
				uint	numParticles,
				uint	particleRangeEnd,
				float	slength,
				float	influenceradius)
{
	int dummy_shared = 0;
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_SHEPARD, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	#if (__COMPUTE__ >= 20)
	dummy_shared = 2560;
	#endif

	cuforces::shepardDevice<kerneltype, boundarytype><<< numBlocks, numThreads, dummy_shared >>>
		(pos, newVel, particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Shepard kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}
};

template<KernelType kerneltype, BoundaryType boundarytype>
struct CUDAFilterEngineHelper<MLS_FILTER, kerneltype, boundarytype>
{
	static void process(
		const	float4	*pos,
		const	float4	*oldVel,
				float4	*newVel,
		const	particleinfo	*info,
		const	hashKey	*particleHash,
		const	uint	*cellStart,
		const	neibdata*neibsList,
				uint	numParticles,
				uint	particleRangeEnd,
				float	slength,
				float	influenceradius)
{
	int dummy_shared = 0;
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_MLS, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	#if (__COMPUTE__ >= 20)
	dummy_shared = 2560;
	#endif

	cuforces::MlsDevice<kerneltype, boundarytype><<< numBlocks, numThreads, dummy_shared >>>
		(pos, newVel, particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Mls kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}
};

// The instances that we want to actually instantiate are defined
// in a programmatically-generated file:

#ifndef FILTERS_INSTANCE_FILE
#error "No instance file defined for filters!"
#else
#include STR(FILTERS_INSTANCE_FILE)
#endif


template<KernelType kerneltype>
void
CUDAPostProcessEngine<kerneltype>::
vorticity(const	float4*		pos,
		const	float4*		vel,
			float3*		vort,
		const	particleinfo	*info,
		const	hashKey*		particleHash,
		const	uint*		cellStart,
		const	neibdata*	neibsList,
			uint		numParticles,
			uint		particleRangeEnd,
			float		slength,
			float		influenceradius)
{
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_CALCVORT, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	cuforces::calcVortDevice<kerneltype><<< numBlocks, numThreads >>>
		(pos, vort, particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Vorticity kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

//Testpoints
template<KernelType kerneltype>
void
CUDAPostProcessEngine<kerneltype>::
testpoints( const float4*	pos,
			float4*			newVel,
			float*			newTke,
			float*			newEpsilon,
			const particleinfo	*info,
			const hashKey*		particleHash,
			const uint*			cellStart,
			const neibdata*		neibsList,
			uint			numParticles,
			uint			particleRangeEnd,
			float			slength,
			float			influenceradius)
{
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_CALCTEST, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, newVel, numParticles*sizeof(float4)));
	if (newTke)
		CUDA_SAFE_CALL(hipBindTexture(0, keps_kTex, newTke, numParticles*sizeof(float)));
	if (newEpsilon)
		CUDA_SAFE_CALL(hipBindTexture(0, keps_eTex, newEpsilon, numParticles*sizeof(float)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	cuforces::calcTestpointsVelocityDevice<kerneltype><<< numBlocks, numThreads >>>
		(pos, newVel, newTke, newEpsilon, particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("test kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	if (newTke)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_kTex));
	if (newEpsilon)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_eTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

// Free surface detection
template<KernelType kerneltype>
void
CUDAPostProcessEngine<kerneltype>::
surfaceparticle(const	float4*		pos,
				const	float4*     vel,
					float4*		normals,
				const	particleinfo	*info,
					particleinfo	*newInfo,
				const	hashKey*		particleHash,
				const	uint*		cellStart,
				const	neibdata*	neibsList,
					uint		numParticles,
					uint		particleRangeEnd,
					float		slength,
					float		influenceradius,
					bool		savenormals)
{
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_CALCTEST, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	if (savenormals) {
		cuforces::calcSurfaceparticleDevice<kerneltype, true><<< numBlocks, numThreads >>>
			(pos, normals, newInfo, particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);
	} else {
		cuforces::calcSurfaceparticleDevice<kerneltype, false><<< numBlocks, numThreads >>>
			(pos, normals, newInfo, particleHash, cellStart, neibsList, particleRangeEnd, slength, influenceradius);
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("surface kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

template<KernelType kerneltype>
void
CUDAPostProcessEngine<kerneltype>::
calcPrivate(const	float4*			pos,
			const	float4*			vel,
			const	particleinfo*	info,
					float*			priv,
			const	hashKey*		particleHash,
			const	uint*			cellStart,
			const	neibdata*		neibsList,
					float			slength,
					float			inflRadius,
					uint			numParticles,
					uint			particleRangeEnd)
{
	uint numThreads = min(BLOCK_SIZE_FORCES, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));

	//execute kernel
	cuforces::calcPrivateDevice<<<numBlocks, numThreads>>>
		(	pos,
			priv,
			particleHash,
			cellStart,
			neibsList,
			slength,
			inflRadius,
			numParticles);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("UpdatePositions kernel execution failed");
}



/* Reductions */
void set_reduction_params(void* buffer, size_t blocks,
		size_t blocksize_max, size_t shmem_max)
{
	reduce_blocks = blocks;
	// in the second step of a reduction, a single block is launched, whose size
	// should be the smallest power of two that covers the number of blocks used
	// in the previous reduction run
	reduce_bs2 = 32;
	while (reduce_bs2 < blocks)
		reduce_bs2<<=1;

	reduce_blocksize_max = blocksize_max;
	reduce_shmem_max = shmem_max;
	reduce_buffer = buffer;
}

void unset_reduction_params()
{
	CUDA_SAFE_CALL(hipFree(reduce_buffer));
	reduce_buffer = NULL;
}

// Compute system energy
void calc_energy(
		float4			*output,
	const	float4		*pos,
	const	float4		*vel,
	const	particleinfo	*pinfo,
	const	hashKey		*particleHash,
		uint			numParticles,
		uint			numFluids)
{
	// shmem needed by a single thread
	size_t shmem_thread = numFluids*sizeof(float4)*2;
	size_t blocksize_max = reduce_shmem_max/shmem_thread;
	if (blocksize_max > reduce_blocksize_max)
		blocksize_max = reduce_blocksize_max;

	size_t blocksize = 32;
	while (blocksize*2 < blocksize_max)
		blocksize<<=1;

	cuforces::calcEnergiesDevice<<<reduce_blocks, blocksize, blocksize*shmem_thread>>>(
			pos, vel, pinfo, particleHash, numParticles, numFluids, (float4*)reduce_buffer);
	CUT_CHECK_ERROR("System energy stage 1 failed");

	cuforces::calcEnergies2Device<<<1, reduce_bs2, reduce_bs2*shmem_thread>>>(
			(float4*)reduce_buffer, reduce_blocks, numFluids);
	CUT_CHECK_ERROR("System energy stage 2 failed");
	CUDA_SAFE_CALL(hipMemcpy(output, reduce_buffer, numFluids*sizeof(float4), hipMemcpyDeviceToHost));
}

#define COND_RET(ret_type) \
template< \
	KernelType kerneltype, \
	ViscosityType visctype, \
	BoundaryType boundarytype, \
	flag_t simflags \
> \
ret_type \
CUDABoundaryConditionsEngine<kerneltype, visctype, boundarytype, simflags>::


COND_RET(void)
disableOutgoingParts(		float4*			pos,
							vertexinfo*		vertices,
					const	particleinfo*	info,
					const	uint			numParticles,
					const	uint			particleRangeEnd)
{
	uint numThreads = min(BLOCK_SIZE_FORCES, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	//execute kernel
	cuforces::disableOutgoingPartsDevice<<<numBlocks, numThreads>>>
		(	pos,
			vertices,
			numParticles);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("UpdatePositions kernel execution failed");
}

COND_RET(void)
saSegmentBoundaryConditions(
			float4*			oldPos,
			float4*			oldVel,
			float*			oldTKE,
			float*			oldEps,
			float4*			oldEulerVel,
			float4*			oldGGam,
			vertexinfo*		vertices,
	const	uint*			vertIDToIndex,
	const	float2	* const vertPos[],
	const	float4*			boundelement,
	const	particleinfo*	info,
	const	hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	uint			particleRangeEnd,
	const	float			deltap,
	const	float			slength,
	const	float			influenceradius,
	const	bool			initStep)
{
	uint numThreads = min(BLOCK_SIZE_FORCES, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	cuforces::saSegmentBoundaryConditions<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(oldPos, oldVel, oldTKE, oldEps, oldEulerVel, oldGGam, vertices, vertIDToIndex, vertPos[0], vertPos[1], vertPos[2], particleHash, cellStart, neibsList, particleRangeEnd, deltap, slength, influenceradius, initStep, simflags & ENABLE_INLET_OUTLET);

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("saSegmentBoundaryConditions kernel execution failed");
}

COND_RET(void)
saVertexBoundaryConditions(
			float4*			oldPos,
			float4*			oldVel,
			float*			oldTKE,
			float*			oldEps,
			float4*			oldGGam,
			float4*			oldEulerVel,
			float4*			forces,
			float2*			contupd,
	const	float4*			boundelement,
			vertexinfo*		vertices,
	const	uint*			vertIDToIndex,
			particleinfo*	info,
			hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
			uint*			newNumParticles,
	const	uint			particleRangeEnd,
	const	float			dt,
	const	int				step,
	const	float			deltap,
	const	float			slength,
	const	float			influenceradius,
	const	uint&			newIDsOffset,
	const	bool			initStep)
{
	int dummy_shared = 0;

	uint numThreads = min(BLOCK_SIZE_SHEPARD, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_newIDsOffset), &newIDsOffset, sizeof(uint)));

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	// execute the kernel
	cuforces::saVertexBoundaryConditions<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(oldPos, oldVel, oldTKE, oldEps, oldGGam, oldEulerVel, forces, contupd, vertices, vertIDToIndex, info, particleHash, cellStart, neibsList, particleRangeEnd, newNumParticles, dt, step, deltap, slength, influenceradius, initStep);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("saVertexBoundaryConditions kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}

COND_RET(void)
downloadIOwaterdepth(
			uint*	h_IOwaterdepth,
	const	uint*	d_IOwaterdepth,
	const	uint	numObjects)
{
	CUDA_SAFE_CALL(hipMemcpy(h_IOwaterdepth, d_IOwaterdepth, numObjects*sizeof(int), hipMemcpyDeviceToHost));
}

COND_RET(void)
uploadIOwaterdepth(
	const	uint*	h_IOwaterdepth,
			uint*	d_IOwaterdepth,
	const	uint	numObjects)
{
	CUDA_SAFE_CALL(hipMemcpy(d_IOwaterdepth, h_IOwaterdepth, numObjects*sizeof(int), hipMemcpyHostToDevice));
}

COND_RET(void)
saIdentifyCornerVertices(
	const	float4*			oldPos,
	const	float4*			boundelement,
			particleinfo*	info,
	const	hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	uint			particleRangeEnd,
	const	float			deltap,
	const	float			eps)
{
	int dummy_shared = 0;

	uint numThreads = min(BLOCK_SIZE_SHEPARD, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	// execute the kernel
	cuforces::saIdentifyCornerVertices<<< numBlocks, numThreads, dummy_shared >>> (
		oldPos,
		info,
		particleHash,
		cellStart,
		neibsList,
		numParticles,
		deltap,
		eps);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("saIdentifyCornerVertices kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}

COND_RET(void)
saFindClosestVertex(
	const	float4*			oldPos,
			particleinfo*	info,
			vertexinfo*		vertices,
	const	uint*			vertIDToIndex,
	const	hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	uint			particleRangeEnd)
{
	int dummy_shared = 0;

	uint numThreads = min(BLOCK_SIZE_SHEPARD, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	cuforces::saFindClosestVertex<<< numBlocks, numThreads, dummy_shared >>>(
				oldPos,
				info,
				vertices,
				vertIDToIndex,
				particleHash,
				cellStart,
				neibsList,
				numParticles);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("saFindClosestVertex kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

// The instances that we want to actually instantiate are defined
// in a programmatically-generated file:

#ifndef BOUND_INSTANCE_FILE
#error "No instance file defined for boundary conditions!"
#else
#include STR(BOUND_INSTANCE_FILE)
#endif

