#include "hip/hip_runtime.h"
/*  Copyright 2015 Giuseppe Bilotta, Alexis Herault, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/* Device functions and constants pertaining open boundaries */

#ifndef _BOUNDS_KERNEL_
#define _BOUNDS_KERNEL_

#include "particledefine.h"

/*!
 * \namespace cubounds
 * \brief Contains all device functions/kernels/constants related to open boundaries and domain geometry.
 *
 * The namespace contains the device side of boundary handling
 *	- domain size, origin and cell grid properties and related functions
 *	- open boundaries properties and related functions
 */
namespace cubounds {

// Grid data
#include "cellgrid.cuh"
#include "planes.h"

/// \name Device constants
/// @{

texture<float, 2, hipReadModeElementType> demTex;	// DEM

/* DEM constants */
// TODO switch to float2s
__constant__ float	d_ewres;		///< east-west resolution (x)
__constant__ float	d_nsres;		///< north-south resolution (y)
__constant__ float	d_demdx;		///< ∆x increment of particle position for normal computation
__constant__ float	d_demdy;		///< ∆y increment of particle position for normal computation
__constant__ float	d_demdxdy;		///< ∆x*∆y
__constant__ float	d_demzmin;		///< minimum distance from DEM for normal computation

/* Constants for geometrical planar boundaries */
__constant__ uint	d_numplanes;
__constant__ plane_t d_plane[MAX_PLANES];

/// Number of open boundaries (both inlets and outlets)
__constant__ uint d_numOpenBoundaries;

// host-computed id offset used for id generation
__constant__ uint	d_newIDsOffset;

/// @}

/** \name Device functions
 *  @{ */

//! Given a point in grid + pos coordinates, and a plane defined by
//! a normal and a point (in grid + pos coordinates) on the plane,
//! returns the (signed) distance of the point to the plane.
//! NOTE: 2*signedDistance*plane.normal gives the distance vector
//! to the reflection of the point across the plane
__device__ __forceinline__ float
signedPlaneDistance(
	const int3&		gridPos,
	const float3&	pos,
	const plane_t&	plane)
{
	// Relative position of the point to the reference point of the plane
	const float3 relPos = globalDistance(gridPos, pos,
		plane.gridPos, plane.pos);

	return dot(relPos, plane.normal);
}

//! \see signedPlaneDistance, but returns the (unsigned) distance
__device__ __forceinline__ float
PlaneDistance(	const int3&		gridPos,
				const float3&	pos,
				const plane_t&	plane)
{
	return abs(signedPlaneDistance(gridPos, pos, plane));
}

/**! Convert an xy grid + local position into a DEM cell position
 * This is done assuming that the worldOrigin is at DEM coordinates (0, 0).
 * NOTE: the function accepts anything as grid and local pos,
 * but GridPosType should be an int2 or int3 and LocalPosType should be
 * a float2 or float3.
 * TODO use type traits to enforce this.
 */
template<typename GridPosType, typename LocalPosType>
__device__ __forceinline__ float2
DemPos(GridPosType const& gridPos, LocalPosType const& pos)
{
	// note that we separate the grid conversion part from the pos conversion part,
	// for improved accuracy. The final 0.5f is because texture values are assumed to be
	// at the center of the DEM cell.
	return make_float2(
		(gridPos.x + 0.5f)*(d_cellSize.x/d_ewres) + pos.x/d_ewres + 0.5f,
		(gridPos.y + 0.5f)*(d_cellSize.y/d_nsres) + pos.y/d_nsres + 0.5f);
}

/**! Interpolate DEM texref for a point at DEM cell pos demPos,
  plus an optional multiple of (∆x, ∆y).
  NOTE: the returned z coordinate is GLOBAL, not LOCAL!
  TODO for improved homogeneous accuracy, maybe have a texture for grid cells and a
  texture for local z coordinates?
 */
__device__ __forceinline__ float
DemInterpol(const texture<float, 2, hipReadModeElementType> texref,
	const float2& demPos, int dx=0, int dy=0)
{
	return tex2D(texref, demPos.x + dx*d_demdx/d_ewres, demPos.y + dy*d_demdy/d_nsres);
}

/*!
 * Create a new particle, cloning an existing particle
 * This returns the index of the generated particle, initializing new_info
 * for a FLUID particle of the same fluid as the generator, no associated
 * object or inlet, and a new id generated in a way which is multi-GPU
 * compatible.
 *
 * All other particle properties (position, velocity, etc) should be
 * set by the caller.
 */
__device__ __forceinline__
uint
createNewFluidParticle(
	/// [out] particle info of the generated particle
			particleinfo	&new_info,
	/// [in] particle info of the generator particle
	const	particleinfo	&info,
	/// [in] number of particles at the start of the current timestep
	const	uint			numParticles,
	/// [in] number of devices
	const	uint			numDevices,
	/// [in,out] number of particles including all the ones already created in this timestep
			uint			*newNumParticles)
{
	const uint new_index = atomicAdd(newNumParticles, 1);
	// number of new particles that were created on this device in this
	// time step
	const uint newNumPartsOnDevice = new_index + 1 - numParticles;
	// the i-th device can only allocate an id that satisfies id%n == i, where
	// n = number of total devices
	const uint new_id = newNumPartsOnDevice*numDevices + d_newIDsOffset;

	new_info = make_particleinfo_by_ids(
		PT_FLUID,
		fluid_num(info), 0, // copy the fluid number, not the object number
		new_id);
	return new_index;
}

/** @} */

} // namespace cubounds

#endif
