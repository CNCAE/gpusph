#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdexcept>

#include "define_buffers.h"
#include "engine_integration.h"
#include "utils.h"

#include "euler_kernel.cu"

#define BLOCK_SIZE_INTEGRATE	256

template<SPHFormulation sph_formulation, BoundaryType boundarytype, bool xsphcorr>
class CUDAPredCorrEngine : public AbstractIntegrationEngine
{

void
setconstants(const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams->epsxsph, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_gridSize), &gridSize, sizeof(uint3)));
}

void
getconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
}

void
setrbcg(const int3* cgGridPos, const float3* cgPos, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgGridPos), cgGridPos, numbodies*sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgPos), cgPos, numbodies*sizeof(float3)));
}

void
setrbtrans(const float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}

void
setrblinearvel(const float3* linearvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rblinearvel), linearvel, numbodies*sizeof(float3)));
}

void
setrbangularvel(const float3* angularvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbangularvel), angularvel, numbodies*sizeof(float3)));
}

void
setrbsteprot(const float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}

void
basicstep(
		MultiBufferList::const_iterator bufread,
		MultiBufferList::iterator bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	float	dt2,
		const	int		step,
		const	float	t)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float4  *oldPos = bufread->getData<BUFFER_POS>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const float4  *oldVel = bufread->getData<BUFFER_VEL>();
	const float4  *oldVol = bufread->getData<BUFFER_VOLUME>();
	const float4 *oldEulerVel = bufread->getData<BUFFER_EULERVEL>();
	const float4 *oldgGam = bufread->getData<BUFFER_GRADGAMMA>();
	const float *oldTKE = bufread->getData<BUFFER_TKE>();
	const float *oldEps = bufread->getData<BUFFER_EPSILON>();
	const particleinfo *info = bufread->getData<BUFFER_INFO>();

	const float4 *forces = bufread->getData<BUFFER_FORCES>();
	const float2 *contupd = bufread->getData<BUFFER_CONTUPD>();
	const float3 *keps_dkde = bufread->getData<BUFFER_DKDE>();
	const float4 *xsph = bufread->getData<BUFFER_XSPH>();

	float4 *newPos = bufwrite->getData<BUFFER_POS>();
	float4 *newVel = bufwrite->getData<BUFFER_VEL>();
	float4 *newVol = bufwrite->getData<BUFFER_VOLUME>();
	float4 *newEulerVel = bufwrite->getData<BUFFER_EULERVEL>();
	float4 *newgGam = bufwrite->getData<BUFFER_GRADGAMMA>();
	float *newTKE = bufwrite->getData<BUFFER_TKE>();
	float *newEps = bufwrite->getData<BUFFER_EPSILON>();
	// boundary elements are updated in-place; only used for rotation in the second step
	float4 *newBoundElement = bufwrite->getData<BUFFER_BOUNDELEMENTS>();

#define ARGS oldPos, particleHash, oldVel, oldVol, oldEulerVel, oldgGam, oldTKE, oldEps, \
	info, forces, contupd, keps_dkde, xsph, newPos, newVel, newVol, newEulerVel, newgGam, newTKE, newEps, newBoundElement, particleRangeEnd, dt, dt2, t

	if (step == 1) {
		cueuler::eulerDevice<1, xsphcorr, sph_formulation, boundarytype><<< numBlocks, numThreads >>>(ARGS);
	} else if (step == 2) {
		cueuler::eulerDevice<2, xsphcorr, sph_formulation, boundarytype><<< numBlocks, numThreads >>>(ARGS);
	} else {
		throw std::invalid_argument("unsupported predcorr timestep");
	}

#undef ARGS

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Euler kernel execution failed");
}

};

