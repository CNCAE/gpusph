#include "hip/hip_runtime.h"
#ifndef PROBLEM_BC_CU
#define PROBLEM_BC_CU

#include <math.h>
#include <string>
#include <iostream>

#include "InputProblem.h"
#include "GlobalData.h"
#include "textures.cuh"
#include "utils.h"
#include "Problem.h"

namespace cuInputProblem
{
#include "cellgrid.h"
// Core SPH functions
#include "sph_core_utils.cuh"

__device__
void
InputProblem_imposeBoundaryCondition(
	const	particleinfo	info,
	const	float3			absPos,
			float			waterdepth,
	const	float			t,
			float4&			vel,
			float4&			eulerVel,
			float&			tke,
			float&			eps)
{
	vel = make_float4(0.0f);
	eulerVel = make_float4(0.0f);
	tke = 0.0f;
	eps = 0.0f;

	if (IO_BOUNDARY(info)) {
		if (VEL_IO(info)) {
#if SPECIFIC_PROBLEM == SmallChannelFlowIO
			// third order approximation to the flow in a rectangular duct
			const float y2 = absPos.y*absPos.y;
			const float z2 = absPos.z*absPos.z;
			const float y4 = y2*y2;
			const float z4 = z2*z2;
			const float y6 = y2*y4;
			const float z6 = z2*z4;
			const float y8 = y4*y4;
			const float z8 = z4*z4;
			eulerVel.x = (461.0f+y8-392.0f*z2-28.0f*y6*z2-70.0f*z4+z8+70.0f*y4*(z4-1.0f)-28.0f*y2*(14.0f-15.0f*z2+z6))/461.0f;
			eulerVel.x = fmax(eulerVel.x, 0.0f);
#elif SPECIFIC_PROBLEM == IOWithoutWalls
			eulerVel.x = 1.0f;
#elif SPECIFIC_PROBLEM == SmallChannelFlowIOPer
			eulerVel.x = 1.0f-absPos.z*absPos.z;
#elif SPECIFIC_PROBLEM == SmallChannelFlowIOKeps
			// the 0.025 is deltap*0.5 = 0.05*0.5
			eulerVel.x = log(fmax(1.0f-fabs(absPos.z), 0.025f)/0.0015625f)/0.41f+5.2f;
#else
			eulerVel.x = 0.0f;
#endif
		}
		else {
#if SPECIFIC_PROBLEM == LaPalisseSmallTest
			if (INFLOW(info))
				waterdepth = 0.255; // set inflow waterdepth to 0.21 (with respect to world_origin)
			const float localdepth = fmax(waterdepth - absPos.z, 0.0f);
			const float pressure = 9.81e3f*localdepth;
			eulerVel.w = RHO(pressure, PART_FLUID_NUM(info));
#elif SPECIFIC_PROBLEM == IOWithoutWalls
			if (INFLOW(info))
				eulerVel.w = 1002.0f;
			else
				eulerVel.w = 1002.0f;
				//eulerVel.w = 1000.0f;
#else
			eulerVel.w = 1000.0f;
#endif
		}

		// impose tangential velocity
		if (INFLOW(info)) {
			eulerVel.y = 0.0f;
			eulerVel.z = 0.0f;
#if SPECIFIC_PROBLEM == SmallChannelFlowIOKeps
			// k and eps based on Versteeg & Malalasekera (2001)
			// turbulent intensity (between 1% and 6%)
			const float Ti = 0.01f;
			// in case of a pressure inlet eulerVel.x = 0 so we set u to 1 to multiply it later once
			// we know the correct velocity
			const float u = eulerVel.x > 1e-6f ? eulerVel.x : 1.0f;
			tke = 3.0f/2.0f*(u*Ti)*(u*Ti);
			tke = 3.33333f;
			// length scale of the flow
			const float L = 1.0f;
			// constant is C_\mu^(3/4)/0.07*sqrt(3/2)
			// formula is epsilon = C_\mu^(3/4) k^(3/2)/(0.07 L)
			eps = 2.874944542f*tke*u*Ti/L;
			eps = 1.0f/0.41f/fmax(1.0f-fabs(absPos.z),0.025f);
#endif
		}
	}

	// forced moving boundaries
	else if (MOVING(info) && !FLOATING(info)) {
		;// placeholder if only
	}
}

__global__ void
InputProblem_imposeBoundaryConditionDevice(
			float4*		newVel,
			float4*		newEulerVel,
			float*		newTke,
			float*		newEpsilon,
	const	float4*		oldPos,
	const	uint*		IOwaterdepth,
	const	float		t,
	const	uint		numParticles,
	const	hashKey*	particleHash)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	float4 vel = make_float4(0.0f);			// imposed velocity for moving objects
	float4 eulerVel = make_float4(0.0f);	// imposed velocity/pressure for open boundaries
	float tke = 0.0f;						// imposed turbulent kinetic energy for open boundaries
	float eps = 0.0f;						// imposed turb. diffusivity for open boundaries

	if(index < numParticles) {
		const particleinfo info = tex1Dfetch(infoTex, index);
		// open boundaries and forced moving objects
		if (VERTEX(info) && IO_BOUNDARY(info)) {
			const float3 absPos = d_worldOrigin + as_float3(oldPos[index])
									+ calcGridPosFromParticleHash(particleHash[index])*d_cellSize
									+ 0.5f*d_cellSize;
			// when pressure outlets require the water depth compute it from the IOwaterdepth integer
			float waterdepth = 0.0f;
			if (!VEL_IO(info) && !INFLOW(info) && IOwaterdepth) {
				waterdepth = ((float)IOwaterdepth[object(info)-1])/((float)UINT_MAX); // now between 0 and 1
				waterdepth *= d_cellSize.z*d_gridSize.z; // now between 0 and world size
				waterdepth += d_worldOrigin.z; // now absolute z position
			}
			// this now calls the virtual function that is problem specific
			InputProblem_imposeBoundaryCondition(info, absPos, waterdepth, t, vel, eulerVel, tke, eps);
			// copy values to arrays
			newVel[index] = vel;
			newEulerVel[index] = eulerVel;
			if(newTke)
				newTke[index] = tke;
			if(newEpsilon)
				newEpsilon[index] = eps;
		}
		// all other vertex particles had their eulerVel set in euler already
	}
}

} // end of cuInputProblem namespace

extern "C"
{

void
InputProblem::setboundconstants(
	const	PhysParams	*physparams,
	float3	const&		worldOrigin,
	uint3	const&		gridSize,
	float3	const&		cellSize)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_gridSize), &gridSize, sizeof(uint3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_rho0), &physparams->rho0, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_bcoeff), &physparams->bcoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_gammacoeff), &physparams->gammacoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuInputProblem::d_sscoeff), &physparams->sscoeff, MAX_FLUID_TYPES*sizeof(float)));

}

}

void
InputProblem::imposeBoundaryConditionHost(
			float4*			newVel,
			float4*			newEulerVel,
			float*			newTke,
			float*			newEpsilon,
	const	particleinfo*	info,
	const	float4*			oldPos,
			uint			*IOwaterdepth,
	const	float			t,
	const	uint			numParticles,
	const	uint			numObjects,
	const	uint			particleRangeEnd,
	const	hashKey*		particleHash)
{
	uint numThreads = min(BLOCK_SIZE_IOBOUND, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	cuInputProblem::InputProblem_imposeBoundaryConditionDevice<<< numBlocks, numThreads, dummy_shared >>>
		(newVel, newEulerVel, newTke, newEpsilon, oldPos, IOwaterdepth, t, numParticles, particleHash);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// reset waterdepth calculation
	if (IOwaterdepth) {
		uint h_IOwaterdepth[numObjects];
		for (uint i=0; i<numObjects; i++)
			h_IOwaterdepth[i] = 0;
		CUDA_SAFE_CALL(hipMemcpy(IOwaterdepth, h_IOwaterdepth, numObjects*sizeof(int), hipMemcpyHostToDevice));
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("imposeBoundaryCondition kernel execution failed");
}

#endif
