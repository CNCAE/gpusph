#include "hip/hip_runtime.h"
/*
 * Device code.
 */

#ifndef _EULER_KERNEL_
#define _EULER_KERNEL_

#include "particledefine.h"
#include "textures.cuh"

__constant__ float	d_epsxsph;
__constant__ float3	d_maxlimit;
__constant__ float3	d_minlimit;
__constant__ float3 d_dispvect3;
__constant__ float4	d_mbdata[MAXMOVINGBOUND];


/*
 * Device code.
 */

#undef XSPH_KERNEL
#define EULER_KERNEL_NAME eulerDevice
#include "euler_kernel.def"
#undef EULER_KERNEL_NAME

#define XSPH_KERNEL 1
#define EULER_KERNEL_NAME eulerXsphDevice
#include "euler_kernel.def"
#undef XPSH_KERNEL
#undef EULER_KERNEL_NAME

#endif
