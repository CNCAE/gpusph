#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>

#include "euler.cuh"
#include "euler_kernel.cu"

#include "utils.h"

extern "C"
{
void
seteulerconstants(const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams->epsxsph, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_gridSize), &gridSize, sizeof(uint3)));
}


void
geteulerconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
}


void
setmbdata(const float4* MbData, uint size)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_mbdata), MbData, size));
}


void
seteulerrbcg(const float3* cg, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcg), cg, numbodies*sizeof(float3)));
}


void
seteulerrbtrans(const float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}


void
seteulerrblinearvel(const float3* linearvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rblinearvel), linearvel, numbodies*sizeof(float3)));
	//printf("Upload linear vel: %e %e %e\n", linearvel[0].x, linearvel[0].y, linearvel[0].z);
}


void
seteulerrbangularvel(const float3* angularvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbangularvel), angularvel, numbodies*sizeof(float3)));
	//printf("Upload angular vel: %e %e %e\n", angularvel[0].x, angularvel[0].y, angularvel[0].z);
}


void
seteulerrbsteprot(const float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}


void
euler(	const float4*		oldPos,
		const hashKey*		particleHash,
		const float4*		oldVel,
		const float*		oldTKE,
		const float*		oldEps,
		const particleinfo* info,
		const float4*		forces,
		float2*				keps_dkde,
		const float4*		xsph,
		float4*				newPos,
		float4*				newVel,
		float*				newTKE,
		float*				newEps,
		const uint			numParticles,
		const uint			particleRangeEnd,
		const float			dt,
		const float			dt2,
		const int			step,
		const float			t,
		const bool			xsphcorr,
		BoundaryType		boundarytype)
{
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_INTEGRATE, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

#define ARGS oldPos, particleHash, oldVel, oldTKE, oldEps, \
	info, forces, keps_dkde, xsph, newPos, newVel, newTKE, newEps, particleRangeEnd, dt, dt2, t

	// execute the kernel
	if (boundarytype == DYN_BOUNDARY) {
		if (step == 1) {
			if (xsphcorr)
				cueuler::eulerDevice<1, true, true><<< numBlocks, numThreads >>>(ARGS);
			else
				cueuler::eulerDevice<1, false, true><<< numBlocks, numThreads >>>(ARGS);
		} else if (step == 2) {
			if (xsphcorr)
				cueuler::eulerDevice<2, true, true><<< numBlocks, numThreads >>>(ARGS);
			else
				cueuler::eulerDevice<2, false, true><<< numBlocks, numThreads >>>(ARGS);
		}
	} else {
		if (step == 1) {
			if (xsphcorr)
				cueuler::eulerDevice<1, true, false><<< numBlocks, numThreads >>>(ARGS);
			else
				cueuler::eulerDevice<1, false, false><<< numBlocks, numThreads >>>(ARGS);
		} else if (step == 2) {
			if (xsphcorr)
				cueuler::eulerDevice<2, true, false><<< numBlocks, numThreads >>>(ARGS);
			else
				cueuler::eulerDevice<2, false, false><<< numBlocks, numThreads >>>(ARGS);
		}
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Euler kernel execution failed");
}
}
