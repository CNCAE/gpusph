#include "hip/hip_runtime.h"
/*  Copyright 2011 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

	Istituto de Nazionale di Geofisica e Vulcanologia
          Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>

#include "euler.cuh"
#include "euler_kernel.cu"

extern "C"
{
void
seteulerconstants(const PhysParams & physparams)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams.epsxsph, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_dispvect), &physparams.dispvect, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_minlimit), &physparams.minlimit, sizeof(float3)));
}


void
geteulerconstants(PhysParams & physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams.epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams.maxlimit, HIP_SYMBOL(cueuler::d_maxlimit), sizeof(float3), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams.minlimit, HIP_SYMBOL(cueuler::d_minlimit), sizeof(float3), 0));
}


void
setmbdata(float4* MbData, uint size)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_mbdata), MbData, size));
}


void
seteulerrbcg(float3* cg, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcg), cg, numbodies*sizeof(float3)));
}


void
seteulerrbtrans(float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}


void
seteulerrbsteprot(float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}


void
euler(	float4*		oldPos,
		float4*		oldVel,
		particleinfo* info,
		float4*		forces,
		float4*		xsph,
		float4*		newPos,
		float4*		newVel,
		uint		numParticles,
		float		dt,
		float		dt2,
		int			step,
		float		t,
		bool		xsphcorr,
		bool		periodicbound)
{
	// thread per particle
	int numThreads = min(BLOCK_SIZE_INTEGRATE, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	// execute the kernel
	if (step == 1) {
		if (periodicbound) {
			if (xsphcorr)
				cueuler::eulerXsphDevice<1, true><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt2, dt2, t);
			else
				cueuler::eulerDevice<1, true><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt2, dt2, t);
		} else {
			if (xsphcorr)
				cueuler::eulerXsphDevice<1, false><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt2, dt2, t);
			else
				cueuler::eulerDevice<1, false><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt2, dt2, t);
		}
	} else if (step == 2) {
		if (periodicbound) {
			if (xsphcorr)
				cueuler::eulerXsphDevice<2, true><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt, dt2, t);
			else
				cueuler::eulerDevice<2, true><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt, dt2, t);
		} else {
			if (xsphcorr)
				cueuler::eulerXsphDevice<2, false><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt, dt2, t);
			else
				cueuler::eulerDevice<2, false><<< numBlocks, numThreads >>>(oldPos, oldVel, info,
									forces, xsph,
									newPos, newVel,
									numParticles, dt, dt2, t);
		}
	} // if (step == 2)

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Euler kernel execution failed");
}
}
