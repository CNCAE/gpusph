#include "hip/hip_runtime.h"
/*  Copyright 2011 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

	Istituto de Nazionale di Geofisica e Vulcanologia
          Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>

#include "euler.cuh"
#include "euler_kernel.cu"

#include "utils.h"

extern "C"
{
void
seteulerconstants(const PhysParams *physparams, const uint3 gridSize, const float3 cellSize)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams->epsxsph, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_gridSize), &gridSize, sizeof(uint3)));
}


void
geteulerconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
}


void
setmbdata(const float4* MbData, uint size)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_mbdata), MbData, size));
}


void
seteulerrbcg(const float3* cg, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcg), cg, numbodies*sizeof(float3)));
}


void
seteulerrbtrans(const float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}


void
seteulerrbsteprot(const float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}


void
euler(	const float4*		oldPos,
		const hashKey*		particleHash,
		const float4*		oldVel,
		const particleinfo* info,
		const float4*		forces,
		const float4*		xsph,
		float4*				newPos,
		float4*				newVel,
		const uint			numParticles,
		const float			dt,
		const float			dt2,
		const int			step,
		const float			t,
		const bool			xsphcorr)
{
	// thread per particle
	uint numThreads = min(BLOCK_SIZE_INTEGRATE, numParticles);
	uint numBlocks = div_up(numParticles, numThreads);

	// execute the kernel
	if (step == 1) {
		if (xsphcorr)
			cueuler::eulerXsphDevice<1><<< numBlocks, numThreads >>>(oldPos, particleHash, oldVel,
								info, forces, xsph, newPos, newVel, numParticles, dt2, dt2, t);
		else
			cueuler::eulerDevice<1><<< numBlocks, numThreads >>>(oldPos, particleHash, oldVel,
								info, forces, xsph, newPos, newVel, numParticles, dt2, dt2, t);
	} else if (step == 2) {
		if (xsphcorr)
			cueuler::eulerXsphDevice<2><<< numBlocks, numThreads >>>(oldPos, particleHash, oldVel,
								info, forces, xsph, newPos, newVel, numParticles, dt, dt2, t);
		else
			cueuler::eulerDevice<2><<< numBlocks, numThreads >>>(oldPos, particleHash, oldVel,
								info, forces, xsph, newPos, newVel, numParticles, dt, dt2, t);
	} // if (step == 2)

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Euler kernel execution failed");
}
}
