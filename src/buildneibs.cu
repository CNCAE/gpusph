#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <stdio.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include "textures.cuh"
#include "buildneibs.cuh"
#include "buildneibs_kernel.cu"

#include "utils.h"

extern "C"
{

void
setneibsconstants(const SimParams *simparams, const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxneibsnum), &simparams->maxneibsnum, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));


	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_gridSize), &gridSize, sizeof(uint3)));
}


void
getneibsconstants(SimParams *simparams, PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&simparams->maxneibsnum, HIP_SYMBOL(cuneibs::d_maxneibsnum), sizeof(uint), 0));
}


void
resetneibsinfo(void)
{
	uint temp = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_numInteractions), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxNeibs), &temp, sizeof(int)));
}


void
getneibsinfo(TimingInfo & timingInfo)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.numInteractions, HIP_SYMBOL(cuneibs::d_numInteractions), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.maxNeibs, HIP_SYMBOL(cuneibs::d_maxNeibs), sizeof(int), 0));
}


void
calcHash(float4*	pos,
		 hashKey*	particleHash,
		 uint*		particleIndex,
		 const particleinfo* particleInfo,
#if HASH_KEY_SIZE >= 64
		 uint*		compactDeviceMap,
#endif
		 const uint		numParticles,
		 const Periodicity	periodicbound)
{
	uint numThreads = min(BLOCK_SIZE_CALCHASH, numParticles);
	uint numBlocks = div_up(numParticles, numThreads);

	switch (periodicbound) {
		case PERIODIC_NONE:
			cuneibs::calcHashDevice<PERIODIC_NONE><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_X:
			cuneibs::calcHashDevice<PERIODIC_X><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_Y:
			cuneibs::calcHashDevice<PERIODIC_Y><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_XY:
			cuneibs::calcHashDevice<PERIODIC_XY><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_Z:
			cuneibs::calcHashDevice<PERIODIC_Z><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_XZ:
			cuneibs::calcHashDevice<PERIODIC_XZ><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_YZ:
			cuneibs::calcHashDevice<PERIODIC_YZ><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		case PERIODIC_XYZ:
			cuneibs::calcHashDevice<PERIODIC_XYZ><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
					   particleInfo,
#if HASH_KEY_SIZE >= 64
					   compactDeviceMap,
#endif
					   numParticles);
			break;

		default:
			throw std::runtime_error("Incorrect value of periodicbound!");
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("CalcHash kernel execution failed");
}

void
inverseParticleIndex (	uint*	particleIndex,
			uint*	inversedParticleIndex,
			uint	numParticles)
{
	int numThreads = min(BLOCK_SIZE_REORDERDATA, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	cuneibs::inverseParticleIndexDevice<<< numBlocks, numThreads >>>(particleIndex, inversedParticleIndex, numParticles);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("InverseParticleIndex kernel execution failed");
}

void reorderDataAndFindCellStart(	uint*				cellStart,			// output: cell start index
									uint*				cellEnd,			// output: cell end index
#if HASH_KEY_SIZE >= 64
									uint*			segmentStart,
#endif
									float4*				newPos,				// output: sorted positions
									float4*				newVel,				// output: sorted velocities
									particleinfo*		newInfo,			// output: sorted info
									float4*				newBoundElement,	// output: sorted boundary elements
									float4*				newGradGamma,		// output: sorted gradient gamma
									vertexinfo*			newVertices,		// output: sorted vertices
									float*				newTKE,				// output: k for k-e model
									float*				newEps,				// output: e for k-e model
									float*				newTurbVisc,		// output: eddy viscosity
									const hashKey*		particleHash,		// input: sorted grid hashes
									const uint*			particleIndex,		// input: sorted particle indices
									const float4*		oldPos,				// input: unsorted positions
									const float4*		oldVel,				// input: unsorted velocities
									const particleinfo*	oldInfo,			// input: unsorted info
									const float4*		oldBoundElement,	// input: sorted boundary elements
									const float4*		oldGradGamma,		// input: sorted gradient gamma
									const vertexinfo*	oldVertices,		// input: sorted vertices
									const float*		oldTKE,				// input: k for k-e model
									const float*		oldEps,				// input: e for k-e model
									const float*		oldTurbVisc,		// input: eddy viscosity
									const uint			numParticles,
									const uint			numGridCells,
									uint*				inversedParticleIndex)
{
	uint numThreads = min(BLOCK_SIZE_REORDERDATA, numParticles);
	uint numBlocks = div_up(numParticles, numThreads);

	// now in a separate function
	// CUDA_SAFE_CALL(hipMemset(cellStart, 0xffffffff, numGridCells*sizeof(uint)));

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, oldInfo, numParticles*sizeof(particleinfo)));

	// TODO reduce these conditionals

	if (oldBoundElement)
		CUDA_SAFE_CALL(hipBindTexture(0, boundTex, oldBoundElement, numParticles*sizeof(float4)));
	if (oldGradGamma)
		CUDA_SAFE_CALL(hipBindTexture(0, gamTex, oldGradGamma, numParticles*sizeof(float4)));
	if (oldVertices)
		CUDA_SAFE_CALL(hipBindTexture(0, vertTex, oldVertices, numParticles*sizeof(vertexinfo)));

	if (oldTKE)
		CUDA_SAFE_CALL(hipBindTexture(0, keps_kTex, oldTKE, numParticles*sizeof(float)));
	if (oldEps)
		CUDA_SAFE_CALL(hipBindTexture(0, keps_eTex, oldEps, numParticles*sizeof(float)));
	if (oldTurbVisc)
		CUDA_SAFE_CALL(hipBindTexture(0, tviscTex, oldTurbVisc, numParticles*sizeof(float)));

	uint smemSize = sizeof(uint)*(numThreads+1);
	cuneibs::reorderDataAndFindCellStartDevice<<< numBlocks, numThreads, smemSize >>>(cellStart, cellEnd,
#if HASH_KEY_SIZE >= 64
													segmentStart,
#endif
		newPos, newVel, newInfo, newBoundElement, newGradGamma, newVertices, newTKE, newEps, newTurbVisc,
												particleHash, particleIndex, numParticles, inversedParticleIndex);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("ReorderDataAndFindCellStart kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	if (oldBoundElement)
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	if (oldGradGamma)
		CUDA_SAFE_CALL(hipUnbindTexture(gamTex));
	if (oldVertices)
		CUDA_SAFE_CALL(hipUnbindTexture(vertTex));

	if (oldTKE)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_kTex));
	if (oldEps)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_eTex));
	if (oldTurbVisc)
		CUDA_SAFE_CALL(hipUnbindTexture(tviscTex));
}


void
buildNeibsList(	neibdata*			neibsList,
				const float4*		pos,
				const particleinfo*	info,
				vertexinfo*			vertices,
				const float4		*boundelem,
				float2*				vertPos[],
				const hashKey*		particleHash,
				const uint*			cellStart,
				const uint*			cellEnd,
				const uint			numParticles,
				const uint			particleRangeEnd,
				const uint			gridCells,
				const float			sqinfluenceradius,
				const float			sqdpo2,
				const Periodicity	periodicbound)
{
	const uint numThreads = min(BLOCK_SIZE_BUILDNEIBS, particleRangeEnd);
	const uint numBlocks = div_up(particleRangeEnd, numThreads);

	// bind textures to read all particles, not only internal ones
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, cellStartTex, cellStart, gridCells*sizeof(uint)));
	CUDA_SAFE_CALL(hipBindTexture(0, cellEndTex, cellEnd, gridCells*sizeof(uint)));
	if (vertices)
		CUDA_SAFE_CALL(hipBindTexture(0, vertTex, vertices, numParticles*sizeof(vertexinfo)));
	if (boundelem)
		CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelem, numParticles*sizeof(float4)));

	float2 *vertPos0, *vertPos1, *vertPos2;
	if (vertPos) {
		vertPos0 = vertPos[0];
		vertPos1 = vertPos[1];
		vertPos2 = vertPos[2];
	} else
		vertPos0 = vertPos1 = vertPos2 = NULL;


	switch (periodicbound) {
		case PERIODIC_NONE:
			cuneibs::buildNeibsListDevice<PERIODIC_NONE, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
		break;

		case PERIODIC_X:
				cuneibs::buildNeibsListDevice<PERIODIC_X, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;

		case PERIODIC_Y:
				cuneibs::buildNeibsListDevice<PERIODIC_Y, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;

		case PERIODIC_XY:
				cuneibs::buildNeibsListDevice<PERIODIC_XY, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;

		case PERIODIC_Z:
				cuneibs::buildNeibsListDevice<PERIODIC_Z, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;

		case PERIODIC_XZ:
				cuneibs::buildNeibsListDevice<PERIODIC_XZ, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;

		case PERIODIC_YZ:
				cuneibs::buildNeibsListDevice<PERIODIC_YZ, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;

		case PERIODIC_XYZ:
				cuneibs::buildNeibsListDevice<PERIODIC_XYZ, true><<< numBlocks, numThreads >>>(
						#if (__COMPUTE__ >= 20)
						pos,
						#endif
						vertPos0,
						vertPos1,
						vertPos2,
						particleHash,
						neibsList,
						particleRangeEnd,
						sqinfluenceradius,
						sqdpo2);
				break;
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("BuildNeibs kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(cellStartTex));
	CUDA_SAFE_CALL(hipUnbindTexture(cellEndTex));
	if (vertices)
		CUDA_SAFE_CALL(hipUnbindTexture(vertTex));
	if (boundelem)
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
}

void
sort(hashKey*	particleHash, uint*	particleIndex, uint	numParticles)
{
	thrust::device_ptr<hashKey> particleHash_devptr = thrust::device_pointer_cast(particleHash);
	thrust::device_ptr<uint> particleIndex_devptr = thrust::device_pointer_cast(particleIndex);

	thrust::sort_by_key(particleHash_devptr, particleHash_devptr + numParticles, particleIndex_devptr);

	CUT_CHECK_ERROR("thrust sort failed");

}
}
