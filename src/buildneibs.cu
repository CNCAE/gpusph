#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdexcept>

#include <stdio.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include "textures.cuh"
#include "buildneibs.cuh"

#include "buildneibs_params.h"
#include "buildneibs_kernel.cu"

#include "utils.h"

extern "C"
{

void
setneibsconstants(const SimParams *simparams, const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxneibsnum), &simparams->maxneibsnum, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));


	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_gridSize), &gridSize, sizeof(uint3)));
}


void
getneibsconstants(SimParams *simparams, PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&simparams->maxneibsnum, HIP_SYMBOL(cuneibs::d_maxneibsnum), sizeof(uint), 0));
}


void
resetneibsinfo(void)
{
	uint temp = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_numInteractions), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxNeibs), &temp, sizeof(int)));
}


void
getneibsinfo(TimingInfo & timingInfo)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.numInteractions, HIP_SYMBOL(cuneibs::d_numInteractions), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.maxNeibs, HIP_SYMBOL(cuneibs::d_maxNeibs), sizeof(int), 0));
}


void
calcHash(float4*	pos,
		 hashKey*	particleHash,
		 uint*		particleIndex,
		 const particleinfo* particleInfo,
		 uint*		compactDeviceMap,
		 const uint		numParticles,
		 const Periodicity	periodicbound)
{
	uint numThreads = BLOCK_SIZE_CALCHASH;
	uint numBlocks = div_up(numParticles, numThreads);

	switch (periodicbound) {
		case PERIODIC_NONE:
			cuneibs::calcHashDevice<PERIODIC_NONE><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_X:
			cuneibs::calcHashDevice<PERIODIC_X><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_Y:
			cuneibs::calcHashDevice<PERIODIC_Y><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_XY:
			cuneibs::calcHashDevice<PERIODIC_XY><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_Z:
			cuneibs::calcHashDevice<PERIODIC_Z><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_XZ:
			cuneibs::calcHashDevice<PERIODIC_XZ><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_YZ:
			cuneibs::calcHashDevice<PERIODIC_YZ><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		case PERIODIC_XYZ:
			cuneibs::calcHashDevice<PERIODIC_XYZ><<< numBlocks, numThreads >>>(pos, particleHash, particleIndex,
						particleInfo, compactDeviceMap, numParticles);
			break;

		default:
			throw std::runtime_error("Incorrect value of periodicbound!");
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("CalcHash kernel execution failed");
}

void
fixHash(hashKey*	particleHash,
		 uint*		particleIndex,
		 const particleinfo* particleInfo,
		 uint*		compactDeviceMap,
		 const uint		numParticles)
{
	uint numThreads = BLOCK_SIZE_CALCHASH;
	uint numBlocks = div_up(numParticles, numThreads);

	cuneibs::fixHashDevice<<< numBlocks, numThreads >>>(particleHash, particleIndex,
				particleInfo, compactDeviceMap, numParticles);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("FixHash kernel execution failed");
}


void reorderDataAndFindCellStart(	uint*				cellStart,			// output: cell start index
									uint*				cellEnd,			// output: cell end index
									uint*				segmentStart,
									float4*				newPos,				// output: sorted positions
									float4*				newVel,				// output: sorted velocities
									particleinfo*		newInfo,			// output: sorted info
									float4*				newBoundElement,	// output: sorted boundary elements
									float4*				newGradGamma,		// output: sorted gradient gamma
									vertexinfo*			newVertices,		// output: sorted vertices
									float*				newTKE,				// output: k for k-e model
									float*				newEps,				// output: e for k-e model
									float*				newTurbVisc,		// output: eddy viscosity
									float4*				newEulerVel,		// output: eulerian velocity
									const hashKey*		particleHash,		// input: sorted grid hashes
									const uint*			particleIndex,		// input: sorted particle indices
									const float4*		oldPos,				// input: unsorted positions
									const float4*		oldVel,				// input: unsorted velocities
									const particleinfo*	oldInfo,			// input: unsorted info
									const float4*		oldBoundElement,	// input: sorted boundary elements
									const float4*		oldGradGamma,		// input: sorted gradient gamma
									const vertexinfo*	oldVertices,		// input: sorted vertices
									const float*		oldTKE,				// input: k for k-e model
									const float*		oldEps,				// input: e for k-e model
									const float*		oldTurbVisc,		// input: eddy viscosity
									const float4*		oldEulerVel,		// input: eulerian velocity
									const uint			numParticles,
									uint*				newNumParticles)	// output: number of active particles found
{
	uint numThreads = BLOCK_SIZE_REORDERDATA;
	uint numBlocks = div_up(numParticles, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, oldInfo, numParticles*sizeof(particleinfo)));

	// TODO reduce these conditionals

	if (oldBoundElement)
		CUDA_SAFE_CALL(hipBindTexture(0, boundTex, oldBoundElement, numParticles*sizeof(float4)));
	if (oldGradGamma)
		CUDA_SAFE_CALL(hipBindTexture(0, gamTex, oldGradGamma, numParticles*sizeof(float4)));
	if (oldVertices)
		CUDA_SAFE_CALL(hipBindTexture(0, vertTex, oldVertices, numParticles*sizeof(vertexinfo)));

	if (oldTKE)
		CUDA_SAFE_CALL(hipBindTexture(0, keps_kTex, oldTKE, numParticles*sizeof(float)));
	if (oldEps)
		CUDA_SAFE_CALL(hipBindTexture(0, keps_eTex, oldEps, numParticles*sizeof(float)));
	if (oldTurbVisc)
		CUDA_SAFE_CALL(hipBindTexture(0, tviscTex, oldTurbVisc, numParticles*sizeof(float)));
	if (oldEulerVel)
		CUDA_SAFE_CALL(hipBindTexture(0, eulerVelTex, oldEulerVel, numParticles*sizeof(float4)));

	uint smemSize = sizeof(uint)*(numThreads+1);
	cuneibs::reorderDataAndFindCellStartDevice<<< numBlocks, numThreads, smemSize >>>(cellStart, cellEnd, segmentStart,
		newPos, newVel, newInfo, newBoundElement, newGradGamma, newVertices, newTKE, newEps, newTurbVisc,
		newEulerVel, particleHash, particleIndex, numParticles, newNumParticles);

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("ReorderDataAndFindCellStart kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	if (oldBoundElement)
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	if (oldGradGamma)
		CUDA_SAFE_CALL(hipUnbindTexture(gamTex));
	if (oldVertices)
		CUDA_SAFE_CALL(hipUnbindTexture(vertTex));

	if (oldTKE)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_kTex));
	if (oldEps)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_eTex));
	if (oldTurbVisc)
		CUDA_SAFE_CALL(hipUnbindTexture(tviscTex));

	if (oldEulerVel)
		CUDA_SAFE_CALL(hipUnbindTexture(eulerVelTex));
}

void
updateVertIDToIndex(particleinfo*	particleInfo,
					uint*			vertIDToIndex,
					const uint		numParticles)
{
	uint numThreads = BLOCK_SIZE_REORDERDATA;
	uint numBlocks = div_up(numParticles, numThreads);

	cuneibs::updateVertIDToIndexDevice<<< numBlocks, numThreads>>>(particleInfo, vertIDToIndex, numParticles);
}

void
buildNeibsList(	neibdata*			neibsList,
				const float4*		pos,
				const particleinfo*	info,
				vertexinfo*			vertices,
				const float4		*boundelem,
				float2*				vertPos[],
				const uint*			vertIDToIndex,
				const hashKey*		particleHash,
				const uint*			cellStart,
				const uint*			cellEnd,
				const uint			numParticles,
				const uint			particleRangeEnd,
				const uint			gridCells,
				const float			sqinfluenceradius,
				const float			boundNlSqInflRad,
				const BoundaryType	boundarytype,
				const Periodicity	periodicbound)
{
	// vertices, boundeleme and vertPos must be either all NULL or all not-NULL.
	// throw otherwise
	if (vertices || boundelem || vertPos) {
		if (!vertices || !boundelem || ! vertPos) {
			fprintf(stderr, "%p vs %p vs %p\n", vertices, boundelem, vertPos);
			throw std::invalid_argument("inconsistent params to buildNeibsList");
		}
	}

	if (boundarytype == SA_BOUNDARY && !vertices) {
		fprintf(stderr, "%s boundary type selected, but no vertices!\n",
			BoundaryName[boundarytype]);
		throw std::invalid_argument("missing data");
	}

	const uint numThreads = BLOCK_SIZE_BUILDNEIBS;
	const uint numBlocks = div_up(particleRangeEnd, numThreads);

	// bind textures to read all particles, not only internal ones
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, cellStartTex, cellStart, gridCells*sizeof(uint)));
	CUDA_SAFE_CALL(hipBindTexture(0, cellEndTex, cellEnd, gridCells*sizeof(uint)));

#define BUILDNEIBS_CASE(btype, periodic) \
	case periodic: \
		cuneibs::buildNeibsListDevice<btype, periodic, true><<<numBlocks, numThreads>>>(params); \
		break;

#define BUILDNEIBS_SWITCH(btype) \
	switch(periodicbound) { \
		BUILDNEIBS_CASE(btype, PERIODIC_NONE); \
		BUILDNEIBS_CASE(btype, PERIODIC_X); \
		BUILDNEIBS_CASE(btype, PERIODIC_Y); \
		BUILDNEIBS_CASE(btype, PERIODIC_XY); \
		BUILDNEIBS_CASE(btype, PERIODIC_Z); \
		BUILDNEIBS_CASE(btype, PERIODIC_XZ); \
		BUILDNEIBS_CASE(btype, PERIODIC_YZ); \
		BUILDNEIBS_CASE(btype, PERIODIC_XYZ); \
	}

	if (boundarytype == SA_BOUNDARY) {
		CUDA_SAFE_CALL(hipBindTexture(0, vertTex, vertices, numParticles*sizeof(vertexinfo)));
		CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelem, numParticles*sizeof(float4)));

		buildneibs_params<true> params(neibsList, pos, particleHash, particleRangeEnd, sqinfluenceradius,
			vertPos, vertIDToIndex, boundNlSqInflRad);

		BUILDNEIBS_SWITCH(SA_BOUNDARY);

		CUDA_SAFE_CALL(hipUnbindTexture(vertTex));
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	} else {
		buildneibs_params<false> params(neibsList, pos, particleHash, particleRangeEnd, sqinfluenceradius,
			vertPos, vertIDToIndex, boundNlSqInflRad);

		// In non-SA boundary case, the only difference is between DYN and non-DYN
		// boundary (because DYN_BOUNDARY needs to build neib list for boundary particles too).
		// To avoid building too many variants of the kernels we will collect all
		// non-SA, non-DYN boundary into the LJ case, since they all behave the same
		if (boundarytype == DYN_BOUNDARY) {
			BUILDNEIBS_SWITCH(DYN_BOUNDARY);
		} else {
			BUILDNEIBS_SWITCH(LJ_BOUNDARY);
		}
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("BuildNeibs kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(cellStartTex));
	CUDA_SAFE_CALL(hipUnbindTexture(cellEndTex));
}

void
sort(hashKey*	particleHash, uint*	particleIndex, uint	numParticles)
{
	thrust::device_ptr<hashKey> particleHash_devptr = thrust::device_pointer_cast(particleHash);
	thrust::device_ptr<uint> particleIndex_devptr = thrust::device_pointer_cast(particleIndex);

	thrust::sort_by_key(particleHash_devptr, particleHash_devptr + numParticles, particleIndex_devptr);

	CUT_CHECK_ERROR("thrust sort failed");

}
}
