#include "hip/hip_runtime.h"
/*  Copyright 2011 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

	Istituto de Nazionale di Geofisica e Vulcanologia
          Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/functional.h>

#include "textures.cuh"
#include "forces.cuh"

hipArray*  dDem = NULL;

/* Auxiliary data for parallel reductions */
size_t	reduce_blocks = 0;
size_t	reduce_blocksize_max = 0;
size_t	reduce_bs2 = 0;
size_t	reduce_shmem_max = 0;
void*	reduce_buffer = NULL;

/* These defines give a shorthand for the kernel with a given correction,
   viscosity, xsph and dt options. They will be used in forces.cu for
   consistency */
#define _FORCES_KERNEL_NAME(visc, xsph, dt) forces_##visc##_##xsph##dt##Device
#define FORCES_KERNEL_NAME(visc, xsph, dt) _FORCES_KERNEL_NAME(visc, xsph, dt)

#include "forces_kernel.cu"

#define NOT_IMPLEMENTED_CHECK(what, arg) \
		default: \
			fprintf(stderr, #what " %s (%u) not implemented\n", what##Name[arg], arg); \
			exit(1)

#define KERNEL_CHECK(kernel, boundarytype, periodic, formulation, visc, dem) \
	case kernel: \
		if (!dtadapt && !xsphcorr) \
				cuforces::FORCES_KERNEL_NAME(visc,,)<kernel, boundarytype, periodic, dem, formulation><<< numBlocks, numThreads, dummy_shared >>>\
						(pos, forces, keps_dkde, turbvisc, neibsList, numParticles, slength, influenceradius, rbforces, rbtorques); \
		else if (!dtadapt && xsphcorr) \
				cuforces::FORCES_KERNEL_NAME(visc, Xsph,)<kernel, boundarytype, periodic, dem, formulation><<< numBlocks, numThreads, dummy_shared >>>\
						(pos, forces, keps_dkde, turbvisc, xsph, neibsList, numParticles, slength, influenceradius, rbforces, rbtorques); \
		else if (dtadapt && !xsphcorr) \
				cuforces::FORCES_KERNEL_NAME(visc,, Dt)<kernel, boundarytype, periodic, dem, formulation><<< numBlocks, numThreads, dummy_shared >>>\
						(pos, forces, keps_dkde, turbvisc, neibsList, numParticles, slength, influenceradius, rbforces, rbtorques, cfl, cflGamma); \
		else if (dtadapt && xsphcorr) \
				cuforces::FORCES_KERNEL_NAME(visc, Xsph, Dt)<kernel, boundarytype, periodic, dem, formulation><<< numBlocks, numThreads, dummy_shared >>>\
						(pos, forces, keps_dkde, turbvisc, xsph, neibsList, numParticles, slength, influenceradius, rbforces, rbtorques, cfl, cflGamma); \
		break

#define KERNEL_SWITCH(formulation, boundarytype, periodic, visc, dem) \
	switch (kerneltype) { \
		KERNEL_CHECK(CUBICSPLINE,	boundarytype, periodic, formulation, visc, dem); \
		KERNEL_CHECK(WENDLAND,		boundarytype, periodic, formulation, visc, dem); \
		NOT_IMPLEMENTED_CHECK(Kernel, kerneltype); \
	}

#define FORMULATION_CHECK(formulation, boundarytype, periodic, visc, dem) \
	case formulation: \
		KERNEL_SWITCH(formulation, boundarytype, periodic, visc, dem) \
		break

#define FORMULATION_SWITCH(boundarytype, periodic, visc, dem) \
	switch (sph_formulation) { \
		FORMULATION_CHECK(SPH_F1, boundarytype, periodic, visc, dem); \
		FORMULATION_CHECK(SPH_F2, boundarytype, periodic, visc, dem); \
		NOT_IMPLEMENTED_CHECK(SPHFormulation, sph_formulation); \
	}

#define VISC_CHECK(boundarytype, periodic, visc, dem) \
	case visc: \
		FORMULATION_SWITCH(boundarytype, periodic, visc, dem) \
		break

#define VISC_CHECK_STANDARD(boundarytype, periodic, dem) \
		VISC_CHECK(boundarytype, periodic, ARTVISC, dem); \
		VISC_CHECK(boundarytype, periodic, DYNAMICVISC, dem); \
		VISC_CHECK(boundarytype, periodic, KINEMATICVISC, dem);\
		VISC_CHECK(boundarytype, periodic, SPSVISC, dem); \
		VISC_CHECK(boundarytype, periodic, KEPSVISC, dem);

#define VISC_SWITCH(boundarytype, periodic, dem) \
	switch (visctype) { \
		VISC_CHECK_STANDARD(boundarytype, periodic, dem); \
		NOT_IMPLEMENTED_CHECK(Viscosity, visctype); \
	}

//		VISC_CHECK_SPS(boundarytype, periodic, dem); \

#define BOUNDARY_CHECK(boundary, periodic, dem) \
	case boundary: \
		VISC_SWITCH(boundary, periodic, dem) \
		break

#define BOUNDARY_SWITCH(periodic, dem) \
	switch (boundarytype) { \
		BOUNDARY_CHECK(LJ_BOUNDARY, periodic, dem); \
		BOUNDARY_CHECK(MK_BOUNDARY, periodic, dem); \
		BOUNDARY_CHECK(MF_BOUNDARY, periodic, dem); \
		NOT_IMPLEMENTED_CHECK(Boundary, boundarytype); \
	}

#define SPS_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::SPSstressMatrixDevice<kernel, periodic><<< numBlocks, numThreads, dummy_shared >>> \
				(pos, tau[0], tau[1], tau[2], neibsList, numParticles, slength, influenceradius); \
		break

#define KEPS_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::MeanScalarStrainRateDevice<kernel, periodic><<< numBlocks, numThreads, dummy_shared >>> \
				(pos, strainrate, neibsList, numParticles, slength, influenceradius); \
		break

#define SHEPARD_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::shepardDevice<kernel, periodic><<< numBlocks, numThreads, dummy_shared >>> \
				 (pos, newVel, neibsList, numParticles, slength, influenceradius); \
	break

#define MLS_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::MlsDevice<kernel, periodic><<< numBlocks, numThreads, dummy_shared >>> \
				(pos, newVel, neibsList, numParticles, slength, influenceradius); \
	break

#define VORT_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::calcVortDevice<kernel, periodic><<< numBlocks, numThreads >>> \
				 (vort, neibsList, numParticles, slength, influenceradius); \
	break

//Testpoints
#define TEST_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::calcTestpointsVelocityDevice<kernel, periodic><<< numBlocks, numThreads >>> \
				(newVel, neibsList, numParticles, slength, influenceradius); \
	break

// Free surface detection
#define SURFACE_CHECK(kernel, periodic, savenormals) \
	case kernel: \
		cuforces::calcSurfaceparticleDevice<kernel, periodic, savenormals><<< numBlocks, numThreads >>> \
				(normals, newInfo, neibsList, numParticles, slength, influenceradius); \
	break

#define INITGRADGAMMA_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::initGradGammaDevice<kernel, periodic><<< numBlocks, numThreads>>> \
				(newPos, virtualVel, gradGamma, neibsList, numParticles, deltap, slength, inflRadius); \
	break

#define UPDATEGAMMA_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::updateGammaDevice<kernel, periodic><<< numBlocks, numThreads>>> \
				(newGam, neibsList, numParticles, slength, inflRadius, virtDt); \
	break

#define UPDATEGAMMAPRCOR_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::updateGammaPrCorDevice<kernel, periodic><<< numBlocks, numThreads>>> \
				(newPos, newGam, neibsList, numParticles, slength, inflRadius, virtDt); \
	break

#define DYNBOUNDARY_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::dynamicBoundConditionsDevice<kernel, periodic><<< numBlocks, numThreads, dummy_shared >>> \
				 (oldPos, oldVel, oldPressure, neibsList, numParticles, slength, influenceradius); \
	break

#define CALCPROBE_CHECK(kernel, periodic) \
	case kernel: \
		cuforces::calcProbeDevice<kernel, periodic><<< numBlocks, numThreads, dummy_shared >>> \
				 (oldPos, oldVel, oldPressure, neibsList, numParticles, slength, influenceradius); \
	break

extern "C"
{
void
setforcesconstants(const SimParams *simparams, const PhysParams *physparams)
{
	// Setting kernels and kernels derivative factors
	float h = simparams->slength;
	float h3 = h*h*h;
	float h4 = h3*h;
	float h5 = h4*h;
	float kernelcoeff = 1.0f/(M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_wcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_wcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 21.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_wcoeff_wendland), &kernelcoeff, sizeof(float)));

	kernelcoeff = 3.0f/(4.0f*M_PI*h4);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_fcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(32.0f*M_PI*h4);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_fcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 105.0f/(128.0f*M_PI*h5);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_fcoeff_wendland), &kernelcoeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_numfluids), &physparams->numFluids, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_rho0), &physparams->rho0, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_bcoeff), &physparams->bcoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gammacoeff), &physparams->gammacoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_sscoeff), &physparams->sscoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_sspowercoeff), &physparams->sspowercoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gravity), &physparams->gravity, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_dcoeff), &physparams->dcoeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_p1coeff), &physparams->p1coeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_p2coeff), &physparams->p2coeff, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_MK_K), &physparams->MK_K, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_MK_d), &physparams->MK_d, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_MK_beta), &physparams->MK_beta, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_r0), &physparams->r0, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_visccoeff), &physparams->visccoeff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_epsartvisc), &physparams->epsartvisc, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_dispvect), &physparams->dispvect, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_ewres), &physparams->ewres, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_nsres), &physparams->nsres, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demdx), &physparams->demdx, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demdy), &physparams->demdy, sizeof(float)));
	float demdxdy = physparams->demdx*physparams->demdy;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demdxdy), &demdxdy, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_demzmin), &physparams->demzmin, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_smagfactor), &physparams->smagfactor, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_kspsfactor), &physparams->kspsfactor, sizeof(float)));

	float partsurf = physparams->partsurf;
	if (partsurf == 0.0f)
		partsurf = physparams->r0*physparams->r0;
		// partsurf = (6.0 - M_PI)*physparams->r0*physparams->r0/4;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_partsurf), &partsurf, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_cosconeanglefluid), &physparams->cosconeanglefluid, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_cosconeanglenonfluid), &physparams->cosconeanglenonfluid, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_objectobjectdf), &physparams->objectobjectdf, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_objectboundarydf), &physparams->objectboundarydf, sizeof(float)));

	uint maxneibs_time_neibinterleave = simparams->maxneibsnum*NEIBINDEX_INTERLEAVE;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_maxneibsnum_time_neibindexinterleave), &maxneibs_time_neibinterleave, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_ferrari), &simparams->ferrari, sizeof(float)));
}


void
getforcesconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->numFluids, HIP_SYMBOL(cuforces::d_numfluids), sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->rho0, HIP_SYMBOL(cuforces::d_rho0), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->gravity, HIP_SYMBOL(cuforces::d_gravity), sizeof(float3), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->bcoeff, HIP_SYMBOL(cuforces::d_bcoeff), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->gammacoeff, HIP_SYMBOL(cuforces::d_gammacoeff), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->sscoeff, HIP_SYMBOL(cuforces::d_sscoeff), MAX_FLUID_TYPES*sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->sspowercoeff, HIP_SYMBOL(cuforces::d_sspowercoeff), MAX_FLUID_TYPES*sizeof(float), 0));

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->dcoeff, HIP_SYMBOL(cuforces::d_dcoeff), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->p1coeff, HIP_SYMBOL(cuforces::d_p1coeff), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->p2coeff, HIP_SYMBOL(cuforces::d_p2coeff), sizeof(float), 0));

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->MK_K, HIP_SYMBOL(cuforces::d_MK_K), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->MK_d, HIP_SYMBOL(cuforces::d_MK_d), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->MK_beta, HIP_SYMBOL(cuforces::d_MK_beta), sizeof(float), 0));

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->r0, HIP_SYMBOL(cuforces::d_r0), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->visccoeff, HIP_SYMBOL(cuforces::d_visccoeff), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsartvisc, HIP_SYMBOL(cuforces::d_epsartvisc), sizeof(float), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->ewres, HIP_SYMBOL(cuforces::d_ewres), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->nsres, HIP_SYMBOL(cuforces::d_nsres), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->demdx, HIP_SYMBOL(cuforces::d_demdx), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->demdy, HIP_SYMBOL(cuforces::d_demdy), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->demzmin, HIP_SYMBOL(cuforces::d_demzmin), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->smagfactor, HIP_SYMBOL(cuforces::d_smagfactor), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->kspsfactor, HIP_SYMBOL(cuforces::d_kspsfactor), sizeof(float)));
}


void
setplaneconstants(int numPlanes, const float* PlanesDiv, const float4* Planes)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_planes), Planes, numPlanes*sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_plane_div), PlanesDiv, numPlanes*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_numplanes), &numPlanes, sizeof(uint)));
}


void
setgravity(float3 const& gravity)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_gravity), &gravity, sizeof(float3)));
}


void
setforcesrbcg(const float3* cg, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_rbcg), cg, numbodies*sizeof(float3)));
}


void
setforcesrbstart(const uint* rbfirstindex, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuforces::d_rbstartindex), rbfirstindex, numbodies*sizeof(uint)));
}


float
forces(	float4*			pos,
		float4*			vel,
		float4*			forces,
		float4*			gradgam,
		float4*			boundelem,
		float*			pressure,
		float4*			rbforces,
		float4*			rbtorques,
		float4*			xsph,
		particleinfo	*info,
		uint*			neibsList,
		uint			numParticles,
		float			slength,
		float			dt,
		bool			dtadapt,
		float			dtadaptfactor,
		bool			xsphcorr,
		KernelType		kerneltype,
		float			influenceradius,
		ViscosityType	visctype,
		float			visccoeff,
		float*			strainrate,
		float*			turbvisc,
		float*			keps_tke,
		float*			keps_eps,
		float2*			keps_dkde,
		float*			cfl,
		float*			cflGamma,
		float*			tempCfl,
		uint			numPartsFmax,
		float2*			tau[],
		bool			periodicbound,
		SPHFormulation	sph_formulation,
		BoundaryType	boundarytype,
		bool			usedem)
{
	int dummy_shared = 0;
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, gamTex, gradgam, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelem, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, presTex, pressure, numParticles*sizeof(float)));
	CUDA_SAFE_CALL(hipBindTexture(0, keps_kTex, keps_tke, numParticles*sizeof(float)));
	CUDA_SAFE_CALL(hipBindTexture(0, keps_eTex, keps_eps, numParticles*sizeof(float)));

	// execute the kernel for computing SPS stress matrix, if needed
	if (visctype == SPSVISC) {	// thread per particle
		int numThreads = min(BLOCK_SIZE_SPS, numParticles);
		int numBlocks = (int) ceil(numParticles / (float) numThreads);
		#if (__COMPUTE__ == 20)
		dummy_shared = 2560;
		#endif
		if (periodicbound) {
			switch (kerneltype) {
				SPS_CHECK(CUBICSPLINE, true);
				SPS_CHECK(QUADRATIC, true);
				SPS_CHECK(WENDLAND, true);
			}
		} else {
			switch (kerneltype) {
				SPS_CHECK(CUBICSPLINE, false);
				SPS_CHECK(QUADRATIC, false);
				SPS_CHECK(WENDLAND, false);
			}
		}
		// check if kernel invocation generated an error
		CUT_CHECK_ERROR("SPS kernel execution failed");
		
		CUDA_SAFE_CALL(hipBindTexture(0, tau0Tex, tau[0], numParticles*sizeof(float2)));
		CUDA_SAFE_CALL(hipBindTexture(0, tau1Tex, tau[1], numParticles*sizeof(float2)));
		CUDA_SAFE_CALL(hipBindTexture(0, tau2Tex, tau[2], numParticles*sizeof(float2)));
	}

	// execute the kernel for computing mean scalar strain rate for k-e model
	if (visctype == KEPSVISC) {
		int numThreads = min(BLOCK_SIZE_SPS, numParticles);
		int numBlocks = (int) ceil(numParticles / (float) numThreads);
		#if (__COMPUTE__ == 20)
		dummy_shared = 2560;
		#endif
		if (periodicbound) {
			switch (kerneltype) {
				KEPS_CHECK(CUBICSPLINE, true);
				//KEPS_CHECK(QUADRATIC, true);
				KEPS_CHECK(WENDLAND, true);
			}
		} else {
			switch (kerneltype) {
				KEPS_CHECK(CUBICSPLINE, false);
				//KEPS_CHECK(QUADRATIC, false);
				KEPS_CHECK(WENDLAND, false);
			}
		}
		// check if kernel invocation generated an error
		CUT_CHECK_ERROR("MeanScalarStrainRate kernel execution failed");

		CUDA_SAFE_CALL(hipBindTexture(0, strainTex, strainrate, numParticles*sizeof(float)));
	}
	
	// thread per particle
	int numThreads = min(BLOCK_SIZE_FORCES, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);		
	#if (__COMPUTE__ == 20)
	if (visctype == SPSVISC)
		dummy_shared = 3328 - dtadapt*BLOCK_SIZE_FORCES*4;
	else
		dummy_shared = 2560 - dtadapt*BLOCK_SIZE_FORCES*4;
	#endif
	if (usedem) {
		if (periodicbound) {
			BOUNDARY_SWITCH(true, true)
		} else {
			BOUNDARY_SWITCH(false, true)
		}
	} else {
		if (periodicbound) {
			BOUNDARY_SWITCH(true, false)
		} else {
			BOUNDARY_SWITCH(false, false)
		}
	}
	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Forces kernel execution failed");

	if (visctype == SPSVISC) {
		CUDA_SAFE_CALL(hipUnbindTexture(tau0Tex));
		CUDA_SAFE_CALL(hipUnbindTexture(tau1Tex));
		CUDA_SAFE_CALL(hipUnbindTexture(tau2Tex));
	}
	
	if (visctype == KEPSVISC) {
		CUDA_SAFE_CALL(hipUnbindTexture(strainTex));
	}

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(gamTex));
	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	CUDA_SAFE_CALL(hipUnbindTexture(presTex));
	CUDA_SAFE_CALL(hipUnbindTexture(keps_kTex));
	CUDA_SAFE_CALL(hipUnbindTexture(keps_eTex));

	if (dtadapt) {
		float maxcfl = cflmax(numPartsFmax, cfl, tempCfl);
		dt = dtadaptfactor*sqrtf(slength/maxcfl);

		if (visctype != ARTVISC) {
			/* Stability condition from viscosity h²/ν */
			float dt_visc = slength*slength/visccoeff;
			switch (visctype) {
				case KINEMATICVISC:
				case SPSVISC:
				/* ν = visccoeff/4 for kinematic viscosity */
					dt_visc *= 4;
					break;

				case DYNAMICVISC:
				/* ν = visccoeff for dynamic viscosity */
					break;
				}
			dt_visc *= 0.125;
			if (dt_visc < dt)
				dt = dt_visc;
		}

		if(boundarytype == MF_BOUNDARY) {
			float dt_gamma = 0.005/cflmax(numPartsFmax, cflGamma, tempCfl);
			if (dt_gamma < dt)
				dt = dt_gamma;
		}
	}
	return dt;
}


void
shepard(float4*		pos,
		float4*		oldVel,
		float4*		newVel,
		particleinfo	*info,
		uint*		neibsList,
		uint		numParticles,
		float		slength,
		int			kerneltype,
		float		influenceradius,
		bool		periodicbound)
{
	int dummy_shared = 0;
	// thread per particle
	int numThreads = min(BLOCK_SIZE_SHEPARD, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	
	// execute the kernel
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	if (periodicbound) {
		switch (kerneltype) {
			SHEPARD_CHECK(CUBICSPLINE, true);
//			SHEPARD_CHECK(QUADRATIC, true);
			SHEPARD_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			SHEPARD_CHECK(CUBICSPLINE, false);
//			SHEPARD_CHECK(QUADRATIC, false);
			SHEPARD_CHECK(WENDLAND, false);
		}
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Shepard kernel execution failed");
	
	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

}


void
mls(float4*		pos,
	float4*		oldVel,
	float4*		newVel,
	particleinfo	*info,
	uint*		neibsList,
	uint		numParticles,
	float		slength,
	int			kerneltype,
	float		influenceradius,
	bool		periodicbound)
{
	int dummy_shared = 0;
	// thread per particle
	int numThreads = min(BLOCK_SIZE_MLS, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel		
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	if (periodicbound) {
		switch (kerneltype) {
			MLS_CHECK(CUBICSPLINE, true);
//			MLS_CHECK(QUADRATIC, true);
			MLS_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			MLS_CHECK(CUBICSPLINE, false);
//			MLS_CHECK(QUADRATIC, false);
			MLS_CHECK(WENDLAND, false);
		}
	}
	
	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Mls kernel execution failed");

	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

void
vorticity(	float4*		pos,
			float4*		vel,
			float3*		vort,
			particleinfo	*info,
			uint*		neibsList,
			uint		numParticles,
			float		slength,
			int			kerneltype,
			float		influenceradius,
			bool		periodicbound)
{
	// thread per particle
	int numThreads = min(BLOCK_SIZE_CALCVORT, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	if (periodicbound) {
		switch (kerneltype) {
			VORT_CHECK(CUBICSPLINE, true);
//			VORT_CHECK(QUADRATIC, true);
			VORT_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			VORT_CHECK(CUBICSPLINE, false);
//			VORT_CHECK(QUADRATIC, false);
			VORT_CHECK(WENDLAND, false);
		}
	}
	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Shepard kernel execution failed");
	
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

//Testpoints
void
testpoints( float4*		pos,
			float4*		newVel,
			particleinfo	*info,
			uint*		neibsList,
			uint		numParticles,
			float		slength,
			int			kerneltype,
			float		influenceradius,
			bool		periodicbound)
{
	// thread per particle
	int numThreads = min(BLOCK_SIZE_CALCTEST, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, newVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	if (periodicbound) {
		switch (kerneltype) {
			TEST_CHECK(CUBICSPLINE, true);
//			TEST_CHECK(QUADRATIC, true);
			TEST_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			TEST_CHECK(CUBICSPLINE, false);
//			TEST_CHECK(QUADRATIC, false);
			TEST_CHECK(WENDLAND, false);
		}
	}
	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("test kernel execution failed");
	
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}

// Free surface detection
void
surfaceparticle(	float4*		pos,
					float4*     vel,
					float4*		normals,
					particleinfo	*info,
					particleinfo	*newInfo,
					uint*		neibsList,
					uint		numParticles,
					float		slength,
					int			kerneltype,
					float		influenceradius,
					bool		periodicbound,
					bool        savenormals)
{
	// thread per particle
	int numThreads = min(BLOCK_SIZE_CALCTEST, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	if (savenormals){
		if (periodicbound) {
			switch (kerneltype) {
				SURFACE_CHECK(CUBICSPLINE, true, true);
//				SURFACE_CHECK(QUADRATIC, true, true);
				SURFACE_CHECK(WENDLAND, true, true);
			}
		} else {
			switch (kerneltype) {
				SURFACE_CHECK(CUBICSPLINE, false, true);
//				SURFACE_CHECK(QUADRATIC, false, true);
				SURFACE_CHECK(WENDLAND, false, true);
			}
		}
	} else {
		if (periodicbound) {
			switch (kerneltype) {
				SURFACE_CHECK(CUBICSPLINE, true, false);
//				SURFACE_CHECK(QUADRATIC, true, false);
				SURFACE_CHECK(WENDLAND, true, false);
			}
		} else {
			switch (kerneltype) {
				SURFACE_CHECK(CUBICSPLINE, false, false);
//				SURFACE_CHECK(QUADRATIC, false, false);
				SURFACE_CHECK(WENDLAND, false, false);
			}
		}
	}
	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("surface kernel execution failed");
	
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
}


void setDemTexture(const float *hDem, int width, int height)
{
	// Allocating, reading and copying DEM
	unsigned int size = width*height*sizeof(float);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL( hipMallocArray( &dDem, &channelDesc, width, height ));
	CUDA_SAFE_CALL( hipMemcpyToArray( dDem, 0, 0, hDem, size, hipMemcpyHostToDevice));

	demTex.addressMode[0] = hipAddressModeClamp;
	demTex.addressMode[1] = hipAddressModeClamp;
	demTex.filterMode = hipFilterModeLinear;
	demTex.normalized = false;

	CUDA_SAFE_CALL( hipBindTextureToArray(demTex, dDem, channelDesc));
}


void releaseDemTexture()
{
	CUDA_SAFE_CALL(hipFreeArray(dDem));
}


void reduceRbForces(float4*		forces,
					float4*		torques,
					uint*		rbnum,
					uint*		lastindex,
					float3*		totalforce,
					float3*		totaltorque,
					uint		numbodies,
					uint		numBodiesParticles)
{
	thrust::device_ptr<float4> forces_devptr = thrust::device_pointer_cast(forces);
	thrust::device_ptr<float4> torques_devptr = thrust::device_pointer_cast(torques);
	thrust::device_ptr<uint> rbnum_devptr = thrust::device_pointer_cast(rbnum);
	thrust::equal_to<uint> binary_pred;
	thrust::plus<float4> binary_op;

	thrust::inclusive_scan_by_key(rbnum_devptr, rbnum_devptr + numBodiesParticles, 
				forces_devptr, forces_devptr, binary_pred, binary_op);
	thrust::inclusive_scan_by_key(rbnum_devptr, rbnum_devptr + numBodiesParticles, 
				torques_devptr, torques_devptr, binary_pred, binary_op);
	
	for (int i = 0; i < numbodies; i++) {
		float4 temp;
		void * ddata = (void *) (forces + lastindex[i]);
		CUDA_SAFE_CALL(hipMemcpy((void *) &temp, ddata, sizeof(float4), hipMemcpyDeviceToHost));
		totalforce[i] = as_float3(temp);
		
		ddata = (void *) (torques + lastindex[i]);
		CUDA_SAFE_CALL(hipMemcpy((void *) &temp, ddata, sizeof(float4), hipMemcpyDeviceToHost));
		totaltorque[i] = as_float3(temp);
		}
}


void 
reducefmax(	const int	size, 
			const int	threads, 
			const int	blocks, 
			float		*d_idata, 
			float		*d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps 
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

	switch (threads)
	{
		case 512:
			cuforces::fmaxDevice<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 256:
			cuforces::fmaxDevice<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 128:
			cuforces::fmaxDevice<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 64:
			cuforces::fmaxDevice<64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 32:
			cuforces::fmaxDevice<32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case 16:
			cuforces::fmaxDevice<16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  8:
			cuforces::fmaxDevice<8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  4:
			cuforces::fmaxDevice<4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  2:
			cuforces::fmaxDevice<2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
		case  1:
			cuforces::fmaxDevice<1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	}
}


uint nextPow2(uint x ) 
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


#define MIN(x,y) ((x < y) ? x : y)
void getNumBlocksAndThreads(const uint	n, 
							const uint	maxBlocks, 
							const uint	maxThreads, 
							uint		&blocks, 
							uint		&threads)
{
	threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
	blocks = MIN(maxBlocks, blocks);
}


uint
getNumPartsFmax(const uint n)
{
	return (int) ceil(n / (float) min(BLOCK_SIZE_FORCES, n));
}
	

uint
getFmaxTempStorageSize(const uint n)
{
	uint numBlocks, numThreads;
	getNumBlocksAndThreads(n, MAX_BLOCKS_FMAX, BLOCK_SIZE_FMAX, numBlocks, numThreads);
	return numBlocks*sizeof(float);
}


float
cflmax( const uint	n,
		float*		cfl,
		float*		tempCfl)
{
	uint numBlocks = 0;
	uint numThreads = 0;
	float max = 0.0f;

	getNumBlocksAndThreads(n, MAX_BLOCKS_FMAX, BLOCK_SIZE_FMAX, numBlocks, numThreads);

	// execute the kernel
	reducefmax(n, numThreads, numBlocks, cfl, tempCfl);

	// check if kernel execution generated an error
	CUT_CHECK_ERROR("fmax kernel execution failed");

	uint s = numBlocks;
	while(s > 1) 
	{
		uint threads = 0, blocks = 0;
		getNumBlocksAndThreads(s, MAX_BLOCKS_FMAX, BLOCK_SIZE_FMAX, blocks, threads);

		reducefmax(s, threads, blocks, tempCfl, tempCfl); //FIXME: incorrect parameters
		CUT_CHECK_ERROR("fmax kernel execution failed");

		s = (s + (threads*2-1)) / (threads*2);
	}

	CUDA_SAFE_CALL(hipMemcpy(&max, tempCfl, sizeof(float), hipMemcpyDeviceToHost));
	
	return max;
}

/* Reductions */
void set_reduction_params(void* buffer, size_t blocks,
		size_t blocksize_max, size_t shmem_max)
{
	reduce_blocks = blocks;
	// in the second step of a reduction, a single block is launched, whose size
	// should be the smallest power of two that covers the number of blocks used
	// in the previous reduction run
	reduce_bs2 = 32;
	while (reduce_bs2 < blocks)
		reduce_bs2<<=1;

	reduce_blocksize_max = blocksize_max;
	reduce_shmem_max = shmem_max;
	reduce_buffer = buffer;
}

void unset_reduction_params()
{
	CUDA_SAFE_CALL(hipFree(reduce_buffer));
	reduce_buffer = NULL;
}

// Compute system energy
void calc_energy(
		float4*			output,
		float4	const*	pos,
		float4	const*	vel,
	particleinfo const*	pinfo,
		uint			numParticles,
		uint			numFluids)
{
	// shmem needed by a single thread
	size_t shmem_thread = numFluids*sizeof(float4)*2;
	size_t blocksize_max = reduce_shmem_max/shmem_thread;
	if (blocksize_max > reduce_blocksize_max)
		blocksize_max = reduce_blocksize_max;

	size_t blocksize = 32;
	while (blocksize*2 < blocksize_max)
		blocksize<<=1;

	cuforces::calcEnergies<<<reduce_blocks, blocksize, blocksize*shmem_thread>>>(
			pos, vel, pinfo, numParticles, numFluids, (float4*)reduce_buffer);
	CUT_CHECK_ERROR("System energy stage 1 failed");

	cuforces::calcEnergies2<<<1, reduce_bs2, reduce_bs2*shmem_thread>>>(
			(float4*)reduce_buffer, reduce_blocks, numFluids);
	CUT_CHECK_ERROR("System energy stage 2 failed");
	CUDA_SAFE_CALL(hipMemcpy(output, reduce_buffer, numFluids*sizeof(float4), hipMemcpyDeviceToHost));
}

void
initGradGamma(	float4*		oldPos,
		float4*		newPos,
		float4*		virtualVel,
		particleinfo*	info,
		float4*		boundElement,
		float4*		gradGamma,
		uint*		neibsList,
		uint		numParticles,
		float		deltap,
		float		slength,
		float		inflRadius,
		int		kerneltype,
		bool		periodicbound)
{
	int numThreads = min(BLOCK_SIZE_FORCES, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);
	
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundElement, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	
	//execute kernel
	if (periodicbound) {
		switch (kerneltype) {
			INITGRADGAMMA_CHECK(CUBICSPLINE, true);
//			INITGRADGAMMA_CHECK(QUADRATIC, true);
			INITGRADGAMMA_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			INITGRADGAMMA_CHECK(CUBICSPLINE, false);
//			INITGRADGAMMA_CHECK(QUADRATIC, false);
			INITGRADGAMMA_CHECK(WENDLAND, false);
		}
	}
	
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("InitGradGamma kernel execution failed");
}

void
updateGamma(	float4*		oldPos,
		float4*		newPos,
		float4*		virtualVel,
		particleinfo*	info,
		float4*		boundElement,
		float4*		oldGam,
		float4*		newGam,
		uint*		neibsList,
		uint		numParticles,
		float		slength,
		float		inflRadius,
		float		virtDt,
		bool		predcor,
		int		kerneltype,
		bool		periodicbound)
{
	int numThreads = min(BLOCK_SIZE_FORCES, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);
	
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundElement, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, virtualVel, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, gamTex, oldGam, numParticles*sizeof(float4)));
	
	//execute kernel
	if(predcor) {
		if (periodicbound) {
			switch (kerneltype) {
				UPDATEGAMMAPRCOR_CHECK(CUBICSPLINE, true);
//				UPDATEGAMMAPRCOR_CHECK(QUADRATIC, true);
				UPDATEGAMMAPRCOR_CHECK(WENDLAND, true);
			}
		} else {
			switch (kerneltype) {
				UPDATEGAMMAPRCOR_CHECK(CUBICSPLINE, false);
//				UPDATEGAMMAPRCOR_CHECK(QUADRATIC, false);
				UPDATEGAMMAPRCOR_CHECK(WENDLAND, false);
			}
		}
	}
	else {
		if (periodicbound) {
			switch (kerneltype) {
				UPDATEGAMMA_CHECK(CUBICSPLINE, true);
//				UPDATEGAMMA_CHECK(QUADRATIC, true);
				UPDATEGAMMA_CHECK(WENDLAND, true);
			}
		} else {
			switch (kerneltype) {
				UPDATEGAMMA_CHECK(CUBICSPLINE, false);
//				UPDATEGAMMA_CHECK(QUADRATIC, false);
				UPDATEGAMMA_CHECK(WENDLAND, false);
			}
		}
	}

	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));
	CUDA_SAFE_CALL(hipUnbindTexture(gamTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("UpdateGamma kernel execution failed");
}

void
updatePositions(	float4*		oldPos,
			float4*		newPos,
			float4*		virtualVel,
			particleinfo*	info,
			float		virtDt,
			uint		numParticles)
{
	int numThreads = min(BLOCK_SIZE_FORCES, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, virtualVel, numParticles*sizeof(float4)));

	//execute kernel
	cuforces::updatePositionsDevice<<<numBlocks, numThreads>>>(newPos, virtDt, numParticles);

	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("UpdatePositions kernel execution failed");
}

void
updateBoundValues(	float4*		oldVel,
			float*		oldPressure,
			vertexinfo*	vertices,
			particleinfo*	info,
			uint		numParticles,
			bool		initStep)
{
	int numThreads = min(BLOCK_SIZE_FORCES, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, vertTex, vertices, numParticles*sizeof(vertexinfo)));

	//execute kernel
	cuforces::updateBoundValuesDevice<<<numBlocks, numThreads>>>(oldVel, oldPressure, numParticles, initStep);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(vertTex));

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("UpdateBoundValues kernel execution failed");
}

void
dynamicBoundConditions(	const float4*		oldPos,
			float4*			oldVel,
			float*			oldPressure,
			const particleinfo*	info,
			const uint*		neibsList,
			const uint		numParticles,
			const float		slength,
			const int		kerneltype,
			const float		influenceradius,
			const bool		periodicbound)
{
	int dummy_shared = 0;

	int numThreads = min(BLOCK_SIZE_SHEPARD, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	// execute the kernel
	if (periodicbound) {
		switch (kerneltype) {
			DYNBOUNDARY_CHECK(CUBICSPLINE, true);
//			DYNBOUNDARY_CHECK(QUADRATIC, true);
			DYNBOUNDARY_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			DYNBOUNDARY_CHECK(CUBICSPLINE, false);
//			DYNBOUNDARY_CHECK(QUADRATIC, false);
			DYNBOUNDARY_CHECK(WENDLAND, false);
		}
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("DynamicBoundConditions kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

}

void
calcProbe(	float4*			oldPos,
		float4*			oldVel,
		float*			oldPressure,
		const particleinfo*	info,
		const uint*		neibsList,
		const uint		numParticles,
		const float		slength,
		const int		kerneltype,
		const float		influenceradius,
		const bool		periodicbound)
{
	int dummy_shared = 0;

	int numThreads = min(BLOCK_SIZE_SHEPARD, numParticles);
	int numBlocks = (int) ceil(numParticles / (float) numThreads);

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	// execute the kernel
	if (periodicbound) {
		switch (kerneltype) {
			CALCPROBE_CHECK(CUBICSPLINE, true);
//			CALCPROBE_CHECK(QUADRATIC, true);
			CALCPROBE_CHECK(WENDLAND, true);
		}
	} else {
		switch (kerneltype) {
			CALCPROBE_CHECK(CUBICSPLINE, false);
//			CALCPROBE_CHECK(QUADRATIC, false);
			CALCPROBE_CHECK(WENDLAND, false);
		}
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("CalcProbe kernel execution failed");

	#if (__COMPUTE__ < 20)
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

}

} // extern "C"

#undef KERNEL_CHECK
#undef KERNEL_SWITCH
#undef VISC_CHECK
#undef VISC_SWITCH
#undef XSPH_CHECK
#undef SHEPARD_CHECK
#undef MLS_CHECK
#undef SPS_CHECK
#undef KEPS_CHECK
#undef VORT_CHECK
#undef TEST_CHECK
#undef SURFACE_CHECK
#undef INITGRADGAMMA_CHECK
#undef UPDATEGAMMA_CHECK

/* These were defined in forces_kernel.cu */
#undef _FORCES_KERNEL_NAME
#undef FORCES_KERNEL_NAME
