#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */
// TODO :
// We can also plan to have separate arrays for boundary parts
// one for the fixed boundary that is sorted only one time in the simulation
// an other one for moving boundary that will be sort with fluid particle
// and a last one for fluid particles. In this way we will compute interactions
// only on fluid particles.

#ifndef _BUILDNEIBS_KERNEL_
#define _BUILDNEIBS_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "vector_math.h"
// CELLTYPE_MASK_*
#include "multi_gpu_defines.h"

namespace cuneibs {
__constant__ uint d_maxneibsnum;
__constant__ idx_t d_neiblist_stride;
__device__ int d_numInteractions;
__device__ int d_maxNeibs;

#include "cellgrid.h"

/// Clamp grid position to edges according to periodicity
/*! This function clamp grid position to edges according to the chosen
 * periodicity, returns the new grid position and update the grid offset.
 *
 *	\param[in] gridPos : grid position to be clamped
 *	\param[in] gridOffset : grid offset
 *	\param[out] toofar : has the gridPos been clamped when the offset was of more than 1 cell?
 *
 * 	\pparam periodicbound : use periodic boundaries (0 ... 7)
 *
 * 	\return : new grid position
 */
// TODO: verify periodicity along multiple axis
template <Periodicity periodicbound>
__device__ __forceinline__ int3
clampGridPos(const int3& gridPos, int3& gridOffset, bool *toofar)
{
	int3 newGridPos = gridPos + gridOffset;
	// For the axis involved in periodicity the new grid position reflects
	// the periodicity and should not be clamped and the grid offset remains
	// unchanged.
	// For the axis not involved in periodicity the new grid position
	// is equal to the clamped old one and the grid offset is updated.

	// periodicity in x
	if (periodicbound & PERIODIC_X) {
		if (newGridPos.x < 0) newGridPos.x += d_gridSize.x;
		if (newGridPos.x >= d_gridSize.x) newGridPos.x -= d_gridSize.x;
	} else {
		newGridPos.x = min(max(0, newGridPos.x), d_gridSize.x-1);
		if (abs(gridOffset.x) > 1 && newGridPos.x == gridPos.x)
			*toofar = true;
		gridOffset.x = newGridPos.x - gridPos.x;
	}

	// periodicity in y
	if (periodicbound & PERIODIC_Y) {
		if (newGridPos.y < 0) newGridPos.y += d_gridSize.y;
		if (newGridPos.y >= d_gridSize.y) newGridPos.y -= d_gridSize.y;
	} else {
		newGridPos.y = min(max(0, newGridPos.y), d_gridSize.y-1);
		if (abs(gridOffset.y) > 1 && newGridPos.y == gridPos.y)
			*toofar = true;
		gridOffset.y = newGridPos.y - gridPos.y;
	}

	// periodicity in z
	if (periodicbound & PERIODIC_Z) {
		if (newGridPos.z < 0) newGridPos.z += d_gridSize.z;
		if (newGridPos.z >= d_gridSize.z) newGridPos.z -= d_gridSize.z;
	} else {
		newGridPos.z = min(max(0, newGridPos.z), d_gridSize.z-1);
		if (abs(gridOffset.z) > 1 && newGridPos.z == gridPos.z)
			*toofar = true;
		gridOffset.z = newGridPos.z - gridPos.z;
	}

	return newGridPos;
}

/// Clamp grid position to edges without periodicity
/*! This function clamp grid position to edges according and
 * returns the new grid position and an updated grid offset.
 *
 *	\param[in] gridPos : grid position to be clamped
 *	\param[in/out] gridOffset : grid offset
 *	\param[out] toofar : has the gridPos been clamped when the offset was of more than 1 cell?
 *
 * 	\return : new grid position
 */
template <>
__device__ __forceinline__ int3
clampGridPos<PERIODIC_NONE>(const int3& gridPos, int3& gridOffset, bool *toofar)
{
	int3 newGridPos = gridPos + gridOffset;

	// Without periodicity the new grid position is clamped to edges
	newGridPos.x = min(max(0, newGridPos.x), d_gridSize.x-1);
	newGridPos.y = min(max(0, newGridPos.y), d_gridSize.y-1);
	newGridPos.z = min(max(0, newGridPos.z), d_gridSize.z-1);
	if ((abs(gridOffset.x) > 1 && newGridPos.x == gridPos.x) ||
		(abs(gridOffset.y) > 1 && newGridPos.y == gridPos.y) ||
		(abs(gridOffset.z) > 1 && newGridPos.z == gridPos.z))
		*toofar = true;

	// In case of change in grid position the grid offset is updated
	gridOffset = newGridPos - gridPos;

	return newGridPos;
}

/// Updates particles hash value of particles and prepare the index table
/*! This kernel should be called before the sort. It
 * 		- updates hash values and relative positions for fluid and
 * 		object particles
 * 		- fill the particle's indexes array with current index
 *
 *	\param[in,out] posArray : particle's positions
 *	\param[in,out] particleHash : particle's hashes
 *	\param[out] particleIndex : particle's indexes
 *	\param[in] particleInfo : particle's informations
 *	\param[in] numParticles : total number of particles
 *
 *	\pparam periodicbound : use periodic boundaries (0 ... 7)
 */
#define MOVINGNOTFLUID (PISTONPART | PADDLEPART | GATEPART | OBJECTPART | VERTEXPART) //TODO-AM the *PART defines are not flags
template <Periodicity periodicbound>
__global__ void
__launch_bounds__(BLOCK_SIZE_CALCHASH, MIN_BLOCKS_CALCHASH)
calcHashDevice(float4*			posArray,		///< particle's positions (in, out)
			   hashKey*			particleHash,	///< particle's hashes (in, out)
			   uint*			particleIndex,	///< particle's indexes (out)
			   const particleinfo*	particelInfo,	///< particle's informations (in)
			   uint				*compactDeviceMap,
			   const uint		numParticles)	///< total number of particles
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// Getting new pos relative to old cell
	float4 pos = posArray[index];
	const particleinfo info = particelInfo[index];

	// we compute new hash only for fluid and moving not fluid particles (object, moving boundaries)
	if ((FLUID(info) || (type(info) & MOVINGNOTFLUID))) {
		// Getting the old grid hash
		uint gridHash = cellHashFromParticleHash( particleHash[index] );

		// Getting grid address of old cell (computed from old hash)
		const int3 gridPos = calcGridPosFromCellHash(gridHash);

		// Computing grid offset from new pos relative to old hash
		int3 gridOffset = make_int3(floor((as_float3(pos) + 0.5f*d_cellSize)/d_cellSize));

		// has the particle flown out of the domain by more than a cell? clamping
		// its position will set this to true if necessary
		bool toofar = false;
		// Compute new grid pos relative to cell, adjust grid offset and compute new cell hash
		gridHash = calcGridHash(clampGridPos<periodicbound>(gridPos, gridOffset, &toofar));

		// mark the cell as inner/outer and/or edge by setting the high bits
		// the value in the compact device map is a CELLTYPE_*_SHIFTED, so 32 bit with high bits set
		if (compactDeviceMap)
			gridHash |= compactDeviceMap[gridHash];

		// Adjust position
		as_float3(pos) -= gridOffset*d_cellSize;
		// if the particle would have flown out of the domain by more than a cell, disable it
		if (toofar)
			disable_particle(pos);

		// mark with special hash if inactive
		if (INACTIVE(pos))
			gridHash = CELL_HASH_MAX;

		// Store grid hash, particle index and position relative to cell
		particleHash[index] = makeParticleHash(gridHash, info);
		posArray[index] = pos;
	}



	// Preparing particle index array for the sort phase
	particleIndex[index] = index;
}

// Similar to calcHash but specific for 1st iteration in MULTI_DEVICE simulations: does not change the cellHash,
// but only sets the high bits according to the compact device map. also, initializes particleIndex
__global__ void
__launch_bounds__(BLOCK_SIZE_CALCHASH, MIN_BLOCKS_CALCHASH)
fixHashDevice(hashKey*			particleHash,	///< particle's hashes (in, out)
			   uint*			particleIndex,	///< particle's indexes (out)
			   const particleinfo*	particelInfo,	///< particle's informations (in)
			   uint				*compactDeviceMap,
			   const uint		numParticles)	///< total number of particles
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = particelInfo[index];

	// We compute new hash only for fluid and moving not fluid particles (object, moving boundaries).
	// Also, if particleHash is NULL we just want to set particleIndex (see comment in GPUWorker::kernel_calcHash())
	if ((FLUID(info) || (type(info) & MOVINGNOTFLUID)) && particleHash) {

		uint gridHash = cellHashFromParticleHash( particleHash[index] );

		// mark the cell as inner/outer and/or edge by setting the high bits
		// the value in the compact device map is a CELLTYPE_*_SHIFTED, so 32 bit with high bits set
		if (compactDeviceMap)
			particleHash[index] = particleHash[index] | ((hashKey)compactDeviceMap[gridHash] << 32);
	}

	// Preparing particle index array for the sort phase
	particleIndex[index] = index;
}

#undef MOVINGNOTFLUID

/// Reorders particles data after the sort and updates cells informations
/*! This kernel should be called after the sort. It
 * 		- computes the index of the first and last particle of
 * 		each grid cell
 * 		- reorders the particle's data (position, velocity, ...)
 * 		according to particles index that have been previously
 * 		sorted during the sort phase
 *
 *	\param[out] cellStart : index of cells first particle
 *	\param[out] cellEnd : index of cells last particle
 *	\param[out] sortedPos : new sorted particle's positions
 *	\param[out] sortedVel : new sorted particle's velocities
 *	\param[out] sortedInfo : new sorted particle's informations
 *	\param[in] particleHash : previously sorted particle's hashes
 *	\param[in] particleIndex : previously sorted particle's indexes
 *	\param[in] numParticles : total number of particles
 *
 * In order to avoid WAR issues we use double buffering : the unsorted data
 * are read trough texture fetches and the sorted one written in a coalesced
 * way in global memory.
 */
__global__
__launch_bounds__(BLOCK_SIZE_REORDERDATA, MIN_BLOCKS_REORDERDATA)
void reorderDataAndFindCellStartDevice( uint*			cellStart,		///< index of cells first particle (out)
										uint*			cellEnd,		///< index of cells last particle (out)
										uint*			segmentStart,
										float4*			sortedPos,		///< new sorted particle's positions (out)
										float4*			sortedVel,		///< new sorted particle's velocities (out)
										particleinfo*	sortedInfo,		///< new sorted particle's informations (out)
										float4*			sortedBoundElements,	// output: sorted boundary elements
										float4*			sortedGradGamma,	// output: sorted gradient gamma
										vertexinfo*		sortedVertices,		// output: sorted vertices
										float*			sortedTKE,			// output: k for k-e model
										float*			sortedEps,			// output: e for k-e model
										float*			sortedTurbVisc,		// output: eddy viscosity
										float4*			sortedEulerVel,		// output: sorted euler vel
										const hashKey*	particleHash,	///< previously sorted particle's hashes (in)
										const uint*		particleIndex,	///< previously sorted particle's hashes (in)
										const uint		numParticles,	///< total number of particles
										uint*			newNumParticles)	// output: number of active particles
{
	// Shared hash array of dimension blockSize + 1
	extern __shared__ uint sharedHash[];

	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	// initialize segmentStarts
	if (segmentStart && index < 4) segmentStart[index] = EMPTY_SEGMENT;

	uint cellHash;
	// Handle the case when number of particles is not multiple of block size
	if (index < numParticles) {
		// To find where cells start/end we only need the cell part of the hash.
		// Note: we do not reset the high bits since we need them to find the segments
		// (aka where the outer particles begin)
		cellHash = cellHashFromParticleHash(particleHash[index], true);

		// Load hash data into shared memory so that we can look
		// at neighboring particle's hash value without loading
		// two hash values per thread
		sharedHash[threadIdx.x + 1] = cellHash;

		if (index > 0 && threadIdx.x == 0) {
			// first thread in block must load neighbor particle hash
			sharedHash[0] = cellHashFromParticleHash(particleHash[index - 1], true);
		}
	}

	__syncthreads();

	if (index < numParticles) {
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell
		// or the first inactive particle.
		// Store the index of this particle as the new cell start and as
		// the previous cell end

		// Note: we need to reset the high bits of the cell hash if the particle hash is 64 bits wide
		// everytime we use a cell hash to access an element of CellStart or CellEnd

		if (index == 0 || cellHash != sharedHash[threadIdx.x]) {

			// new cell, otherwise, it's the number of active particles (short hash: compare with 32 bits max)
			if (cellHash != CELL_HASH_MAX)
				// if it isn't an inactive particle, it is also the start of the cell
				cellStart[cellHash & CELLTYPE_BITMASK] = index;
			else
				*newNumParticles = index;

			// If it isn't the first particle, it must also be the end of the previous cell
			if (index > 0)
				cellEnd[sharedHash[threadIdx.x] & CELLTYPE_BITMASK] = index;
		}

		// if we are an inactive particle, we're done (short hash: compare with 32 bits max)
		if (cellHash == CELL_HASH_MAX)
			return;

		if (index == numParticles - 1) {
			// ditto
			cellEnd[cellHash & CELLTYPE_BITMASK] = index + 1;
			*newNumParticles = numParticles;
		}

		if (segmentStart) {
			// if segment start is given, hash key size is 64 and we detect the segments
			uchar curr_type = cellHash >> 30;
			uchar prev_type = sharedHash[threadIdx.x] >> 30;
			if (index == 0 || curr_type != prev_type)
				segmentStart[curr_type] = index;
		}

		// Now use the sorted index to reorder particle's data
		const uint sortedIndex = particleIndex[index];
		const float4 pos = tex1Dfetch(posTex, sortedIndex);
		const float4 vel = tex1Dfetch(velTex, sortedIndex);
		const particleinfo info = tex1Dfetch(infoTex, sortedIndex);

		sortedPos[index] = pos;
		sortedVel[index] = vel;
		sortedInfo[index] = info;

		if (sortedBoundElements) {
			sortedBoundElements[index] = tex1Dfetch(boundTex, sortedIndex);
		}

		if (sortedGradGamma) {
			sortedGradGamma[index] = tex1Dfetch(gamTex, sortedIndex);
		}

		if (sortedVertices) {
			if (BOUNDARY(info)) {
				const vertexinfo vertices = tex1Dfetch(vertTex, sortedIndex);
				sortedVertices[index] = make_vertexinfo(
					vertices.x,
					vertices.y,
					vertices.z,
					vertices.w);
			}
			else
				sortedVertices[index] = make_vertexinfo(0, 0, 0, 0);
		}

		if (sortedTKE) {
			sortedTKE[index] = tex1Dfetch(keps_kTex, sortedIndex);
		}

		if (sortedEps) {
			sortedEps[index] = tex1Dfetch(keps_eTex, sortedIndex);
		}

		if (sortedTurbVisc) {
			sortedTurbVisc[index] = tex1Dfetch(tviscTex, sortedIndex);
		}

		if (sortedEulerVel) {
			sortedEulerVel[index] = tex1Dfetch(eulerVelTex, sortedIndex);
		}

	}
}

/// Update ID-to-particleIndex lookup table (BUFFER_VERTIDINDEX)
/*! This kernel should be called after the reorder.
 *
 *	\param[in] particleInfo : particleInfo
 *	\param[out] vertIDToIndex : ID-to-particleIndex lookup table, overwritten
 *	\param[in] numParticles : total number of particles
 */
__global__
__launch_bounds__(BLOCK_SIZE_REORDERDATA, MIN_BLOCKS_REORDERDATA)
void updateVertIDToIndexDevice(	particleinfo*	particleInfo,	///< particle's informations
								uint*			vertIDToIndex,	///< vertIDToIndex array (out)
								const uint		numParticles)	///< total number of particles
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	// Handle the case when number of particles is not multiple of block size
	if (index >= numParticles)
		return;

	// assuming vertIDToIndex is allocated, since this kernel is called only with SA bounds
	particleinfo info = particleInfo[index];

	// only vertex particles need to have this information, it should not be done
	// fluid particles as their ids can grow and cause buffer overflows
	if(VERTEX(info))
		// as the vertex particles never change their id (which is <= than the initial
		// particle count, this buffer does not overflow
		vertIDToIndex[ id(info) ] = index;
}

/// Compute the grid position for a neighbor cell
/*! This function computes the grid position for a neighbor cell,
 * according to periodicity.
 *
 * Returns true if the new cell is in the domain, false otherwise.
 */
template <Periodicity periodicbound>
__device__ __forceinline__ bool
calcNeibCell(
		int3 &gridPos, ///< current grid position
		int3 const& gridOffset) ///< cell offset from current grid position
{
	// Compute the grid position of the current cell
	gridPos += gridOffset;

	// With periodic boundary when the neighboring cell grid position lies
	// outside the domain size we wrap it to the d_gridSize or 0 according
	// with the chosen periodicity
	// TODO: verify periodicity along multiple axis
	if (periodicbound) {
		// Periodicity along x axis
		if (gridPos.x < 0) {
			if (periodicbound & PERIODIC_X)
				gridPos.x = d_gridSize.x - 1;
			else
				return false;
		}
		else if (gridPos.x >= d_gridSize.x) {
			if (periodicbound & PERIODIC_X)
				gridPos.x = 0;
			else
				return false;
		}

		// Periodicity along y axis
		if (gridPos.y < 0) {
			if (periodicbound & PERIODIC_Y)
				gridPos.y = d_gridSize.y - 1;
			else
				return false;
		}
		else if (gridPos.y >= d_gridSize.y) {
			if (periodicbound & PERIODIC_Y)
				gridPos.y = 0;
			else
				return false;
		}

		// Periodicity along z axis
		if (gridPos.z < 0) {
			if (periodicbound & PERIODIC_Z)
				gridPos.z = d_gridSize.z - 1;
			else
				return false;
		}
		else if (gridPos.z >= d_gridSize.z) {
			if (periodicbound & PERIODIC_Z)
				gridPos.z = 0;
			else
				return false;
		}
	}
	// Without periodic boundary when the neighboring cell grid position lies
	// outside the domain size there is nothing to do
	else {
		if ((gridPos.x < 0) || (gridPos.x >= d_gridSize.x) ||
			(gridPos.y < 0) || (gridPos.y >= d_gridSize.y) ||
			(gridPos.z < 0) || (gridPos.z >= d_gridSize.z))
				return false;
	}
	// if we get here, the new gridPos was computed correctly, we are
	// still in the domain
	return true;

}

/// variables found in all specializations of neibsInCell
struct common_niC_vars
{
	const	uint	gridHash;		// hash value of grid position
	const	uint	bucketStart;	// index of first particle in cell
	const	uint	bucketEnd;		// index of last particle in cell

	__device__ __forceinline__
	common_niC_vars(int3 const& gridPos) :
		gridHash(calcGridHash(gridPos)),
		bucketStart(tex1Dfetch(cellStartTex, gridHash)),
		bucketEnd(tex1Dfetch(cellEndTex, gridHash))
	{}
};

/// variables found in use_sa_boundary specialization of neibsInCell
struct sa_boundary_niC_vars
{
	vertexinfo	vertices;
	const	float4		boundElement;
	const	uint		j;
	const	float4		coord2;

	__device__ __forceinline__
	sa_boundary_niC_vars(const uint index, buildneibs_params<true> const& bparams) :
		vertices(tex1Dfetch(vertTex, index)),
		boundElement(tex1Dfetch(boundTex, index)),
		// j is 0, 1 or 2 depending on which is smaller (in magnitude) between
		// boundElement.{x,y,z}
		j(
			(fabs(boundElement.z) < fabs(boundElement.y) &&
			fabs(boundElement.z) < fabs(boundElement.x)) ? 2 :
			(fabs(boundElement.y) < fabs(boundElement.x) ? 1 : 0)
		 ),
		// compute second coordinate which is equal to n_s x e_j
		coord2(
			j == 0 ?
			make_float4(0.0f, boundElement.z, -boundElement.y, 0.0f) :
			j == 1 ?
			make_float4(-boundElement.z, 0.0f, boundElement.x, 0.0f) :
			// j == 2
			make_float4(boundElement.y, -boundElement.x, 0.0f, 0.0f)
			)
		{
			// here local copy of part IDs of vertices are replaced by the correspondent part indices
			vertices.x = bparams.vertIDToIndex[vertices.x];
			vertices.y = bparams.vertIDToIndex[vertices.y];
			vertices.z = bparams.vertIDToIndex[vertices.z];
		}
};

/// all neibsInCell variables
template<bool use_sa_boundary>
struct niC_vars :
	common_niC_vars,
	COND_STRUCT(use_sa_boundary, sa_boundary_niC_vars)
{
	__device__ __forceinline__
	niC_vars(int3 const& gridPos, const uint index, buildneibs_params<use_sa_boundary> const& bparams) :
		common_niC_vars(gridPos),
		COND_STRUCT(use_sa_boundary, sa_boundary_niC_vars)(index, bparams)
	{}
};

/// check if a particle at distance relPos is close enough to be considered for neibslist inclusion
template<bool use_sa_boundary>
__device__ __forceinline__
bool isCloseEnough(float3 const& relPos, particleinfo const& neibInfo,
	buildneibs_params<use_sa_boundary> params)
{
	return sqlength(relPos) < params.sqinfluenceradius; // default check: against the influence radius
}

/// SA_BOUNDARY specialization
template<>
__device__ __forceinline__
bool isCloseEnough<true>(float3 const& relPos, particleinfo const& neibInfo,
	buildneibs_params<true> params)
{
	const float rp2(sqlength(relPos));
	// include BOUNDARY neighbors which are a little further than sqinfluenceradius
	return (rp2 < params.sqinfluenceradius ||
		(rp2 < params.boundNlSqInflRad && BOUNDARY(neibInfo)));
}

/// process SA_BOUNDARY segments in neibsInCell
template<bool use_sa_boundary>
__device__ __forceinline__
void process_niC_segment(const uint index, const uint neib_index, float3 const& relPos,
	buildneibs_params<use_sa_boundary> const& params,
	niC_vars<use_sa_boundary> const& var)
{ /* do nothing by default */ }

template<>
__device__ __forceinline__
void process_niC_segment<true>(const uint index, const uint neib_index, float3 const& relPos,
	buildneibs_params<true> const& params,
	niC_vars<true> const& var)
{
	int i = -1;
	if (neib_index == var.vertices.x)
		i = 0;
	else if (neib_index == var.vertices.y)
		i = 1;
	else if (neib_index == var.vertices.z)
		i = 2;
	if (i>-1) {
		// relPosProj is the projected relative position of the vertex to the segment.
		// the first coordinate system is given by the following two vectors:
		// 1. The unit vector e_j, where j is the coordinate for which n_s is minimal
		// 2. The cross product between n_s and e_j
		float2 relPosProj = make_float2(0.0);
		// relPosProj.x = relPos . e_j
		relPosProj.x = var.j==0 ? relPos.x : (var.j==1 ? relPos.y : relPos.z);
		// relPosProj.y = relPos . (n_s x e_j)
		relPosProj.y = dot(relPos, as_float3(var.coord2));
		// save relPosProj in vertPos buffer
		if (i==0)
			params.vertPos0[index] = relPosProj;
		else if (i==1)
			params.vertPos1[index] = relPosProj;
		else
			params.vertPos2[index] = relPosProj;
	}
}

/// Find neighbors in a given cell
/*! This function look for neighbors of the current particle in
 * a given cell
 *
 *	\param[in] buildneibs_params : parameters to buildneibs
 *	\param[in] gridPos : current particle grid position
 *	\param[in] gridOffset : cell offset from current particle cell
 *	\param[in] cell : cell number
 *	\param[in] index : index of the current particle
 *	\param[in] pos : position of the current particle
 *	\param[in, out] neibs_num : current number of neighbors found for current particle
 *
 *	\pparam use_sa_boundary : use SA_BOUNDARY
 *	\pparam periodicbound : use periodic boundaries (0 ... 7)
 *
 * First and last particle index for grid cells and particle's informations
 * are read through texture fetches.
 */
template <bool use_sa_boundary, Periodicity periodicbound>
__device__ __forceinline__ void
neibsInCell(
			buildneibs_params<use_sa_boundary>
				const& params,	///< buildneibs params
			int3			gridPos,	///< current particle grid position
			const int3		gridOffset,	///< cell offset from current particle grid position
			const uchar		cell,		///< cell number (0 ... 26)
			const uint		index,		///< current particle index
			float3			pos,		///< current particle position
			uint&			neibs_num,	///< number of neighbors for the current particle
			const bool		segment)	///< if a segment is searching we are also looking for the three vertices
{
	// Compute the grid position of the current cell, and return if it's
	// outside the domain
	if (!calcNeibCell<periodicbound>(gridPos, gridOffset))
		return;

	niC_vars<use_sa_boundary> var(gridPos, index, params);

	// Return if the cell is empty
	if (var.bucketStart == 0xffffffff)
		return;

	// Substract gridOffset*cellsize to pos so we don't need to do it each time
	// we compute relPos respect to potential neighbor
	pos -= gridOffset*d_cellSize;

	// Iterate over all particles in the cell
	bool encode_cell = true;

	for (uint neib_index = var.bucketStart; neib_index < var.bucketEnd; neib_index++) {

		// no self-interaction
		if (neib_index == index)
			continue;

		const particleinfo neibInfo = tex1Dfetch(infoTex, neib_index);

		// testpoints have a neibs list, but are not considered in the neibs list of other
		// points
		if (TESTPOINTS(neibInfo))
			continue;

		// Compute relative position between particle and potential neighbor
		// NOTE: using as_float3 instead of make_float3 result in a 25% performance loss
		#if (__COMPUTE__ >= 20)
		const float4 neib_pos = params.posArray[neib_index];
		#else
		const float4 neib_pos = tex1Dfetch(posTex, neib_index);
		#endif

		// skip inactive particles
		if (INACTIVE(neib_pos))
			continue;

		const float3 relPos = pos - make_float3(neib_pos);

		// Check if the squared distance is smaller than the squared influence radius
		// used for neighbor list construction
		bool close_enough = isCloseEnough(relPos, neibInfo, params);

		if (close_enough) {
			if (neibs_num < d_maxneibsnum) {
				params.neibsList[neibs_num*d_neiblist_stride + index] =
						neib_index - var.bucketStart + ((encode_cell) ? ENCODE_CELL(cell) : 0);
				encode_cell = false;
			}
			neibs_num++;
		}
		if (segment) {
			process_niC_segment(index, neib_index, relPos, params, var);
		}

	}

	return;
}


/// Builds particles neighbors list
/*! This kernel computes the neighbor's indexes of all particles.
 *
 *	\pparam boundarytype : the boundary type (determines which particles have a neib list)
 *	\pparam periodicbound : use periodic boundaries (0 ... 7)
 *	\pparam neibcount : compute maximum neighbor number (0, 1)
 *
 * First and last particle index for grid cells and particle's informations
 * are read through texture fetches.
 */
template<BoundaryType boundarytype, Periodicity periodicbound, bool neibcount>
__global__ void
__launch_bounds__( BLOCK_SIZE_BUILDNEIBS, MIN_BLOCKS_BUILDNEIBS)
buildNeibsListDevice(buildneibs_params<boundarytype == SA_BOUNDARY> params)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	uint neibs_num = 0;		// Number of neighbors for the current particle

	// rather than nesting if's, use a do { } while (0) loop with breaks for early bailouts
	do {
		if (index >= params.numParticles)
			break;

		// Read particle info from texture
		const particleinfo info = tex1Dfetch(infoTex, index);

		// the neighbor list is only constructed for fluid, testpoint, and object particles.
		// if we use SA_BOUNDARY, also for vertex and boundary particles
		bool build_nl = FLUID(info) || TESTPOINTS(info) || OBJECT(info);
		if (boundarytype == DYN_BOUNDARY)
			build_nl = build_nl || BOUNDARY(info);
		if (boundarytype == SA_BOUNDARY)
			build_nl = build_nl || VERTEX(info) || BOUNDARY(info);
		if (!build_nl)
			break; // nothing to do for other particles

		// Get particle position
		#if (__COMPUTE__ >= 20)
		const float4 pos = params.posArray[index];
		#else
		const float4 pos = tex1Dfetch(posTex, index);
		#endif

		if (INACTIVE(pos))
			break; // no NL for inactive particles

		const float3 pos3 = make_float3(pos);

		// Get particle grid position computed from particle hash
		const int3 gridPos = calcGridPosFromParticleHash(params.particleHash[index]);

		for(int z=-1; z<=1; z++) {
			for(int y=-1; y<=1; y++) {
				for(int x=-1; x<=1; x++) {
					neibsInCell<boundarytype == SA_BOUNDARY, periodicbound>(params,
						gridPos,
						make_int3(x, y, z),
						(x + 1) + (y + 1)*3 + (z + 1)*9,
						index,
						pos3,
						neibs_num,
						BOUNDARY(info));
				}
			}
		}
	} while (0);

	// Setting the end marker. Must be done here so that
	// particles for which the neighbor list is not built actually
	// have an empty neib list. Otherwise, particles which are
	// marked inactive will keep their old neiblist.
	if (index < params.numParticles && neibs_num < d_maxneibsnum) {
		params.neibsList[neibs_num*d_neiblist_stride + index] = 0xffff;
	}

	if (neibcount) {
		// Shared memory reduction of per block maximum number of neighbors
		__shared__ volatile uint sm_neibs_num[BLOCK_SIZE_BUILDNEIBS];
		__shared__ volatile uint sm_neibs_max[BLOCK_SIZE_BUILDNEIBS];

		sm_neibs_num[threadIdx.x] = neibs_num;
		sm_neibs_max[threadIdx.x] = neibs_num;
		__syncthreads();

		uint i = blockDim.x/2;
		while (i != 0) {
			if (threadIdx.x < i) {
				sm_neibs_num[threadIdx.x] += sm_neibs_num[threadIdx.x + i];
				const float n1 = sm_neibs_max[threadIdx.x];
				const float n2 = sm_neibs_max[threadIdx.x + i];
				if (n2 > n1)
					sm_neibs_max[threadIdx.x] = n2;
			}
			__syncthreads();
			i /= 2;
		}

		if (!threadIdx.x) {
			atomicAdd(&d_numInteractions, sm_neibs_num[0]);
			atomicMax(&d_maxNeibs, sm_neibs_max[0]);
		}
	}
	return;
}
}
#endif
