#include "hip/hip_runtime.h"
/*  Copyright 2011 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

	Istituto de Nazionale di Geofisica e Vulcanologia
          Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */
// TODO :
// We can also plan to have separate arrays for boundary parts
// one for the fixed boundary that is sorted only one time in the simulation
// an other one for moving boundary that will be sort with fluid particle
// and a last one for fluid particles. In this way we will compute interactions
// only on fluid particles.

#ifndef _BUILDNEIBS_KERNEL_
#define _BUILDNEIBS_KERNEL_

#include "particledefine.h"
#include "textures.cuh"

namespace cuneibs {
__constant__ uint d_maxneibsnum;
__constant__ uint d_maxneibsnum_time_neibindexinterleave;
__device__ int d_numInteractions;
__device__ int d_maxNeibs;
__constant__ float3 d_dispvect;

// calculate position in uniform grid
__device__ __forceinline__ int3
calcGridPos(float3			pos,
			const float3	worldOrigin,
			const float3	cellSize)
{
	int3 gridPos;
	gridPos.x = floor((pos.x - worldOrigin.x) / cellSize.x);
	gridPos.y = floor((pos.y - worldOrigin.y) / cellSize.y);
	gridPos.z = floor((pos.z - worldOrigin.z) / cellSize.z);

	return gridPos;
}


// calculate address in grid from position (clamping to edges)
__device__ __forceinline__ uint
calcGridHash(int3			gridPos,
			 const uint3	gridSize)
{
	gridPos.x = max(0, min(gridPos.x, gridSize.x-1));
	gridPos.y = max(0, min(gridPos.y, gridSize.y-1));
	gridPos.z = max(0, min(gridPos.z, gridSize.z-1));
	return INTMUL(INTMUL(gridPos.z, gridSize.y), gridSize.x) + INTMUL(gridPos.y, gridSize.x) + gridPos.x;
}


// calculate grid hash value for each particle
__global__ void
__launch_bounds__(BLOCK_SIZE_CALCHASH, MIN_BLOCKS_CALCHASH)
calcHashDevice(const float4*	posArray,
			   uint*			particleHash,
			   uint*			particleIndex,
			   const uint3		gridSize,
			   const float3		cellSize,
			   const float3		worldOrigin,
			   const uint		numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const float4 pos = posArray[index];

	// get address in grid
	const int3 gridPos = calcGridPos(make_float3(pos), worldOrigin, cellSize);
	const uint gridHash = calcGridHash(gridPos, gridSize);

	// store grid hash and particle index
	particleHash[index] = gridHash;
	particleIndex[index] = index;
}

__global__
__launch_bounds__(BLOCK_SIZE_REORDERDATA, MIN_BLOCKS_REORDERDATA)
void inverseParticleIndexDevice (	uint*	particleIndex,
					uint*	inversedParticleIndex,
					uint	numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	
	if (index < numParticles) {
		int oldindex = particleIndex[index];
		inversedParticleIndex[oldindex] = index;
	}
}
__global__
__launch_bounds__(BLOCK_SIZE_REORDERDATA, MIN_BLOCKS_REORDERDATA)
void reorderDataAndFindCellStartDevice( uint*			cellStart,		// output: cell start index
										uint*			cellEnd,		// output: cell end index
										float4*			sortedPos,		// output: sorted positions
										float4*			sortedVel,		// output: sorted velocities
										particleinfo*		sortedInfo,		// output: sorted info
										float4*			sortedBoundElements,	// output: sorted boundary elements
										float4*			sortedGradGamma,	// output: sorted gradient gamma
										vertexinfo*		sortedVertices,		// output: sorted vertices
										uint*			particleHash,	// input: sorted grid hashes
										uint*			particleIndex,	// input: sorted particle indices
										uint			numParticles,
										uint*			inversedParticleIndex)
{
	extern __shared__ uint sharedHash[];	// blockSize + 1 elements

	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	uint hash;
	// handle case when no. of particles not multiple of block size
	if (index < numParticles) {
		hash = particleHash[index];

		// Load hash data into shared memory so that we can look
		// at neighboring particle's hash value without loading
		// two hash values per thread
		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0) {
			// first thread in block must load neighbor particle hash
			sharedHash[0] = particleHash[index-1];
			}
	}

	__syncthreads();

	if (index < numParticles) {
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell,
		// so store the index of this particle in the cell.
		// As it isn't the first particle, it must also be the cell end of
		// the previous particle's cell

		if (index == 0 || hash != sharedHash[threadIdx.x]) {
			cellStart[hash] = index;
			if (index > 0)
				cellEnd[sharedHash[threadIdx.x]] = index;
			}

		if (index == numParticles - 1) {
			cellEnd[hash] = index + 1;
			}

		// Now use the sorted index to reorder the pos and vel data
		uint sortedIndex = particleIndex[index];
		float4 pos = tex1Dfetch(posTex, sortedIndex);	   // macro does either global read or texture fetch
		float4 vel = tex1Dfetch(velTex, sortedIndex);	   // see particles_kernel.cuh
		particleinfo info = tex1Dfetch(infoTex, sortedIndex);
		float4 boundelement = tex1Dfetch(boundTex, sortedIndex);
		float4 gradgamma = tex1Dfetch(gamTex, sortedIndex);
		vertexinfo vertices = tex1Dfetch(vertTex, sortedIndex);

		sortedPos[index] = pos;
		sortedVel[index] = vel;
		sortedInfo[index] = info;
		sortedBoundElements[index] = boundelement;
		sortedGradGamma[index] = gradgamma;
		
		sortedVertices[index].x = inversedParticleIndex[vertices.x];
		sortedVertices[index].y = inversedParticleIndex[vertices.y];
		sortedVertices[index].z = inversedParticleIndex[vertices.z];
	}
}


template <bool periodicbound>
__device__ __forceinline__ void
neibsInCell(
			#if (__COMPUTE__ >= 20)			
			const float4*	posArray,
			#endif
			int3			gridPos,
			const uint		index,
			const float3	pos,
			const uint3		gridSize,
			const uint		numParticles,
			const float		sqinfluenceradius,
			uint*			neibsList,
			uint&			neibs_num,
			const uint		lane,
			const uint		offset)
{
	int3 periodic = make_int3(0);
	if (periodicbound) {
		if (gridPos.x < 0) {
			if (d_dispvect.x) {
				gridPos.x = gridSize.x;
				periodic.x = 1;
			} else
				return;
		} else if (gridPos.x >= gridSize.x) {
			if (d_dispvect.x) {
				gridPos.x = 0;
				periodic.x = -1;
			} else
				return;
		}
		if (gridPos.y < 0) {
			if (d_dispvect.y) {
				gridPos.y = gridSize.y;
				periodic.y = 1;
			} else
				return;
		} else if (gridPos.y >= gridSize.y) {
			if (d_dispvect.y) {
				gridPos.y = 0;
				periodic.y = -1;
			} else
				return;
		}
		if (gridPos.z < 0) {
			if (d_dispvect.z) {
				gridPos.z = gridSize.z;
				periodic.z = 1;
			} else
				return;
		} else if (gridPos.z >= gridSize.z) {
			if (d_dispvect.z) {
				gridPos.z = 0;
				periodic.z = -1;
			} else
				return;
		}
	} else {
		if ((gridPos.x < 0) || (gridPos.x >= gridSize.x) ||
			(gridPos.y < 0) || (gridPos.y >= gridSize.y) ||
			(gridPos.z < 0) || (gridPos.z >= gridSize.z))
				return;
	}

	// get hash value of grid position
	const uint gridHash = calcGridHash(gridPos, gridSize);

	// get start of bucket for this cell
	const uint bucketStart = tex1Dfetch(cellStartTex, gridHash);

	if (bucketStart == 0xffffffff)
		return;   // cell empty

	// iterate over particles in this cell
	const uint bucketEnd = tex1Dfetch(cellEndTex, gridHash);
	for(uint neib_index = bucketStart; neib_index < bucketEnd; neib_index++) {

		//Testpoints ( Testpoints are not considered in neighboring list of other particles since they are imaginary particles)
    	const particleinfo info = tex1Dfetch(infoTex, neib_index);
        if (!TESTPOINTS (info)) {
			if (neib_index != index) {			  // check not interacting with self
				#if (__COMPUTE__ >= 20)			
				float3 relPos = pos - make_float3(posArray[neib_index]);
				#else
				float3 relPos = pos - make_float3(tex1Dfetch(posTex, neib_index));
				#endif
				if (periodicbound)
					relPos += periodic*d_dispvect;

				uint mod_index = neib_index;
				if (sqlength(relPos) < sqinfluenceradius) {
					if (periodicbound) {
						if (periodic.x == 1)
							mod_index |= WARPXPLUS;
						else if (periodic.x == -1)
							mod_index |= WARPXMINUS;
						if (periodic.y == 1)
							mod_index |= WARPYPLUS;
						else if (periodic.y == -1)
							mod_index |= WARPYMINUS;
						if (periodic.z == 1)
							mod_index |= WARPZPLUS;
						else if (periodic.z == -1)
							mod_index |= WARPZMINUS;
					}

					if (neibs_num < d_maxneibsnum)
						neibsList[d_maxneibsnum_time_neibindexinterleave*lane + neibs_num*NEIBINDEX_INTERLEAVE + offset] = mod_index;
					neibs_num++;
				}

			}
		} //If  not Testpoints
	}

	return;
}


template<bool periodicbound, bool neibcount>
__global__ void
__launch_bounds__( BLOCK_SIZE_BUILDNEIBS, MIN_BLOCKS_BUILDNEIBS)
buildNeibsListDevice(   
						#if (__COMPUTE__ >= 20)			
						const float4*	posArray,
						#endif
						uint*			neibsList,
						const uint3		gridSize,
						const float3	cellSize,
						const float3	worldOrigin,
						const uint		numParticles,
						const float		sqinfluenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	const uint tid = threadIdx.x;
	const uint lane = index/NEIBINDEX_INTERLEAVE;
	const uint offset = tid & (NEIBINDEX_INTERLEAVE - 1);

	uint neibs_num = 0;

	if (index < numParticles) {
		// read particle info from texture
    	const particleinfo info = tex1Dfetch(infoTex, index);

		// Only fluid particle needs to have a boundary list
		// TODO: this is not true with dynamic boundary particles
		// so change that when implementing dynamics boundary parts
		// This is also not true for "Ferrand et al." boundary model,
		// where vertex particles also need to have a list of neighbours

		// Neighboring list is calculated for testpoints and object points)
		if (FLUID(info) || TESTPOINTS (info) || OBJECT(info)/*TODO: || VERTEX(info) || BOUNDARY(info)*/) {
			// read particle position from global memory or texture according to architecture
			#if (__COMPUTE__ >= 20)
			const float3 pos = make_float3(posArray[index]);
			#else
			const float3 pos = make_float3(tex1Dfetch(posTex, index));
			#endif

			// get address in grid
			const int3 gridPos = calcGridPos(pos, worldOrigin, cellSize);

			// examine only neighboring cells
			for(int z=-1; z<=1; z++) {
				for(int y=-1; y<=1; y++) {
					for(int x=-1; x<=1; x++)
						neibsInCell<periodicbound>(
							#if (__COMPUTE__ >= 20)
							posArray, 
							#endif
							gridPos + make_int3(x, y, z), index, pos, gridSize, numParticles, 
							sqinfluenceradius, neibsList, neibs_num, lane, offset);
				}
			}
		}
		
		if (neibs_num < d_maxneibsnum)
			neibsList[d_maxneibsnum_time_neibindexinterleave*lane + neibs_num*NEIBINDEX_INTERLEAVE + offset] = 0xffffffff;
	}
	
	if (neibcount) {
		// Shared memory reduction of per block maximum number of neighbors
		__shared__ volatile uint sm_neibs_num[BLOCK_SIZE_BUILDNEIBS];
		__shared__ volatile uint sm_neibs_max[BLOCK_SIZE_BUILDNEIBS];

		sm_neibs_num[tid] = neibs_num;	
		sm_neibs_max[tid] = neibs_num;
		__syncthreads();

		uint i = blockDim.x/2;
		while (i != 0) {
			if (tid < i) {
				sm_neibs_num[tid] += sm_neibs_num[tid + i];
				const float n1 = sm_neibs_max[tid];
				const float n2 = sm_neibs_max[tid + i];
				if (n2 > n1)
					sm_neibs_max[tid] = n2;
			}
			__syncthreads();
			i /= 2;
		}

		if (!tid) {
			atomicAdd(&d_numInteractions, sm_neibs_num[0]);
			atomicMax(&d_maxNeibs, sm_neibs_max[0]);
		}
	}
	return;
}
}
#endif
