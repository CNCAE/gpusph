#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */
// TODO :
// We can also plan to have separate arrays for boundary parts
// one for the fixed boundary that is sorted only one time in the simulation
// an other one for moving boundary that will be sort with fluid particle
// and a last one for fluid particles. In this way we will compute interactions
// only on fluid particles.

#ifndef _BUILDNEIBS_KERNEL_
#define _BUILDNEIBS_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "vector_math.h"

namespace cuneibs {
__constant__ uint d_maxneibsnum;
__device__ int d_numInteractions;
__device__ int d_maxNeibs;

#include "cellgrid.h"

/// Clamp grid position to edges according to periodicity
/*! This function clamp grid position to edges according to the chosen
 * periodicity, returns the new grid position and update the grid offset.
 *
 *	\param[in] gridPos : grid position to be clamped
 *	\param[in] gridOffset : grid offset
 *
 * 	\pparam periodicbound : use periodic boundaries (0 ... 7)
 *
 * 	\return : new grid position
 */
// TODO: verify periodicity along multiple axis
template <int periodicbound>
__device__ __forceinline__ int3
clampGridPos(const int3& gridPos, int3& gridOffset)
{
	int3 newGridPos = gridPos + gridOffset;
	// For the axis involved in periodicity the new grid position reflects
	// the periodicity and should not be clamped and the grid offset remains
	// unchanged.
	// For the axis not involved in periodicity the new grid position
	// is equal to the clamped old one and the grid offset is updated.

	// periodicity in x
	if (periodicbound & XPERIODIC) {
		if (newGridPos.x < 0) newGridPos.x += d_gridSize.x;
		if (newGridPos.x >= d_gridSize.x) newGridPos.x -= d_gridSize.x;
	} else {
		newGridPos.x = max(0, min(gridPos.x, d_gridSize.x-1));
		gridOffset.x = newGridPos.x - gridPos.x;
	}

	// periodicity in y
	if (periodicbound & YPERIODIC) {
		if (newGridPos.y < 0) newGridPos.y += d_gridSize.y;
		if (newGridPos.y >= d_gridSize.y) newGridPos.y -= d_gridSize.y;
	} else {
		newGridPos.y = max(0, min(gridPos.y, d_gridSize.y-1));
		gridOffset.y = newGridPos.y - gridPos.y;
	}

	// periodicity in z
	if (periodicbound & ZPERIODIC) {
		if (newGridPos.z < 0) newGridPos.z += d_gridSize.z;
		if (newGridPos.z >= d_gridSize.z) newGridPos.z -= d_gridSize.z;
	} else {
		newGridPos.z = max(0, min(gridPos.z, d_gridSize.z-1));
		gridOffset.z = newGridPos.z - gridPos.z;
	}

	return newGridPos;
}

/// Clamp grid position to edges without periodicity
/*! This function clamp grid position to edges according and
 * returns the new grid position and an updated grid offset.
 *
 *	\param[in] gridPos : grid position to be clamped
 *	\param[in/out] gridOffset : grid offset
 *
 * 	\return : new grid position
 */
template <>
__device__ __forceinline__ int3
clampGridPos<0>(const int3& gridPos, int3& gridOffset)
{
	int3 newGridPos = gridPos + gridOffset;

	// Without periodicity the new grid position is clamped to edges
	newGridPos.x = max(0, min(newGridPos.x, d_gridSize.x-1));
	newGridPos.y = max(0, min(newGridPos.y, d_gridSize.y-1));
	newGridPos.z = max(0, min(newGridPos.z, d_gridSize.z-1));

	// In case of change in grid position the grid offset is updated
	gridOffset = newGridPos - gridPos;

	return newGridPos;
}

/// Updates particles hash value of particles and prepare the index table
/*! This kernel should be called before the sort. It
 * 		- updates hash values and relative positions for fluid and
 * 		object particles
 * 		- fill the particle's indexes array with current index
 *
 *	\param[in,out] posArray : particle's positions
 *	\param[in,out] particleHash : particle's hashes
 *	\param[out] particleIndex : particle's indexes
 *	\param[in] particleInfo : particle's informations
 *	\param[in] numParticles : total number of particles
 *
 *	\pparam periodicbound : use periodic boundaries (0 ... 7)
 */
#define MOVINGNOTFLUID (PISTONPART | PADDLEPART | GATEPART | OBJECTPART | VERTEXPART) //TODO-AM the *PART defines are not flags
template <int periodicbound>
__global__ void
__launch_bounds__(BLOCK_SIZE_CALCHASH, MIN_BLOCKS_CALCHASH)
calcHashDevice(float4*			posArray,		///< particle's positions (in, out)
			   hashKey*			particleHash,	///< particle's hashes (in, out)
			   uint*			particleIndex,	///< particle's indexes (out)
			   const particleinfo*	particelInfo,	///< particle's informations (in)
			   const uint		numParticles)	///< total number of particles
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// Getting new pos relative to old cell
	float4 pos = posArray[index];
	const particleinfo info = particelInfo[index];

	// We compute new hash only for fluid and moving not fluid particles (object, moving boundaries)
	if (FLUID(info) || (type(info) & MOVINGNOTFLUID)) {
	//if (true) {
		// Getting the old grid hash
		hashKey gridHash = particleHash[index];

		// Getting grid address of old cell (computed from old hash)
		const int3 gridPos = calcGridPosFromHash(gridHash);

		// Computing grid offset from new pos relative to old hash
		int3 gridOffset = make_int3(floor((as_float3(pos) + 0.5f*d_cellSize)/d_cellSize));

		// Compute new grid pos relative to cell, adjust grid offset and compute new cell hash
		gridHash = calcGridHash(clampGridPos<periodicbound>(gridPos, gridOffset));
#if HASH_KEY_SIZE >= 64
		gridHash <<= GRIDHASH_BITSHIFT
		gridHash |= id(pinfo[index]);
#endif

		// Adjust position
		as_float3(pos) -= gridOffset*d_cellSize;

		// Store grid hash, particle index and position relative to cell
		particleHash[index] = gridHash;
		posArray[index] = pos;
	}

	// Preparing particle index array for the sort phase
	particleIndex[index] = index;
}
#undef MOVINGNOTFLUID

__global__
__launch_bounds__(BLOCK_SIZE_REORDERDATA, MIN_BLOCKS_REORDERDATA)
void inverseParticleIndexDevice (   uint*   particleIndex,
                    uint*   inversedParticleIndex,
                    uint    numParticles)
{
    const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

    if (index < numParticles) {
        int oldindex = particleIndex[index];
        inversedParticleIndex[oldindex] = index;
    }
}

/// Reorders particles data after the sort and updates cells informations
/*! This kernel should be called after the sort. It
 * 		- computes the index of the first and last particle of
 * 		each grid cell
 * 		- reorders the particle's data (position, velocity, ...)
 * 		according to particles index that have been previously
 * 		sorted during the sort phase
 *
 *	\param[out] cellStart : index of cells first particle
 *	\param[out] cellEnd : index of cells last particle
 *	\param[out] sortedPos : new sorted particle's positions
 *	\param[out] sortedVel : new sorted particle's velocities
 *	\param[out] sortedInfo : new sorted particle's informations
 *	\param[in] particleHash : previously sorted particle's hashes
 *	\param[in] particleIndex : previously sorted particle's indexes
 *	\param[in] numParticles : total number of particles
 *
 * In order to avoid WAR issues we use double buffering : the unsorted data
 * are read trough texture fetches and the sorted one written in a coalesced
 * way in global memory.
 */
__global__
__launch_bounds__(BLOCK_SIZE_REORDERDATA, MIN_BLOCKS_REORDERDATA)
void reorderDataAndFindCellStartDevice( uint*			cellStart,		///< index of cells first particle (out)
										uint*			cellEnd,		///< index of cells last particle (out)
										float4*			sortedPos,		///< new sorted particle's positions (out)
										float4*			sortedVel,		///< new sorted particle's velocities (out)
										particleinfo*	sortedInfo,		///< new sorted particle's informations (out)
										float4*			sortedBoundElements,	// output: sorted boundary elements
										float4*			sortedGradGamma,	// output: sorted gradient gamma
										vertexinfo*		sortedVertices,		// output: sorted vertices
										float*			sortedPressure,		// output: sorted pressure
										float*			sortedTKE,			// output: k for k-e model
										float*			sortedEps,			// output: e for k-e model
										float*			sortedTurbVisc,		// output: eddy viscosity
										float*			sortedStrainRate,	// output: strain rate
										const hashKey*	particleHash,	///< previously sorted particle's hashes (in)
										const uint*		particleIndex,	///< previously sorted particle's hashes (in)
										const uint		numParticles,	///< total number of particles
										const uint*		inversedParticleIndex)
{
	// Shared hash array of dimension blockSize + 1
	extern __shared__ uint sharedHash[];

	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	uint hash;
	// Handle the case when number of particles is not multiple of block size
	if (index < numParticles) {
		hash = (uint)(particleHash[index] >> GRIDHASH_BITSHIFT);

		// Load hash data into shared memory so that we can look
		// at neighboring particle's hash value without loading
		// two hash values per thread
		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0) {
			// first thread in block must load neighbor particle hash
			sharedHash[0] = (uint)(particleHash[index-1] >> GRIDHASH_BITSHIFT);
			}
	}

	__syncthreads();

	if (index < numParticles) {
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell
		// or the first inactive particle.
		// Store the index of this particle as the new cell start and as
		// the previous cell end

		if (index == 0 || hash != sharedHash[threadIdx.x]) {
			cellStart[hash] = index;
			// If it isn't the first particle, it must also be the cell end of
			if (index > 0)
				cellEnd[sharedHash[threadIdx.x]] = index;
		}

		if (index == numParticles - 1)
			cellEnd[hash] = index + 1;

		// Now use the sorted index to reorder particle's data
		const uint sortedIndex = particleIndex[index];
		const float4 pos = tex1Dfetch(posTex, sortedIndex);
		const float4 vel = tex1Dfetch(velTex, sortedIndex);
		const particleinfo info = tex1Dfetch(infoTex, sortedIndex);
		const float4 boundelement = tex1Dfetch(boundTex, sortedIndex);
		const float4 gradgamma = tex1Dfetch(gamTex, sortedIndex);
		const vertexinfo vertices = tex1Dfetch(vertTex, sortedIndex);
		const float pressure = tex1Dfetch(presTex, sortedIndex);

		const float keps_k = tex1Dfetch(keps_kTex, sortedIndex);
		const float keps_e = tex1Dfetch(keps_eTex, sortedIndex);
		const float tvisc = tex1Dfetch(tviscTex, sortedIndex);
		const float strainrate = tex1Dfetch(strainTex, sortedIndex);

		sortedPos[index] = pos;
		sortedVel[index] = vel;
		sortedInfo[index] = info;
		sortedBoundElements[index] = boundelement;
		sortedGradGamma[index] = gradgamma;
		sortedPressure[index] = pressure;
		
		sortedVertices[index].x = inversedParticleIndex[vertices.x];
		sortedVertices[index].y = inversedParticleIndex[vertices.y];
		sortedVertices[index].z = inversedParticleIndex[vertices.z];

		sortedTKE[index] = keps_k;
		sortedEps[index] = keps_e;
		sortedTurbVisc[index] = tvisc;
		sortedStrainRate[index] = strainrate;
	}
}


/// Find neighbors in a given cell
/*! This function look for neighbors of the current particle in
 * a given cell
 *
 *	\param[in] posArray : particle's positions
 *	\param[in] gridPos : current particle grid position
 *	\param[in] gridOffset : cell offset from current particle cell
 *	\param[in] cell : cell number
 *	\param[in] index : index of the current particle
 *	\param[in] pos : position of the current particle
 *	\param[in] numParticles : total number of particles
 *	\param[in] sqinfluenceradius : squared value of the influence radius
 *	\param[out] neibList : neighbor's list
 *	\param[in, out] neibs_num : current number of neighbors found for current particle
 *
 *	\pparam periodicbound : use periodic boundaries (0 ... 7)
 *
 * First and last particle index for grid cells and particle's informations
 * are read trough texture fetches.
 */
template <int periodicbound>
__device__ __forceinline__ void
neibsInCell(
			#if (__COMPUTE__ >= 20)			
			const float4*	posArray,	///< particle's positions (in)
			#endif
			int3			gridPos,	///< current particle grid position
			const int3		gridOffset,	///< cell offset from current particle grid position
			const uchar		cell,		///< cell number (0 ... 26)
			const uint		index,		///< current particle index
			float3			pos,		///< current particle position
			const uint		numParticles,	///< total number of particles
			const float		sqinfluenceradius,	///< squared value of influence radius
			neibdata*		neibsList,	///< neighbor's list (out)
			uint&			neibs_num)	///< number of neighbors for the current particle
{
	// Compute the grid position of the current cell
	gridPos += gridOffset;

	// With periodic boundary when the neighboring cell grid position lies
	// outside the domain size we wrap it to the d_gridSize or 0 according
	// with the chosen periodicity
	// TODO: verify periodicity along multiple axis
	if (periodicbound) {
		// Periodicity along x axis
		if (gridPos.x < 0) {
			if (periodicbound & XPERIODIC)
				gridPos.x = d_gridSize.x - 1;
			else
				return;
		}
		else if (gridPos.x >= d_gridSize.x) {
			if (periodicbound & XPERIODIC)
				gridPos.x = 0;
			else
				return;
		}

		// Periodicity along y axis
		if (gridPos.y < 0) {
			if (periodicbound & YPERIODIC)
				gridPos.y = d_gridSize.y - 1;
			else
				return;
		}
		else if (gridPos.y >= d_gridSize.y) {
			if (periodicbound & YPERIODIC)
				gridPos.y = 0;
			else
				return;
		}

		// Periodicity along z axis
		if (gridPos.z < 0) {
			if (periodicbound & ZPERIODIC)
				gridPos.z = d_gridSize.z - 1;
			else
				return;
		}
		else if (gridPos.z >= d_gridSize.z) {
			if (periodicbound & ZPERIODIC)
				gridPos.z = 0;
			else
				return;
		}
	}
	// Without periodic boundary when the neighboring cell grid position lies
	// outside the domain size there is nothing to do
	else {
		if ((gridPos.x < 0) || (gridPos.x >= d_gridSize.x) ||
			(gridPos.y < 0) || (gridPos.y >= d_gridSize.y) ||
			(gridPos.z < 0) || (gridPos.z >= d_gridSize.z))
				return;
	}

	// Get hash value from grid position
	const uint gridHash = calcGridHash(gridPos);

	// Get the first particle index of the cell
	const uint bucketStart = tex1Dfetch(cellStartTex, gridHash);

	// Return if the cell is empty
	if (bucketStart == 0xffffffff)
		return;

	// Substract gridOffset*cellsize to pos so we don't need to do it each time
	// we compute relPos respect to potential neighbor
	pos -= gridOffset*d_cellSize;

	// Get the last particle index of the cell
	const uint bucketEnd = tex1Dfetch(cellEndTex, gridHash);
	// Iterate over all particles in the cell
	bool encode_cell = true;
	for(uint neib_index = bucketStart; neib_index < bucketEnd; neib_index++) {

		// Test and probe points are not considered in neighboring list of other particles since they are imaginary particles.
    	const particleinfo info = tex1Dfetch(infoTex, neib_index);
        if (!TESTPOINTS (info) && !PROBE(info)) {
        	// Check for self interaction
			if (neib_index != index) {
				// Compute relative position between particle and potential neighbor
				// NOTE: using as_float3 instead of make_float3 result in a 25% performance loss
				#if (__COMPUTE__ >= 20)			
				const float3 relPos = pos - make_float3(posArray[neib_index]);
				#else
				const float3 relPos = pos - make_float3(tex1Dfetch(posTex, neib_index));
				#endif

				// Check if the squared distance is smaller than the squared influence radius
				// used for neighbor list construction
				if (sqlength(relPos) < sqinfluenceradius) {
					if (neibs_num < d_maxneibsnum) {
						neibsList[neibs_num*numParticles + index] =
								neib_index - bucketStart + ((encode_cell) ? ENCODE_CELL(cell) : 0);
						encode_cell = false;
					}
					neibs_num++;
				}

			}
		} // if not Testpoints
	}

	return;
}


/// Builds particles neighbors list
/*! This kernel computes the neighbor's indexes of all particles.
 * In order to have best performance across different compute capabilities
 * particle's positions are read from global memory for compute capability
 * greather or equal to 2.0 and from texture otherwise.
 *
 *	\param[in] posArray : particle's positions
 *	\param[in] particleHash : particle's hashes
 *	\param[out] neibList : neighbor's list
 *	\param[in] numParticles : total number of particles
 *	\param[in] sqinfluenceradius : squared value of the influence radius
 *
 *	\pparam periodicbound : use periodic boundaries (0 ... 7)
 *	\pparam neibcount : compute maximum neighbor number (0, 1)
 *
 * First and last particle index for grid cells and particle's informations
 * are read trough texture fetches.
 */
template<int periodicbound, bool neibcount>
__global__ void
__launch_bounds__( BLOCK_SIZE_BUILDNEIBS, MIN_BLOCKS_BUILDNEIBS)
buildNeibsListDevice(   
						#if (__COMPUTE__ >= 20)			
						const float4*	posArray,				///< particle's positions (in)
						#endif
						const hashKey*	particleHash,			///< particle's hashes (in)
						neibdata*		neibsList,				///< neighbor's list (out)
						const uint		numParticles,			///< total number of particles
						const float		sqinfluenceradius)		///< squared influence radius
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	uint neibs_num = 0;		// Number of neighbors for the current particle

	if (index < numParticles) {
		// Read particle info from texture
    	const particleinfo info = tex1Dfetch(infoTex, index);

		// Only fluid particle needs to have a boundary list
		// TODO: this is not true with dynamic boundary particles
		// so change that when implementing dynamics boundary parts
		// This is also not true for "Ferrand et al." boundary model,
		// where vertex particles also need to have a list of neighbours

		// Neighbor list is build for fluid, object, vertex and probe particles
		if (FLUID(info) || TESTPOINTS (info) || OBJECT(info) || VERTEX(info) || PROBE(info)/*TODO: || BOUNDARY(info)*/) {
			// Get particle position
			#if (__COMPUTE__ >= 20)
			const float3 pos = make_float3(posArray[index]);
			#else
			const float3 pos = make_float3(tex1Dfetch(posTex, index));
			#endif

			// Get particle grid position computed from particle hash
			const int3 gridPos = calcGridPosFromHash(particleHash[index]);

			// Look trough the 26 neighboring cells and the current particle cell
			for(int z=-1; z<=1; z++) {
				for(int y=-1; y<=1; y++) {
					for(int x=-1; x<=1; x++) {
						neibsInCell<periodicbound>(
							#if (__COMPUTE__ >= 20)
							posArray, 
							#endif
							gridPos, make_int3(x, y, z), (x + 1) + (y + 1)*3 + (z + 1)*9, index, pos,
							numParticles, sqinfluenceradius, neibsList, neibs_num);
					}
				}
			}
		}
		
		// Setting the end marker
		if (neibs_num < d_maxneibsnum) {
			neibsList[neibs_num*numParticles + index] = 0xffff;
		}
	}
	
	if (neibcount) {
		// Shared memory reduction of per block maximum number of neighbors
		__shared__ volatile uint sm_neibs_num[BLOCK_SIZE_BUILDNEIBS];
		__shared__ volatile uint sm_neibs_max[BLOCK_SIZE_BUILDNEIBS];

		sm_neibs_num[threadIdx.x] = neibs_num;
		sm_neibs_max[threadIdx.x] = neibs_num;
		__syncthreads();

		uint i = blockDim.x/2;
		while (i != 0) {
			if (threadIdx.x < i) {
				sm_neibs_num[threadIdx.x] += sm_neibs_num[threadIdx.x + i];
				const float n1 = sm_neibs_max[threadIdx.x];
				const float n2 = sm_neibs_max[threadIdx.x + i];
				if (n2 > n1)
					sm_neibs_max[threadIdx.x] = n2;
			}
			__syncthreads();
			i /= 2;
		}

		if (!threadIdx.x) {
			atomicAdd(&d_numInteractions, sm_neibs_num[0]);
			atomicMax(&d_maxNeibs, sm_neibs_max[0]);
		}
	}
	return;
}
}
#endif
